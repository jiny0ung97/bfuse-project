
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(232) conv2d_test_2(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[7424];
  __shared__ float kernel_shared[2048];
  for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
    conv2d_nchw_local[ff_c_inner_init] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 2)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 4)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 6)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 8)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 10)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 12)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_inner_init + 14)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 232) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) % 29) * 2) + ((((int)threadIdx.x) % 116) / 58))) && ((((((int)blockIdx.x) % 29) * 2) + ((((int)threadIdx.x) % 116) / 58)) < 57)) && (1 <= (((int)threadIdx.x) % 58))) && ((((int)threadIdx.x) % 58) < 57)) ? data[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6272) + ((((int)threadIdx.x) / 116) * 3136)) + ((((int)blockIdx.x) % 29) * 112)) + (((((int)threadIdx.x) % 116) / 58) * 56)) + (((int)threadIdx.x) % 58)) - 57)] : 0.000000e+00f);
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 29) + (((int)threadIdx.x) >> 3)) < 128) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 464) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + ((((((int)blockIdx.x) / 29) * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 464)) + (((int)threadIdx.x) * 2)));
    }
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
        conv2d_nchw_local[ff_c_inner] = (conv2d_nchw_local[ff_c_inner] + (pad_temp_shared[(((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58))] * kernel_shared[(((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw_local[(ff_c_inner + 2)] = (conv2d_nchw_local[(ff_c_inner + 2)] + (pad_temp_shared[((((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58)) + 58)] * kernel_shared[(((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw_local[(ff_c_inner + 4)] = (conv2d_nchw_local[(ff_c_inner + 4)] + (pad_temp_shared[(((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58))] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 512)]));
        conv2d_nchw_local[(ff_c_inner + 6)] = (conv2d_nchw_local[(ff_c_inner + 6)] + (pad_temp_shared[((((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 512)]));
        conv2d_nchw_local[(ff_c_inner + 8)] = (conv2d_nchw_local[(ff_c_inner + 8)] + (pad_temp_shared[(((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58))] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 1024)]));
        conv2d_nchw_local[(ff_c_inner + 10)] = (conv2d_nchw_local[(ff_c_inner + 10)] + (pad_temp_shared[((((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 1024)]));
        conv2d_nchw_local[(ff_c_inner + 12)] = (conv2d_nchw_local[(ff_c_inner + 12)] + (pad_temp_shared[(((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58))] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 1536)]));
        conv2d_nchw_local[(ff_c_inner + 14)] = (conv2d_nchw_local[(ff_c_inner + 14)] + (pad_temp_shared[((((rc_outer_inner * 1856) + (rc_inner * 116)) + (((int)threadIdx.x) % 58)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 58) * 128) + (ff_c_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 1536)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    conv2d_nchw[((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 58)] = conv2d_nchw_local[(ff_inner + 2)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 26912)] = conv2d_nchw_local[(ff_inner + 4)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 26970)] = conv2d_nchw_local[(ff_inner + 6)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 53824)] = conv2d_nchw_local[(ff_inner + 8)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 53882)] = conv2d_nchw_local[(ff_inner + 10)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 80736)] = conv2d_nchw_local[(ff_inner + 12)];
    conv2d_nchw[(((((((((int)blockIdx.x) / 29) * 107648) + ((((int)threadIdx.x) / 58) * 6728)) + (ff_inner * 3364)) + ((((int)blockIdx.x) % 29) * 116)) + (((int)threadIdx.x) % 58)) + 80794)] = conv2d_nchw_local[(ff_inner + 14)];
  }
}

extern "C" __global__ void __launch_bounds__(50) softmax_test(float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  float normal_reduce_temp0[1];
  __shared__ float red_buf0[50];
  __shared__ float T_softmax_maxelem[1];
  float normal_reduce_temp0_1[1];
  __shared__ float red_buf0_1[50];
  __shared__ float T_softmax_expsum[1];
  normal_reduce_temp0[0] = -3.402823e+38f;
  for (int k_outer = 0; k_outer < 20; ++k_outer) {
    normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + (k_outer * 50)) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[((int)threadIdx.x)] = normal_reduce_temp0[0];
  __syncthreads();
  if (((int)threadIdx.x) < 18) {
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 32)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    float w_16_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 16)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_16_0;
    float w_8_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 8)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_8_0;
    float w_4_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 4)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_4_0;
    float w_2_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 2)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_2_0;
    float w_1_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 1)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_1_0;
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_maxelem[0] = ((volatile float*)red_buf0)[0];
  }
  normal_reduce_temp0_1[0] = 0.000000e+00f;
  __syncthreads();
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[((((int)blockIdx.x) * 1000) + ((int)threadIdx.x))] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 50)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 100)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 150)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 200)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 250)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 300)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 350)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 400)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 450)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 500)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 550)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 600)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 650)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 700)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 750)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 800)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 850)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 900)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 950)] - T_softmax_maxelem[0])));
  __syncthreads();
  ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = normal_reduce_temp0_1[0];
  __syncthreads();
  if (((int)threadIdx.x) < 18) {
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 32)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    float w_16_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 16)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_16_0_1;
    float w_8_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 8)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_8_0_1;
    float w_4_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 4)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_4_0_1;
    float w_2_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 2)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_2_0_1;
    float w_1_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 1)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_1_0_1;
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_expsum[0] = ((volatile float*)red_buf0_1)[0];
  }
  __syncthreads();
  for (int i1_outer = 0; i1_outer < 20; ++i1_outer) {
    T_softmax_norm[(((((int)blockIdx.x) * 1000) + (i1_outer * 50)) + ((int)threadIdx.x))] = (__expf((data[(((((int)blockIdx.x) * 1000) + (i1_outer * 50)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])) / T_softmax_expsum[0]);
  }
}

