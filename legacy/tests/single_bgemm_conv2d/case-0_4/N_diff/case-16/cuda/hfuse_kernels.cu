
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(224) void bgemm_0_conv2d_4_fused_hfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_4_conv2d_nchw_, float *__restrict conv2d_4_data_, float *__restrict conv2d_4_kernel_)
{
  // bgemm_0
  if (((int)threadIdx.x >= 0 && (int)threadIdx.x < 64) && ((int)blockIdx.x >= 0 && (int)blockIdx.x < 65536))
  {
      int blockIdx_x_ = (int)blockIdx.x % 8;
      int blockIdx_y_ = (int)blockIdx.x / 8 % 8;
      int blockIdx_z_ = (int)blockIdx.x / 64;
      int threadIdx_x_ = ((int)threadIdx.x - 0) % 8;
      int threadIdx_y_ = ((int)threadIdx.x - 0) / 8 % 8;
      int threadIdx_z_ = ((int)threadIdx.x - 0) / 64;

      float T_batch_matmul_NT_local[64];
      static float A_shared[512] __attribute__((shared));
      static float B_shared[512] __attribute__((shared));
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          asm ("bar.sync 1, 64;");
          A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          asm ("bar.sync 1, 64;");
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
  }
  // conv2d_4
  if (((int)threadIdx.x >= 64 && (int)threadIdx.x < 216) && ((int)blockIdx.x >= 0 && (int)blockIdx.x < 350208))
  {
      int blockIdx_x_ = (int)blockIdx.x % 3;
      int blockIdx_y_ = (int)blockIdx.x / 3 % 57;
      int blockIdx_z_ = (int)blockIdx.x / 171;
      int threadIdx_x_ = ((int)threadIdx.x - 64) % 19;
      int threadIdx_y_ = ((int)threadIdx.x - 64) / 19 % 1;
      int threadIdx_z_ = ((int)threadIdx.x - 64) / 19;

      float conv2d_nchw_local[4];
      static float pad_temp_shared[450] __attribute__((shared));
      static float kernel_shared[576] __attribute__((shared));
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
          asm ("bar.sync 2, 160;");
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              if (((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) < 150) {
                  pad_temp_shared[(((((int)threadIdx_z_) * 57) + (((int)threadIdx_x_) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = conv2d_4_data_[((((((((((int)blockIdx_z_) >> 1) * 824464) + (rc_outer * 103058)) + ((((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) / 75) * 51529)) + (((int)blockIdx_y_) * 908)) + (((((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) % 75) / 25) * 227)) + (((int)blockIdx_x_) * 76)) + ((((((int)threadIdx_z_) * 57) + (((int)threadIdx_x_) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 75))];
              }
          }
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) {
              if (((((int)threadIdx_x_) / 18) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 18) {
                      kernel_shared[(((((int)threadIdx_z_) * 72) + (((int)threadIdx_x_) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1)] = conv2d_4_kernel_[((((((((int)blockIdx_z_) & 1) * 4608) + (((int)threadIdx_z_) * 576)) + ((((((int)threadIdx_x_) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 >> 1)) / 9) * 144)) + (rc_outer * 18)) + (((((int)threadIdx_x_) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) % 18))];
                  }
              }
          }
          asm ("bar.sync 2, 160;");
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
                  for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
                      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * kernel_shared[((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
                      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
                      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
                      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
                  }
              }
          }
      }
      conv2d_4_conv2d_nchw_[(((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 25992)] = conv2d_nchw_local[1];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 51984)] = conv2d_nchw_local[2];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 77976)] = conv2d_nchw_local[3];
  }
}
