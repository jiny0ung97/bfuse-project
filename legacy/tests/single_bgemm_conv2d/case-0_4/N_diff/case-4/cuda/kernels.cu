
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) bgemm_0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[64];
  __shared__ float A_shared[512];
  __shared__ float B_shared[512];
  float A_shared_local[8];
  float B_shared_local[8];
  for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
      T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    __syncthreads();
    A_shared[((((int)threadIdx.y) * 64) + ((int)threadIdx.x))] = A[(((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 8)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 64)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 16)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 128)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 24)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 192)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 32)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 256)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 40)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 320)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 48)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 384)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 56)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 448)];
    B_shared[((((int)threadIdx.y) * 64) + ((int)threadIdx.x))] = B[(((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 8)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 64)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 16)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 128)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 24)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 192)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 32)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 256)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 40)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 320)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 48)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 384)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 56)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 448)];
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      A_shared_local[0] = A_shared[((((int)threadIdx.y) * 64) + k_inner)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 8)];
      A_shared_local[2] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 16)];
      A_shared_local[3] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 24)];
      A_shared_local[4] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 32)];
      A_shared_local[5] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 40)];
      A_shared_local[6] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 48)];
      A_shared_local[7] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 56)];
      B_shared_local[0] = B_shared[((((int)threadIdx.x) * 64) + k_inner)];
      B_shared_local[1] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 8)];
      B_shared_local[2] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 16)];
      B_shared_local[3] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 24)];
      B_shared_local[4] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 32)];
      B_shared_local[5] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 40)];
      B_shared_local[6] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 48)];
      B_shared_local[7] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 56)];
      for (int i_c = 0; i_c < 8; ++i_c) {
        T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
        T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
        T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
        T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
        T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
        T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
        T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
        T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
      }
    }
  }
  for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
    T_batch_matmul_NT[((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
  }
}

extern "C" __global__ void __launch_bounds__(152) conv2d_4(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[450];
  __shared__ float kernel_shared[576];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 19) + ((int)threadIdx.x)) < 150) {
        pad_temp_shared[(((((int)threadIdx.z) * 57) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = data[((((((((((int)blockIdx.z) >> 1) * 824464) + (rc_outer * 103058)) + ((((((int)threadIdx.z) * 19) + ((int)threadIdx.x)) / 75) * 51529)) + (((int)blockIdx.y) * 908)) + (((((((int)threadIdx.z) * 19) + ((int)threadIdx.x)) % 75) / 25) * 227)) + (((int)blockIdx.x) * 76)) + ((((((int)threadIdx.z) * 57) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 75))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) {
      if (((((int)threadIdx.x) / 18) + ((int)threadIdx.z)) < 8) {
        if (((int)threadIdx.x) < 18) {
          kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1)] = kernel[((((((((int)blockIdx.z) & 1) * 4608) + (((int)threadIdx.z) * 576)) + ((((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 >> 1)) / 9) * 144)) + (rc_outer * 18)) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) % 18))];
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx.x) * 4)) + rx_inner)] * kernel_shared[((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx.x) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx.x) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx.x) * 4)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
        }
      }
    }
  }
  conv2d_nchw[(((((((int)blockIdx.z) * 103968) + (((int)threadIdx.z) * 3249)) + (((int)blockIdx.y) * 57)) + (((int)blockIdx.x) * 19)) + ((int)threadIdx.x))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((int)blockIdx.z) * 103968) + (((int)threadIdx.z) * 3249)) + (((int)blockIdx.y) * 57)) + (((int)blockIdx.x) * 19)) + ((int)threadIdx.x)) + 25992)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((int)blockIdx.z) * 103968) + (((int)threadIdx.z) * 3249)) + (((int)blockIdx.y) * 57)) + (((int)blockIdx.x) * 19)) + ((int)threadIdx.x)) + 51984)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((int)blockIdx.z) * 103968) + (((int)threadIdx.z) * 3249)) + (((int)blockIdx.y) * 57)) + (((int)blockIdx.x) * 19)) + ((int)threadIdx.x)) + 77976)] = conv2d_nchw_local[3];
}

