
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) bgemm(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[1];
  __shared__ float A_shared[8];
  __shared__ float B_shared[64];
  float A_shared_local[1];
  float B_shared_local[1];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((int)blockIdx.z) * 512) + (k_outer * 8)) + ((int)threadIdx.x))];
    // #pragma unroll
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      B_shared[((ax1_inner * 8) + ((int)threadIdx.x))] = B[(((((((int)blockIdx.z) * 512000) + (((int)blockIdx.x) * 4096)) + (ax1_inner * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      A_shared_local[0] = A_shared[k_inner];
      B_shared_local[0] = B_shared[((((int)threadIdx.x) * 8) + k_inner)];
      T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (A_shared_local[0] * B_shared_local[0]));
    }
  }
  T_batch_matmul_NT[(((((int)blockIdx.z) * 1000) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[0];
}

extern "C" __global__ void __launch_bounds__(58) conv2d(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nchw_local[64];
  __shared__ float pad_temp_shared[232];
  __shared__ float kernel_shared[256];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[56] = 0.000000e+00f;
  conv2d_nchw_local[58] = 0.000000e+00f;
  conv2d_nchw_local[60] = 0.000000e+00f;
  conv2d_nchw_local[62] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  conv2d_nchw_local[57] = 0.000000e+00f;
  conv2d_nchw_local[59] = 0.000000e+00f;
  conv2d_nchw_local[61] = 0.000000e+00f;
  conv2d_nchw_local[63] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((int)threadIdx.z) * 116) + (((int)threadIdx.x) * 4))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) % 58) / 29))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx.x) * 29) + ((((int)threadIdx.x) * 4) % 29)))) && (((((int)blockIdx.x) * 29) + ((((int)threadIdx.x) * 4) % 29)) < 57)) ? data[(((((((((((int)blockIdx.z) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + (((((int)threadIdx.x) * 2) / 29) * 3136)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 4) % 58) / 29) * 56)) + (((int)blockIdx.x) * 29)) + ((((int)threadIdx.x) * 4) % 29)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((((int)threadIdx.z) * 116) + (((int)threadIdx.x) * 4)) + 1)] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 58) / 29))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 1) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 1) % 29)))) && (((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 1) % 29)) < 57)) ? data[(((((((((((int)blockIdx.z) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + (((((int)threadIdx.x) * 2) / 29) * 3136)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 4) + 1) % 58) / 29) * 56)) + (((int)blockIdx.x) * 29)) + (((((int)threadIdx.x) * 4) + 1) % 29)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((((int)threadIdx.z) * 116) + (((int)threadIdx.x) * 4)) + 2)] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 58) / 29))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 2) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 2) % 29)))) && (((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 2) % 29)) < 57)) ? data[(((((((((((int)blockIdx.z) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 2) + 1) / 29) * 3136)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 4) + 2) % 58) / 29) * 56)) + (((int)blockIdx.x) * 29)) + (((((int)threadIdx.x) * 4) + 2) % 29)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((((int)threadIdx.z) * 116) + (((int)threadIdx.x) * 4)) + 3)] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 58) / 29))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + 3) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 3) % 29)))) && (((((int)blockIdx.x) * 29) + (((((int)threadIdx.x) * 4) + 3) % 29)) < 57)) ? data[(((((((((((int)blockIdx.z) * 200704) + (rc_outer * 12544)) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.x) * 2) + 1) / 29) * 3136)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 4) + 3) % 58) / 29) * 56)) + (((int)blockIdx.x) * 29)) + (((((int)threadIdx.x) * 4) + 3) % 29)) - 57)] : 0.000000e+00f);
    if ((((((int)threadIdx.x) * 5) >> 7) + ((int)threadIdx.z)) < 2) {
      if (((int)threadIdx.x) < 26) {
        kernel_shared[((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 5))] = kernel[((((((int)threadIdx.z) * 2048) + (((((int)threadIdx.x) * 5) >> 2) * 64)) + (rc_outer * 4)) + (((int)threadIdx.x) & 3))];
      }
    }
    if (((((((int)threadIdx.x) * 5) + 1) >> 7) + ((int)threadIdx.z)) < 2) {
      if (((int)threadIdx.x) < 26) {
        kernel_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 5)) + 1)] = kernel[((((((int)threadIdx.z) * 2048) + ((((((int)threadIdx.x) * 5) + 1) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx.x) + 1) & 3))];
      }
    }
    if (((((((int)threadIdx.x) * 5) + 2) >> 7) + ((int)threadIdx.z)) < 2) {
      if (((int)threadIdx.x) < 26) {
        kernel_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 5)) + 2)] = kernel[((((((int)threadIdx.z) * 2048) + ((((((int)threadIdx.x) * 5) + 2) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx.x) + 2) & 3))];
      }
    }
    if (((((((int)threadIdx.x) * 5) + 3) >> 7) + ((int)threadIdx.z)) < 2) {
      if (((int)threadIdx.x) < 25) {
        kernel_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 5)) + 3)] = kernel[((((((int)threadIdx.z) * 2048) + ((((((int)threadIdx.x) * 5) + 3) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx.x) + 3) & 3))];
      }
    }
    if (((((((int)threadIdx.x) * 5) + 4) >> 7) + ((int)threadIdx.z)) < 2) {
      if (((int)threadIdx.x) < 25) {
        kernel_shared[(((((int)threadIdx.z) * 128) + (((int)threadIdx.x) * 5)) + 4)] = kernel[(((((((int)threadIdx.z) * 2048) + (((((int)threadIdx.x) * 5) >> 2) * 64)) + (rc_outer * 4)) + (((int)threadIdx.x) & 3)) + 64)];
      }
    }
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 8)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 16)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 24)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 32)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 40)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 48)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 56)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 72)]));
    conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 80)]));
    conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 88)]));
    conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 96)]));
    conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 104)]));
    conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 112)]));
    conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 120)]));
    conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 128)]));
    conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 136)]));
    conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 144)]));
    conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 152)]));
    conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 160)]));
    conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 168)]));
    conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 176)]));
    conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 184)]));
    conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 192)]));
    conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 200)]));
    conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 208)]));
    conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 216)]));
    conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 224)]));
    conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 232)]));
    conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 240)]));
    conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[((((int)threadIdx.z) * 4) + 248)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 8)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 16)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 24)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 32)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 40)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 48)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 56)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 72)]));
    conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 80)]));
    conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 88)]));
    conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 96)]));
    conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 104)]));
    conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 112)]));
    conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 120)]));
    conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 128)]));
    conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 136)]));
    conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 144)]));
    conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 152)]));
    conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 160)]));
    conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 168)]));
    conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 176)]));
    conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 184)]));
    conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 192)]));
    conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 200)]));
    conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 208)]));
    conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 216)]));
    conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 224)]));
    conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 232)]));
    conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 240)]));
    conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (pad_temp_shared[(((int)threadIdx.x) + 29)] * kernel_shared[((((int)threadIdx.z) * 4) + 248)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 9)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 17)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 25)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 33)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 41)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 49)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 57)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 73)]));
    conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 81)]));
    conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 89)]));
    conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 97)]));
    conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 105)]));
    conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 113)]));
    conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 121)]));
    conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 129)]));
    conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 137)]));
    conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 145)]));
    conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 153)]));
    conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 161)]));
    conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 169)]));
    conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 177)]));
    conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 185)]));
    conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 193)]));
    conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 201)]));
    conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 209)]));
    conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 217)]));
    conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 225)]));
    conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 233)]));
    conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 241)]));
    conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (pad_temp_shared[(((int)threadIdx.x) + 58)] * kernel_shared[((((int)threadIdx.z) * 4) + 249)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 9)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 17)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 25)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 33)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 41)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 49)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 57)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 73)]));
    conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 81)]));
    conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 89)]));
    conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 97)]));
    conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 105)]));
    conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 113)]));
    conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 121)]));
    conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 129)]));
    conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 137)]));
    conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 145)]));
    conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 153)]));
    conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 161)]));
    conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 169)]));
    conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 177)]));
    conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 185)]));
    conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 193)]));
    conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 201)]));
    conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 209)]));
    conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 217)]));
    conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 225)]));
    conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 233)]));
    conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 241)]));
    conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (pad_temp_shared[(((int)threadIdx.x) + 87)] * kernel_shared[((((int)threadIdx.z) * 4) + 249)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 10)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 18)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 26)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 34)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 42)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 50)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 58)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 74)]));
    conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 82)]));
    conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 90)]));
    conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 98)]));
    conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 106)]));
    conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 114)]));
    conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 122)]));
    conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 130)]));
    conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 138)]));
    conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 146)]));
    conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 154)]));
    conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 162)]));
    conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 170)]));
    conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 178)]));
    conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 186)]));
    conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 194)]));
    conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 202)]));
    conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 210)]));
    conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 218)]));
    conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 226)]));
    conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 234)]));
    conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 242)]));
    conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (pad_temp_shared[(((int)threadIdx.x) + 116)] * kernel_shared[((((int)threadIdx.z) * 4) + 250)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 10)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 18)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 26)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 34)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 42)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 50)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 58)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 74)]));
    conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 82)]));
    conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 90)]));
    conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 98)]));
    conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 106)]));
    conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 114)]));
    conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 122)]));
    conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 130)]));
    conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 138)]));
    conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 146)]));
    conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 154)]));
    conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 162)]));
    conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 170)]));
    conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 178)]));
    conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 186)]));
    conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 194)]));
    conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 202)]));
    conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 210)]));
    conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 218)]));
    conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 226)]));
    conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 234)]));
    conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 242)]));
    conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (pad_temp_shared[(((int)threadIdx.x) + 145)] * kernel_shared[((((int)threadIdx.z) * 4) + 250)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 11)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 19)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 27)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 35)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 43)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 51)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 59)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 75)]));
    conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 83)]));
    conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 91)]));
    conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 99)]));
    conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 107)]));
    conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 115)]));
    conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 123)]));
    conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 131)]));
    conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 139)]));
    conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 147)]));
    conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 155)]));
    conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 163)]));
    conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 171)]));
    conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 179)]));
    conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 187)]));
    conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 195)]));
    conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 203)]));
    conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 211)]));
    conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 219)]));
    conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 227)]));
    conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 235)]));
    conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 243)]));
    conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (pad_temp_shared[(((int)threadIdx.x) + 174)] * kernel_shared[((((int)threadIdx.z) * 4) + 251)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 11)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 19)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 27)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 35)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 43)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 51)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 59)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 75)]));
    conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 83)]));
    conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 91)]));
    conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 99)]));
    conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 107)]));
    conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 115)]));
    conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 123)]));
    conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 131)]));
    conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 139)]));
    conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 147)]));
    conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 155)]));
    conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 163)]));
    conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 171)]));
    conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 179)]));
    conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 187)]));
    conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 195)]));
    conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 203)]));
    conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 211)]));
    conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 219)]));
    conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 227)]));
    conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 235)]));
    conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 243)]));
    conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (pad_temp_shared[(((int)threadIdx.x) + 203)] * kernel_shared[((((int)threadIdx.z) * 4) + 251)]));
  }
  conv2d_nchw[(((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 6728)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 13456)] = conv2d_nchw_local[4];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 20184)] = conv2d_nchw_local[6];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 26912)] = conv2d_nchw_local[8];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 33640)] = conv2d_nchw_local[10];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 40368)] = conv2d_nchw_local[12];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 47096)] = conv2d_nchw_local[14];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 53824)] = conv2d_nchw_local[16];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 60552)] = conv2d_nchw_local[18];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 67280)] = conv2d_nchw_local[20];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 74008)] = conv2d_nchw_local[22];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 80736)] = conv2d_nchw_local[24];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 87464)] = conv2d_nchw_local[26];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 94192)] = conv2d_nchw_local[28];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 100920)] = conv2d_nchw_local[30];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 107648)] = conv2d_nchw_local[32];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 114376)] = conv2d_nchw_local[34];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 121104)] = conv2d_nchw_local[36];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 127832)] = conv2d_nchw_local[38];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 134560)] = conv2d_nchw_local[40];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 141288)] = conv2d_nchw_local[42];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 148016)] = conv2d_nchw_local[44];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 154744)] = conv2d_nchw_local[46];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 161472)] = conv2d_nchw_local[48];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 168200)] = conv2d_nchw_local[50];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 174928)] = conv2d_nchw_local[52];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 181656)] = conv2d_nchw_local[54];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 188384)] = conv2d_nchw_local[56];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 195112)] = conv2d_nchw_local[58];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 201840)] = conv2d_nchw_local[60];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 208568)] = conv2d_nchw_local[62];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 58)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 6786)] = conv2d_nchw_local[3];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 13514)] = conv2d_nchw_local[5];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 20242)] = conv2d_nchw_local[7];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 26970)] = conv2d_nchw_local[9];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 33698)] = conv2d_nchw_local[11];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 40426)] = conv2d_nchw_local[13];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 47154)] = conv2d_nchw_local[15];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 53882)] = conv2d_nchw_local[17];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 60610)] = conv2d_nchw_local[19];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 67338)] = conv2d_nchw_local[21];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 74066)] = conv2d_nchw_local[23];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 80794)] = conv2d_nchw_local[25];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 87522)] = conv2d_nchw_local[27];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 94250)] = conv2d_nchw_local[29];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 100978)] = conv2d_nchw_local[31];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 107706)] = conv2d_nchw_local[33];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 114434)] = conv2d_nchw_local[35];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 121162)] = conv2d_nchw_local[37];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 127890)] = conv2d_nchw_local[39];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 134618)] = conv2d_nchw_local[41];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 141346)] = conv2d_nchw_local[43];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 148074)] = conv2d_nchw_local[45];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 154802)] = conv2d_nchw_local[47];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 161530)] = conv2d_nchw_local[49];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 168258)] = conv2d_nchw_local[51];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 174986)] = conv2d_nchw_local[53];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 181714)] = conv2d_nchw_local[55];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 188442)] = conv2d_nchw_local[57];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 195170)] = conv2d_nchw_local[59];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 201898)] = conv2d_nchw_local[61];
  conv2d_nchw[((((((((int)blockIdx.z) * 215296) + (((int)threadIdx.z) * 3364)) + (((int)blockIdx.y) * 116)) + (((int)blockIdx.x) * 29)) + ((int)threadIdx.x)) + 208626)] = conv2d_nchw_local[63];
}

