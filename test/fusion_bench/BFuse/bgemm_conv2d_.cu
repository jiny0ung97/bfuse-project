
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(58) void bgemm_conv2d_fused_kernel_bfuse_idx_0(float *__restrict bgemm_A_, float *__restrict bgemm_B_, float *__restrict bgemm_T_batch_matmul_NT_, float *__restrict conv2d_conv2d_nchw_, float *__restrict conv2d_data_, float *__restrict conv2d_kernel_)
{
  /*
   * KernelID_ means...
   * 0: bgemm
   * 1: conv2d
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 3696) && ((((int)blockIdx.x - 0) / 84) % 2 == 0))
  {
    NewBlockIdx_ = 0 + (((int)blockIdx.x - 0) / 168) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 1;
    gridDim_x_ = 2;
    gridDim_y_ = 29;
    gridDim_z_ = 32;
    blockDim_x_ = 29;
    blockDim_y_ = 1;
    blockDim_z_ = 2;
  }
  else if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 3696) && ((((int)blockIdx.x - 0) / 84) % 2 == 1))
  {
    NewBlockIdx_ = 0 + (((int)blockIdx.x - 0) / 168) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 125;
    gridDim_y_ = 1;
    gridDim_z_ = 32;
    blockDim_x_ = 8;
    blockDim_y_ = 1;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x >= 3696 && (int)blockIdx.x < 5796) && ((((int)blockIdx.x - 3696) / 84) % 1 == 0))
  {
    NewBlockIdx_ = 1848 + (((int)blockIdx.x - 3696) / 84) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 125;
    gridDim_y_ = 1;
    gridDim_z_ = 32;
    blockDim_x_ = 8;
    blockDim_y_ = 1;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 5796 && (int)blockIdx.x < 5804)
  {
    NewBlockIdx_ = (int)blockIdx.x - 3948;
    KernelID_  = 1;
    gridDim_x_ = 2;
    gridDim_y_ = 29;
    gridDim_z_ = 32;
    blockDim_x_ = 29;
    blockDim_y_ = 1;
    blockDim_z_ = 2;
  }
  else if ((int)blockIdx.x >= 5804 && (int)blockIdx.x < 5856)
  {
    NewBlockIdx_ = (int)blockIdx.x - 1856;
    KernelID_  = 0;
    gridDim_x_ = 125;
    gridDim_y_ = 1;
    gridDim_z_ = 32;
    blockDim_x_ = 8;
    blockDim_y_ = 1;
    blockDim_z_ = 1;
  }
  blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
  blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
  blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
  threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
  threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
  threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

  static float union_shared_0_[256] __attribute__((shared));
  static float union_shared_1_[232] __attribute__((shared));


  // bgemm
  if ((KernelID_ == 0) && ((int)threadIdx.x < 8))
  {
      float T_batch_matmul_NT_local[1];
      float A_shared_local[1];
      float B_shared_local[1];
      T_batch_matmul_NT_local[0] = 0.F;
      for (int k_outer = 0; k_outer < 64; ++k_outer) {
          __syncthreads();
          union_shared_1_[((int)threadIdx_x_)] = bgemm_A_[(((((int)blockIdx_z_) * 512) + (k_outer * 8)) + ((int)threadIdx_x_))];
          for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
              union_shared_0_[((ax1_inner * 8) + ((int)threadIdx_x_))] = bgemm_B_[(((((((int)blockIdx_z_) * 512000) + (((int)blockIdx_x_) * 4096)) + (ax1_inner * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          }
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = union_shared_1_[k_inner];
              B_shared_local[0] = union_shared_0_[((((int)threadIdx_x_) * 8) + k_inner)];
              T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (A_shared_local[0] * B_shared_local[0]));
          }
      }
      bgemm_T_batch_matmul_NT_[(((((int)blockIdx_z_) * 1000) + (((int)blockIdx_x_) * 8)) + ((int)threadIdx_x_))] = T_batch_matmul_NT_local[0];
  }
  // conv2d
  else if ((KernelID_ == 1) && ((int)threadIdx.x < 58))
  {
      float conv2d_nchw_local[64];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[16] = 0.F;
      conv2d_nchw_local[18] = 0.F;
      conv2d_nchw_local[20] = 0.F;
      conv2d_nchw_local[22] = 0.F;
      conv2d_nchw_local[24] = 0.F;
      conv2d_nchw_local[26] = 0.F;
      conv2d_nchw_local[28] = 0.F;
      conv2d_nchw_local[30] = 0.F;
      conv2d_nchw_local[32] = 0.F;
      conv2d_nchw_local[34] = 0.F;
      conv2d_nchw_local[36] = 0.F;
      conv2d_nchw_local[38] = 0.F;
      conv2d_nchw_local[40] = 0.F;
      conv2d_nchw_local[42] = 0.F;
      conv2d_nchw_local[44] = 0.F;
      conv2d_nchw_local[46] = 0.F;
      conv2d_nchw_local[48] = 0.F;
      conv2d_nchw_local[50] = 0.F;
      conv2d_nchw_local[52] = 0.F;
      conv2d_nchw_local[54] = 0.F;
      conv2d_nchw_local[56] = 0.F;
      conv2d_nchw_local[58] = 0.F;
      conv2d_nchw_local[60] = 0.F;
      conv2d_nchw_local[62] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      conv2d_nchw_local[17] = 0.F;
      conv2d_nchw_local[19] = 0.F;
      conv2d_nchw_local[21] = 0.F;
      conv2d_nchw_local[23] = 0.F;
      conv2d_nchw_local[25] = 0.F;
      conv2d_nchw_local[27] = 0.F;
      conv2d_nchw_local[29] = 0.F;
      conv2d_nchw_local[31] = 0.F;
      conv2d_nchw_local[33] = 0.F;
      conv2d_nchw_local[35] = 0.F;
      conv2d_nchw_local[37] = 0.F;
      conv2d_nchw_local[39] = 0.F;
      conv2d_nchw_local[41] = 0.F;
      conv2d_nchw_local[43] = 0.F;
      conv2d_nchw_local[45] = 0.F;
      conv2d_nchw_local[47] = 0.F;
      conv2d_nchw_local[49] = 0.F;
      conv2d_nchw_local[51] = 0.F;
      conv2d_nchw_local[53] = 0.F;
      conv2d_nchw_local[55] = 0.F;
      conv2d_nchw_local[57] = 0.F;
      conv2d_nchw_local[59] = 0.F;
      conv2d_nchw_local[61] = 0.F;
      conv2d_nchw_local[63] = 0.F;
      for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
          __syncthreads();
          union_shared_1_[((((int)threadIdx_z_) * 116) + (((int)threadIdx_x_) * 4))] = (((((1 <= ((((int)blockIdx_y_) * 2) + (((((int)threadIdx_x_) * 4) % 58) / 29))) && (((((int)blockIdx_y_) * 2) + (((((int)threadIdx_x_) * 4) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx_x_) * 29) + ((((int)threadIdx_x_) * 4) % 29)))) && (((((int)blockIdx_x_) * 29) + ((((int)threadIdx_x_) * 4) % 29)) < 57)) ? conv2d_data_[(((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + (((((int)threadIdx_x_) * 2) / 29) * 3136)) + (((int)blockIdx_y_) * 112)) + ((((((int)threadIdx_x_) * 4) % 58) / 29) * 56)) + (((int)blockIdx_x_) * 29)) + ((((int)threadIdx_x_) * 4) % 29)) - 57)] : 0.F);
          union_shared_1_[(((((int)threadIdx_z_) * 116) + (((int)threadIdx_x_) * 4)) + 1)] = (((((1 <= ((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 1) % 58) / 29))) && (((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 1) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 1) % 29)))) && (((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 1) % 29)) < 57)) ? conv2d_data_[(((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + (((((int)threadIdx_x_) * 2) / 29) * 3136)) + (((int)blockIdx_y_) * 112)) + (((((((int)threadIdx_x_) * 4) + 1) % 58) / 29) * 56)) + (((int)blockIdx_x_) * 29)) + (((((int)threadIdx_x_) * 4) + 1) % 29)) - 57)] : 0.F);
          union_shared_1_[(((((int)threadIdx_z_) * 116) + (((int)threadIdx_x_) * 4)) + 2)] = (((((1 <= ((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 2) % 58) / 29))) && (((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 2) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 2) % 29)))) && (((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 2) % 29)) < 57)) ? conv2d_data_[(((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((((int)threadIdx_x_) * 2) + 1) / 29) * 3136)) + (((int)blockIdx_y_) * 112)) + (((((((int)threadIdx_x_) * 4) + 2) % 58) / 29) * 56)) + (((int)blockIdx_x_) * 29)) + (((((int)threadIdx_x_) * 4) + 2) % 29)) - 57)] : 0.F);
          union_shared_1_[(((((int)threadIdx_z_) * 116) + (((int)threadIdx_x_) * 4)) + 3)] = (((((1 <= ((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 3) % 58) / 29))) && (((((int)blockIdx_y_) * 2) + ((((((int)threadIdx_x_) * 4) + 3) % 58) / 29)) < 57)) && (1 <= ((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 3) % 29)))) && (((((int)blockIdx_x_) * 29) + (((((int)threadIdx_x_) * 4) + 3) % 29)) < 57)) ? conv2d_data_[(((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((((int)threadIdx_x_) * 2) + 1) / 29) * 3136)) + (((int)blockIdx_y_) * 112)) + (((((((int)threadIdx_x_) * 4) + 3) % 58) / 29) * 56)) + (((int)blockIdx_x_) * 29)) + (((((int)threadIdx_x_) * 4) + 3) % 29)) - 57)] : 0.F);
          if ((((((int)threadIdx_x_) * 5) >> 7) + ((int)threadIdx_z_)) < 2) {
              if (((int)threadIdx_x_) < 26) {
                  union_shared_0_[((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 5))] = conv2d_kernel_[((((((int)threadIdx_z_) * 2048) + (((((int)threadIdx_x_) * 5) >> 2) * 64)) + (rc_outer * 4)) + (((int)threadIdx_x_) & 3))];
              }
          }
          if (((((((int)threadIdx_x_) * 5) + 1) >> 7) + ((int)threadIdx_z_)) < 2) {
              if (((int)threadIdx_x_) < 26) {
                  union_shared_0_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_kernel_[((((((int)threadIdx_z_) * 2048) + ((((((int)threadIdx_x_) * 5) + 1) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx_x_) + 1) & 3))];
              }
          }
          if (((((((int)threadIdx_x_) * 5) + 2) >> 7) + ((int)threadIdx_z_)) < 2) {
              if (((int)threadIdx_x_) < 26) {
                  union_shared_0_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_kernel_[((((((int)threadIdx_z_) * 2048) + ((((((int)threadIdx_x_) * 5) + 2) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx_x_) + 2) & 3))];
              }
          }
          if (((((((int)threadIdx_x_) * 5) + 3) >> 7) + ((int)threadIdx_z_)) < 2) {
              if (((int)threadIdx_x_) < 25) {
                  union_shared_0_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_kernel_[((((((int)threadIdx_z_) * 2048) + ((((((int)threadIdx_x_) * 5) + 3) >> 2) * 64)) + (rc_outer * 4)) + ((((int)threadIdx_x_) + 3) & 3))];
              }
          }
          if (((((((int)threadIdx_x_) * 5) + 4) >> 7) + ((int)threadIdx_z_)) < 2) {
              if (((int)threadIdx_x_) < 25) {
                  union_shared_0_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_kernel_[(((((((int)threadIdx_z_) * 2048) + (((((int)threadIdx_x_) * 5) >> 2) * 64)) + (rc_outer * 4)) + (((int)threadIdx_x_) & 3)) + 64)];
              }
          }
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 8)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 16)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 24)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 32)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 40)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 48)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 56)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 72)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 80)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 88)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 96)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 104)]));
          conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 112)]));
          conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 120)]));
          conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 128)]));
          conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 136)]));
          conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 144)]));
          conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 152)]));
          conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 160)]));
          conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 168)]));
          conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 176)]));
          conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 184)]));
          conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 192)]));
          conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 200)]));
          conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 208)]));
          conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 216)]));
          conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 224)]));
          conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 232)]));
          conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 240)]));
          conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_1_[((int)threadIdx_x_)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 248)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 8)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 16)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 24)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 32)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 40)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 48)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 56)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 72)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 80)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 88)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 96)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 104)]));
          conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 112)]));
          conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 120)]));
          conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 128)]));
          conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 136)]));
          conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 144)]));
          conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 152)]));
          conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 160)]));
          conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 168)]));
          conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 176)]));
          conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 184)]));
          conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 192)]));
          conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 200)]));
          conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 208)]));
          conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 216)]));
          conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 224)]));
          conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 232)]));
          conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 240)]));
          conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_1_[(((int)threadIdx_x_) + 29)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 248)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 9)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 17)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 25)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 33)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 41)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 49)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 57)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 73)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 81)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 89)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 97)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 105)]));
          conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 113)]));
          conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 121)]));
          conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 129)]));
          conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 137)]));
          conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 145)]));
          conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 153)]));
          conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 161)]));
          conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 169)]));
          conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 177)]));
          conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 185)]));
          conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 193)]));
          conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 201)]));
          conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 209)]));
          conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 217)]));
          conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 225)]));
          conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 233)]));
          conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 241)]));
          conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_1_[(((int)threadIdx_x_) + 58)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 249)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 9)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 17)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 25)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 33)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 41)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 49)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 57)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 73)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 81)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 89)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 97)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 105)]));
          conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 113)]));
          conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 121)]));
          conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 129)]));
          conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 137)]));
          conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 145)]));
          conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 153)]));
          conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 161)]));
          conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 169)]));
          conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 177)]));
          conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 185)]));
          conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 193)]));
          conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 201)]));
          conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 209)]));
          conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 217)]));
          conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 225)]));
          conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 233)]));
          conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 241)]));
          conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_1_[(((int)threadIdx_x_) + 87)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 249)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 10)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 18)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 26)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 34)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 42)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 50)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 58)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 74)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 82)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 90)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 98)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 106)]));
          conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 114)]));
          conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 122)]));
          conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 130)]));
          conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 138)]));
          conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 146)]));
          conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 154)]));
          conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 162)]));
          conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 170)]));
          conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 178)]));
          conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 186)]));
          conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 194)]));
          conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 202)]));
          conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 210)]));
          conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 218)]));
          conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 226)]));
          conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 234)]));
          conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 242)]));
          conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_1_[(((int)threadIdx_x_) + 116)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 250)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 10)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 18)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 26)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 34)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 42)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 50)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 58)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 74)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 82)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 90)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 98)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 106)]));
          conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 114)]));
          conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 122)]));
          conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 130)]));
          conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 138)]));
          conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 146)]));
          conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 154)]));
          conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 162)]));
          conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 170)]));
          conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 178)]));
          conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 186)]));
          conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 194)]));
          conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 202)]));
          conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 210)]));
          conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 218)]));
          conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 226)]));
          conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 234)]));
          conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 242)]));
          conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_1_[(((int)threadIdx_x_) + 145)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 250)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 11)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 19)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 27)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 35)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 43)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 51)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 59)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 75)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 83)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 91)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 99)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 107)]));
          conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 115)]));
          conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 123)]));
          conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 131)]));
          conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 139)]));
          conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 147)]));
          conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 155)]));
          conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 163)]));
          conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 171)]));
          conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 179)]));
          conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 187)]));
          conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 195)]));
          conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 203)]));
          conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 211)]));
          conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 219)]));
          conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 227)]));
          conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 235)]));
          conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 243)]));
          conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_1_[(((int)threadIdx_x_) + 174)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 251)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 11)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 19)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 27)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 35)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 43)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 51)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 59)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 75)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 83)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 91)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 99)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 107)]));
          conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 115)]));
          conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 123)]));
          conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 131)]));
          conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 139)]));
          conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 147)]));
          conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 155)]));
          conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 163)]));
          conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 171)]));
          conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 179)]));
          conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 187)]));
          conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 195)]));
          conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 203)]));
          conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 211)]));
          conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 219)]));
          conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 227)]));
          conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 235)]));
          conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 243)]));
          conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_1_[(((int)threadIdx_x_) + 203)] * union_shared_0_[((((int)threadIdx_z_) * 4) + 251)]));
      }
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 6728)] = conv2d_nchw_local[2];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 13456)] = conv2d_nchw_local[4];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 20184)] = conv2d_nchw_local[6];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 26912)] = conv2d_nchw_local[8];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 33640)] = conv2d_nchw_local[10];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 40368)] = conv2d_nchw_local[12];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 47096)] = conv2d_nchw_local[14];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 53824)] = conv2d_nchw_local[16];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 60552)] = conv2d_nchw_local[18];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 67280)] = conv2d_nchw_local[20];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 74008)] = conv2d_nchw_local[22];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 80736)] = conv2d_nchw_local[24];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 87464)] = conv2d_nchw_local[26];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 94192)] = conv2d_nchw_local[28];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 100920)] = conv2d_nchw_local[30];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 107648)] = conv2d_nchw_local[32];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 114376)] = conv2d_nchw_local[34];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 121104)] = conv2d_nchw_local[36];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 127832)] = conv2d_nchw_local[38];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 134560)] = conv2d_nchw_local[40];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 141288)] = conv2d_nchw_local[42];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 148016)] = conv2d_nchw_local[44];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 154744)] = conv2d_nchw_local[46];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 161472)] = conv2d_nchw_local[48];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 168200)] = conv2d_nchw_local[50];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 174928)] = conv2d_nchw_local[52];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 181656)] = conv2d_nchw_local[54];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 188384)] = conv2d_nchw_local[56];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 195112)] = conv2d_nchw_local[58];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 201840)] = conv2d_nchw_local[60];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 208568)] = conv2d_nchw_local[62];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 58)] = conv2d_nchw_local[1];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 6786)] = conv2d_nchw_local[3];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 13514)] = conv2d_nchw_local[5];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 20242)] = conv2d_nchw_local[7];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 26970)] = conv2d_nchw_local[9];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 33698)] = conv2d_nchw_local[11];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 40426)] = conv2d_nchw_local[13];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 47154)] = conv2d_nchw_local[15];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 53882)] = conv2d_nchw_local[17];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 60610)] = conv2d_nchw_local[19];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 67338)] = conv2d_nchw_local[21];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 74066)] = conv2d_nchw_local[23];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 80794)] = conv2d_nchw_local[25];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 87522)] = conv2d_nchw_local[27];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 94250)] = conv2d_nchw_local[29];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 100978)] = conv2d_nchw_local[31];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 107706)] = conv2d_nchw_local[33];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 114434)] = conv2d_nchw_local[35];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 121162)] = conv2d_nchw_local[37];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 127890)] = conv2d_nchw_local[39];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 134618)] = conv2d_nchw_local[41];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 141346)] = conv2d_nchw_local[43];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 148074)] = conv2d_nchw_local[45];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 154802)] = conv2d_nchw_local[47];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 161530)] = conv2d_nchw_local[49];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 168258)] = conv2d_nchw_local[51];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 174986)] = conv2d_nchw_local[53];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 181714)] = conv2d_nchw_local[55];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 188442)] = conv2d_nchw_local[57];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 195170)] = conv2d_nchw_local[59];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 201898)] = conv2d_nchw_local[61];
      conv2d_conv2d_nchw_[((((((((int)blockIdx_z_) * 215296) + (((int)threadIdx_z_) * 3364)) + (((int)blockIdx_y_) * 116)) + (((int)blockIdx_x_) * 29)) + ((int)threadIdx_x_)) + 208626)] = conv2d_nchw_local[63];
  }
}
