
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ conv2d_nhwc, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nhwc_local[64];
  __shared__ float pad_temp_shared[1024];
  __shared__ float kernel_shared[3072];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[32] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[33] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[34] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[35] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[36] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[37] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[38] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[39] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[40] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[41] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[42] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[43] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[44] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[45] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[46] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[47] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[48] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[49] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[50] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[51] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[52] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[53] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[54] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[55] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[56] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[57] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[58] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;
  conv2d_nhwc_local[59] = 0.000000e+00f;
  conv2d_nhwc_local[28] = 0.000000e+00f;
  conv2d_nhwc_local[60] = 0.000000e+00f;
  conv2d_nhwc_local[29] = 0.000000e+00f;
  conv2d_nhwc_local[61] = 0.000000e+00f;
  conv2d_nhwc_local[30] = 0.000000e+00f;
  conv2d_nhwc_local[62] = 0.000000e+00f;
  conv2d_nhwc_local[31] = 0.000000e+00f;
  conv2d_nhwc_local[63] = 0.000000e+00f;
  for (int ry_outer_outer = 0; ry_outer_outer < 3; ++ry_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
      __syncthreads();
      *(float4*)(pad_temp_shared + (((int)threadIdx.x) * 4)) = (((((1 <= (((((int)blockIdx.x) % 98) / 7) + ry_outer_outer)) && ((((((int)blockIdx.x) % 98) / 7) + ry_outer_outer) < 15)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)) < 15)) ? *(float4*)(data + (((((((((((int)blockIdx.x) / 98) * 401408) + ((((int)threadIdx.x) >> 3) * 12544)) + (ry_outer_outer * 896)) + ((((int)blockIdx.x) % 98) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 64)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) - 960)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
      *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 512)) = (((((1 <= (((((int)blockIdx.x) % 98) / 7) + ry_outer_outer)) && ((((((int)blockIdx.x) % 98) / 7) + ry_outer_outer) < 15)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)) < 15)) ? *(float4*)(data + (((((((((((int)blockIdx.x) / 98) * 401408) + ((((int)threadIdx.x) >> 3) * 12544)) + (ry_outer_outer * 896)) + ((((int)blockIdx.x) % 98) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 64)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 199744)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
      *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + (((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)));
      *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(kernel + ((((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)) + 512));
      *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(kernel + ((((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)) + 8192));
      *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(kernel + ((((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)) + 8704));
      *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(kernel + ((((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)) + 16384));
      *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(kernel + ((((ry_outer_outer * 24576) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) * 4)) + 16896));
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (pad_temp_shared[(((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[32] = (conv2d_nhwc_local[32] + (pad_temp_shared[(((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (pad_temp_shared[(((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[33] = (conv2d_nhwc_local[33] + (pad_temp_shared[(((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[34] = (conv2d_nhwc_local[34] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[35] = (conv2d_nhwc_local[35] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 32)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[36] = (conv2d_nhwc_local[36] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 32)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 32)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[37] = (conv2d_nhwc_local[37] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 32)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[38] = (conv2d_nhwc_local[38] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[39] = (conv2d_nhwc_local[39] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 64)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[40] = (conv2d_nhwc_local[40] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 64)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 64)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[41] = (conv2d_nhwc_local[41] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 64)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[42] = (conv2d_nhwc_local[42] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[43] = (conv2d_nhwc_local[43] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 96)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[44] = (conv2d_nhwc_local[44] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 96)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 96)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[45] = (conv2d_nhwc_local[45] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 96)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[46] = (conv2d_nhwc_local[46] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[47] = (conv2d_nhwc_local[47] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 128)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[48] = (conv2d_nhwc_local[48] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 128)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 128)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[49] = (conv2d_nhwc_local[49] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 128)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[50] = (conv2d_nhwc_local[50] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[51] = (conv2d_nhwc_local[51] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 160)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[52] = (conv2d_nhwc_local[52] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 160)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 160)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[53] = (conv2d_nhwc_local[53] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 160)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[54] = (conv2d_nhwc_local[54] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[55] = (conv2d_nhwc_local[55] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 192)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[56] = (conv2d_nhwc_local[56] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 192)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 192)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[57] = (conv2d_nhwc_local[57] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 192)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[58] = (conv2d_nhwc_local[58] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[59] = (conv2d_nhwc_local[59] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 224)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[60] = (conv2d_nhwc_local[60] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 224)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 224)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[61] = (conv2d_nhwc_local[61] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 224)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2))]));
        conv2d_nhwc_local[62] = (conv2d_nhwc_local[62] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
        conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
        conv2d_nhwc_local[63] = (conv2d_nhwc_local[63] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[32] = (conv2d_nhwc_local[32] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[33] = (conv2d_nhwc_local[33] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 8)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[34] = (conv2d_nhwc_local[34] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[35] = (conv2d_nhwc_local[35] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[36] = (conv2d_nhwc_local[36] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[37] = (conv2d_nhwc_local[37] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 40)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[38] = (conv2d_nhwc_local[38] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[39] = (conv2d_nhwc_local[39] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[40] = (conv2d_nhwc_local[40] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[41] = (conv2d_nhwc_local[41] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 72)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[42] = (conv2d_nhwc_local[42] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[43] = (conv2d_nhwc_local[43] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[44] = (conv2d_nhwc_local[44] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[45] = (conv2d_nhwc_local[45] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 104)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[46] = (conv2d_nhwc_local[46] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[47] = (conv2d_nhwc_local[47] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[48] = (conv2d_nhwc_local[48] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[49] = (conv2d_nhwc_local[49] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 136)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[50] = (conv2d_nhwc_local[50] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[51] = (conv2d_nhwc_local[51] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[52] = (conv2d_nhwc_local[52] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[53] = (conv2d_nhwc_local[53] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 168)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[54] = (conv2d_nhwc_local[54] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[55] = (conv2d_nhwc_local[55] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[56] = (conv2d_nhwc_local[56] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[57] = (conv2d_nhwc_local[57] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 200)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[58] = (conv2d_nhwc_local[58] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[59] = (conv2d_nhwc_local[59] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[60] = (conv2d_nhwc_local[60] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[61] = (conv2d_nhwc_local[61] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 232)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
        conv2d_nhwc_local[62] = (conv2d_nhwc_local[62] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
        conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
        conv2d_nhwc_local[63] = (conv2d_nhwc_local[63] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[32] = (conv2d_nhwc_local[32] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[33] = (conv2d_nhwc_local[33] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 16)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 24)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[34] = (conv2d_nhwc_local[34] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 24)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 24)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[35] = (conv2d_nhwc_local[35] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 24)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[36] = (conv2d_nhwc_local[36] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[37] = (conv2d_nhwc_local[37] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 48)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 56)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[38] = (conv2d_nhwc_local[38] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 56)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 56)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[39] = (conv2d_nhwc_local[39] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 56)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[40] = (conv2d_nhwc_local[40] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[41] = (conv2d_nhwc_local[41] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 80)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 88)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[42] = (conv2d_nhwc_local[42] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 88)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 88)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[43] = (conv2d_nhwc_local[43] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 88)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[44] = (conv2d_nhwc_local[44] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[45] = (conv2d_nhwc_local[45] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 112)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 120)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[46] = (conv2d_nhwc_local[46] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 120)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 120)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[47] = (conv2d_nhwc_local[47] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 120)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[48] = (conv2d_nhwc_local[48] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[49] = (conv2d_nhwc_local[49] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 144)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 152)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[50] = (conv2d_nhwc_local[50] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 152)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 152)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[51] = (conv2d_nhwc_local[51] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 152)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[52] = (conv2d_nhwc_local[52] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[53] = (conv2d_nhwc_local[53] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 176)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 184)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[54] = (conv2d_nhwc_local[54] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 184)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 184)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[55] = (conv2d_nhwc_local[55] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 184)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[56] = (conv2d_nhwc_local[56] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[57] = (conv2d_nhwc_local[57] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 208)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 216)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[58] = (conv2d_nhwc_local[58] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 216)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 216)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[59] = (conv2d_nhwc_local[59] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 216)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[28] = (conv2d_nhwc_local[28] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[60] = (conv2d_nhwc_local[60] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[29] = (conv2d_nhwc_local[29] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[61] = (conv2d_nhwc_local[61] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 240)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
        conv2d_nhwc_local[30] = (conv2d_nhwc_local[30] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 248)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2048)]));
        conv2d_nhwc_local[62] = (conv2d_nhwc_local[62] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 248)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2112)]));
        conv2d_nhwc_local[31] = (conv2d_nhwc_local[31] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 248)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2049)]));
        conv2d_nhwc_local[63] = (conv2d_nhwc_local[63] + (pad_temp_shared[((((((int)threadIdx.x) >> 5) * 256) + rc_outer_inner) + 248)] * kernel_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 2113)]));
      }
    }
  }
  for (int nn_inner = 0; nn_inner < 8; ++nn_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
        conv2d_nhwc[((((((((((int)blockIdx.x) / 98) * 802816) + ((((int)threadIdx.x) >> 5) * 200704)) + (nn_inner * 25088)) + ((((int)blockIdx.x) % 98) * 256)) + (xx_inner * 128)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner)] = conv2d_nhwc_local[(((nn_inner * 4) + (xx_inner * 2)) + ff_inner)];
        conv2d_nhwc[(((((((((((int)blockIdx.x) / 98) * 802816) + ((((int)threadIdx.x) >> 5) * 200704)) + (nn_inner * 25088)) + ((((int)blockIdx.x) % 98) * 256)) + (xx_inner * 128)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner) + 64)] = conv2d_nhwc_local[((((nn_inner * 4) + (xx_inner * 2)) + ff_inner) + 32)];
      }
    }
  }
}

