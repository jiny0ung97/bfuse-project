
#include <hip/hip_runtime.h>
 __global__ __launch_bounds__(128, 8) void softmax_shared_1036_softmax_shared_1036_copy_fused_kernel_vfuse_lb_idx_0(float *__restrict T_softmax_norm0, float *__restrict data1, float *__restrict T_softmax_norm22, float *__restrict data23)
 {
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_0 = 128;
    unsigned int threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_0 = 1;
    unsigned int threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_0 = 1;
    unsigned int threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp02[1];
    static float red_result3[257] __attribute__((shared));
    static float T_softmax_maxelem4[257] __attribute__((shared));
    float normal_reduce_temp0_15[1];
    static float red_result_16[257] __attribute__((shared));
    static float T_softmax_expsum7[257] __attribute__((shared));
    normal_reduce_temp02[0] = -3.40282306E+38F;
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0))]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 128)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 256)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 384)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 512)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 640)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 768)]);
    if (((int)threadIdx_x_0) < 104) {
        normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 896)]);
    }
    float red_buf08[1];
    unsigned int mask9[1];
    float t010[1];
    float red_buf0_111[1];
    unsigned int mask_112[1];
    float t0_113[1];
    static float red_buf_staging14[4] __attribute__((shared));
    red_buf0_111[0] = normal_reduce_temp02[0];
    mask_112[0] = __activemask();
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 16, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 8, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 4, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 2, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 1, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging14[(((int)threadIdx_x_0) >> 5)] = red_buf0_111[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf08[0] = red_buf_staging14[((int)threadIdx_x_0)];
    }
    mask9[0] = (__activemask() & (unsigned int)15);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 2, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 1, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result3)[0] = red_buf08[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_maxelem4[0] = ((volatile float *)red_result3)[0];
    }
    normal_reduce_temp0_15[0] = 0.F;
    asm ("bar.sync 1,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            normal_reduce_temp0_15[0] = (normal_reduce_temp0_15[0] + __expf((data1[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])));
        }
    }
    float red_buf0_215[1];
    unsigned int mask_216[1];
    float t0_217[1];
    float red_buf0_318[1];
    unsigned int mask_319[1];
    float t0_320[1];
    static float red_buf_staging_121[4] __attribute__((shared));
    red_buf0_318[0] = normal_reduce_temp0_15[0];
    mask_319[0] = __activemask();
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 16, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 8, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 4, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 2, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 1, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging_121[(((int)threadIdx_x_0) >> 5)] = red_buf0_318[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf0_215[0] = red_buf_staging_121[((int)threadIdx_x_0)];
    }
    mask_216[0] = (__activemask() & (unsigned int)15);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 2, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 1, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result_16)[0] = red_buf0_215[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_expsum7[0] = ((volatile float *)red_result_16)[0];
    }
    asm ("bar.sync 1,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            T_softmax_norm0[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] = (__expf((data1[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])) / T_softmax_expsum7[0]);
        }
    }
}
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_1 = 128;
    unsigned int threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_1 = 1;
    unsigned int threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_1 = 1;
    unsigned int threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp024[1];
    static float red_result25[257] __attribute__((shared));
    static float T_softmax_maxelem26[257] __attribute__((shared));
    float normal_reduce_temp0_127[1];
    static float red_result_128[257] __attribute__((shared));
    static float T_softmax_expsum29[257] __attribute__((shared));
    normal_reduce_temp024[0] = -3.40282306E+38F;
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1))]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 128)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 256)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 384)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 512)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 640)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 768)]);
    if (((int)threadIdx_x_1) < 104) {
        normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 896)]);
    }
    float red_buf030[1];
    unsigned int mask31[1];
    float t032[1];
    float red_buf0_133[1];
    unsigned int mask_134[1];
    float t0_135[1];
    static float red_buf_staging36[4] __attribute__((shared));
    red_buf0_133[0] = normal_reduce_temp024[0];
    mask_134[0] = __activemask();
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 16, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 8, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 4, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 2, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 1, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging36[(((int)threadIdx_x_1) >> 5)] = red_buf0_133[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf030[0] = red_buf_staging36[((int)threadIdx_x_1)];
    }
    mask31[0] = (__activemask() & (unsigned int)15);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 2, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 1, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result25)[0] = red_buf030[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_maxelem26[0] = ((volatile float *)red_result25)[0];
    }
    normal_reduce_temp0_127[0] = 0.F;
    asm ("bar.sync 2,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            normal_reduce_temp0_127[0] = (normal_reduce_temp0_127[0] + __expf((data23[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])));
        }
    }
    float red_buf0_237[1];
    unsigned int mask_238[1];
    float t0_239[1];
    float red_buf0_340[1];
    unsigned int mask_341[1];
    float t0_342[1];
    static float red_buf_staging_143[4] __attribute__((shared));
    red_buf0_340[0] = normal_reduce_temp0_127[0];
    mask_341[0] = __activemask();
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 16, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 8, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 4, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 2, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 1, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging_143[(((int)threadIdx_x_1) >> 5)] = red_buf0_340[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf0_237[0] = red_buf_staging_143[((int)threadIdx_x_1)];
    }
    mask_238[0] = (__activemask() & (unsigned int)15);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 2, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 1, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result_128)[0] = red_buf0_237[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_expsum29[0] = ((volatile float *)red_result_128)[0];
    }
    asm ("bar.sync 2,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            T_softmax_norm22[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] = (__expf((data23[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])) / T_softmax_expsum29[0]);
        }
    }
}
}
 __global__ __launch_bounds__(128, 0) void softmax_shared_1036_softmax_shared_1036_copy_fused_kernel_vfuse_idx_0(float *__restrict T_softmax_norm0, float *__restrict data1, float *__restrict T_softmax_norm22, float *__restrict data23)
 {
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_0 = 128;
    unsigned int threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_0 = 1;
    unsigned int threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_0 = 1;
    unsigned int threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp02[1];
    static float red_result3[257] __attribute__((shared));
    static float T_softmax_maxelem4[257] __attribute__((shared));
    float normal_reduce_temp0_15[1];
    static float red_result_16[257] __attribute__((shared));
    static float T_softmax_expsum7[257] __attribute__((shared));
    normal_reduce_temp02[0] = -3.40282306E+38F;
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0))]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 128)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 256)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 384)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 512)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 640)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 768)]);
    if (((int)threadIdx_x_0) < 104) {
        normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 896)]);
    }
    float red_buf08[1];
    unsigned int mask9[1];
    float t010[1];
    float red_buf0_111[1];
    unsigned int mask_112[1];
    float t0_113[1];
    static float red_buf_staging14[4] __attribute__((shared));
    red_buf0_111[0] = normal_reduce_temp02[0];
    mask_112[0] = __activemask();
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 16, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 8, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 4, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 2, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 1, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging14[(((int)threadIdx_x_0) >> 5)] = red_buf0_111[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf08[0] = red_buf_staging14[((int)threadIdx_x_0)];
    }
    mask9[0] = (__activemask() & (unsigned int)15);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 2, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 1, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result3)[0] = red_buf08[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_maxelem4[0] = ((volatile float *)red_result3)[0];
    }
    normal_reduce_temp0_15[0] = 0.F;
    asm ("bar.sync 1,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            normal_reduce_temp0_15[0] = (normal_reduce_temp0_15[0] + __expf((data1[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])));
        }
    }
    float red_buf0_215[1];
    unsigned int mask_216[1];
    float t0_217[1];
    float red_buf0_318[1];
    unsigned int mask_319[1];
    float t0_320[1];
    static float red_buf_staging_121[4] __attribute__((shared));
    red_buf0_318[0] = normal_reduce_temp0_15[0];
    mask_319[0] = __activemask();
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 16, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 8, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 4, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 2, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 1, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging_121[(((int)threadIdx_x_0) >> 5)] = red_buf0_318[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf0_215[0] = red_buf_staging_121[((int)threadIdx_x_0)];
    }
    mask_216[0] = (__activemask() & (unsigned int)15);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 2, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 1, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result_16)[0] = red_buf0_215[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_expsum7[0] = ((volatile float *)red_result_16)[0];
    }
    asm ("bar.sync 1,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            T_softmax_norm0[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] = (__expf((data1[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])) / T_softmax_expsum7[0]);
        }
    }
}
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_1 = 128;
    unsigned int threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_1 = 1;
    unsigned int threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_1 = 1;
    unsigned int threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp024[1];
    static float red_result25[257] __attribute__((shared));
    static float T_softmax_maxelem26[257] __attribute__((shared));
    float normal_reduce_temp0_127[1];
    static float red_result_128[257] __attribute__((shared));
    static float T_softmax_expsum29[257] __attribute__((shared));
    normal_reduce_temp024[0] = -3.40282306E+38F;
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1))]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 128)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 256)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 384)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 512)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 640)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 768)]);
    if (((int)threadIdx_x_1) < 104) {
        normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 896)]);
    }
    float red_buf030[1];
    unsigned int mask31[1];
    float t032[1];
    float red_buf0_133[1];
    unsigned int mask_134[1];
    float t0_135[1];
    static float red_buf_staging36[4] __attribute__((shared));
    red_buf0_133[0] = normal_reduce_temp024[0];
    mask_134[0] = __activemask();
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 16, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 8, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 4, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 2, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 1, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging36[(((int)threadIdx_x_1) >> 5)] = red_buf0_133[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf030[0] = red_buf_staging36[((int)threadIdx_x_1)];
    }
    mask31[0] = (__activemask() & (unsigned int)15);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 2, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 1, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result25)[0] = red_buf030[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_maxelem26[0] = ((volatile float *)red_result25)[0];
    }
    normal_reduce_temp0_127[0] = 0.F;
    asm ("bar.sync 2,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            normal_reduce_temp0_127[0] = (normal_reduce_temp0_127[0] + __expf((data23[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])));
        }
    }
    float red_buf0_237[1];
    unsigned int mask_238[1];
    float t0_239[1];
    float red_buf0_340[1];
    unsigned int mask_341[1];
    float t0_342[1];
    static float red_buf_staging_143[4] __attribute__((shared));
    red_buf0_340[0] = normal_reduce_temp0_127[0];
    mask_341[0] = __activemask();
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 16, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 8, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 4, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 2, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 1, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging_143[(((int)threadIdx_x_1) >> 5)] = red_buf0_340[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf0_237[0] = red_buf_staging_143[((int)threadIdx_x_1)];
    }
    mask_238[0] = (__activemask() & (unsigned int)15);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 2, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 1, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result_128)[0] = red_buf0_237[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_expsum29[0] = ((volatile float *)red_result_128)[0];
    }
    asm ("bar.sync 2,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            T_softmax_norm22[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] = (__expf((data23[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])) / T_softmax_expsum29[0]);
        }
    }
}
}
 __global__ __launch_bounds__(256, 0) void softmax_shared_1036_softmax_shared_1036_copy_fused_kernel_hfuse_idx_0(float *__restrict T_softmax_norm0, float *__restrict data1, float *__restrict T_softmax_norm22, float *__restrict data23)
 {
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_0 = 128;
    unsigned int threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_0 = 1;
    unsigned int threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_0 = 1;
    unsigned int threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp02[1];
    static float red_result3[257] __attribute__((shared));
    static float T_softmax_maxelem4[257] __attribute__((shared));
    float normal_reduce_temp0_15[1];
    static float red_result_16[257] __attribute__((shared));
    static float T_softmax_expsum7[257] __attribute__((shared));
    normal_reduce_temp02[0] = -3.40282306E+38F;
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0))]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 128)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 256)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 384)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 512)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 640)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 768)]);
    if (((int)threadIdx_x_0) < 104) {
        normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 896)]);
    }
    float red_buf08[1];
    unsigned int mask9[1];
    float t010[1];
    float red_buf0_111[1];
    unsigned int mask_112[1];
    float t0_113[1];
    static float red_buf_staging14[4] __attribute__((shared));
    red_buf0_111[0] = normal_reduce_temp02[0];
    mask_112[0] = __activemask();
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 16, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 8, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 4, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 2, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 1, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging14[(((int)threadIdx_x_0) >> 5)] = red_buf0_111[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf08[0] = red_buf_staging14[((int)threadIdx_x_0)];
    }
    mask9[0] = (__activemask() & (unsigned int)15);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 2, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 1, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result3)[0] = red_buf08[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_maxelem4[0] = ((volatile float *)red_result3)[0];
    }
    normal_reduce_temp0_15[0] = 0.F;
    asm ("bar.sync 1,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            normal_reduce_temp0_15[0] = (normal_reduce_temp0_15[0] + __expf((data1[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])));
        }
    }
    float red_buf0_215[1];
    unsigned int mask_216[1];
    float t0_217[1];
    float red_buf0_318[1];
    unsigned int mask_319[1];
    float t0_320[1];
    static float red_buf_staging_121[4] __attribute__((shared));
    red_buf0_318[0] = normal_reduce_temp0_15[0];
    mask_319[0] = __activemask();
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 16, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 8, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 4, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 2, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 1, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging_121[(((int)threadIdx_x_0) >> 5)] = red_buf0_318[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf0_215[0] = red_buf_staging_121[((int)threadIdx_x_0)];
    }
    mask_216[0] = (__activemask() & (unsigned int)15);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 2, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 1, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result_16)[0] = red_buf0_215[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_expsum7[0] = ((volatile float *)red_result_16)[0];
    }
    asm ("bar.sync 1,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            T_softmax_norm0[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] = (__expf((data1[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])) / T_softmax_expsum7[0]);
        }
    }
}
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=128 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 256)){
    unsigned int blockDim_x_1 = 128;
    unsigned int threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) % 128;
    unsigned int blockDim_y_1 = 1;
    unsigned int threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) / 128 % 1;
    unsigned int blockDim_z_1 = 1;
    unsigned int threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) / 128;
    float normal_reduce_temp024[1];
    static float red_result25[257] __attribute__((shared));
    static float T_softmax_maxelem26[257] __attribute__((shared));
    float normal_reduce_temp0_127[1];
    static float red_result_128[257] __attribute__((shared));
    static float T_softmax_expsum29[257] __attribute__((shared));
    normal_reduce_temp024[0] = -3.40282306E+38F;
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1))]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 128)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 256)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 384)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 512)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 640)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 768)]);
    if (((int)threadIdx_x_1) < 104) {
        normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 896)]);
    }
    float red_buf030[1];
    unsigned int mask31[1];
    float t032[1];
    float red_buf0_133[1];
    unsigned int mask_134[1];
    float t0_135[1];
    static float red_buf_staging36[4] __attribute__((shared));
    red_buf0_133[0] = normal_reduce_temp024[0];
    mask_134[0] = __activemask();
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 16, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 8, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 4, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 2, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 1, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging36[(((int)threadIdx_x_1) >> 5)] = red_buf0_133[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf030[0] = red_buf_staging36[((int)threadIdx_x_1)];
    }
    mask31[0] = (__activemask() & (unsigned int)15);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 2, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 1, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result25)[0] = red_buf030[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_maxelem26[0] = ((volatile float *)red_result25)[0];
    }
    normal_reduce_temp0_127[0] = 0.F;
    asm ("bar.sync 2,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            normal_reduce_temp0_127[0] = (normal_reduce_temp0_127[0] + __expf((data23[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])));
        }
    }
    float red_buf0_237[1];
    unsigned int mask_238[1];
    float t0_239[1];
    float red_buf0_340[1];
    unsigned int mask_341[1];
    float t0_342[1];
    static float red_buf_staging_143[4] __attribute__((shared));
    red_buf0_340[0] = normal_reduce_temp0_127[0];
    mask_341[0] = __activemask();
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 16, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 8, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 4, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 2, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 1, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging_143[(((int)threadIdx_x_1) >> 5)] = red_buf0_340[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf0_237[0] = red_buf_staging_143[((int)threadIdx_x_1)];
    }
    mask_238[0] = (__activemask() & (unsigned int)15);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 2, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 1, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result_128)[0] = red_buf0_237[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_expsum29[0] = ((volatile float *)red_result_128)[0];
    }
    asm ("bar.sync 2,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            T_softmax_norm22[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] = (__expf((data23[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])) / T_softmax_expsum29[0]);
        }
    }
}
}
 __global__ __launch_bounds__(256, 8) void softmax_shared_1036_softmax_shared_1036_copy_fused_kernel_hfuse_lb_idx_0(float *__restrict T_softmax_norm0, float *__restrict data1, float *__restrict T_softmax_norm22, float *__restrict data23)
 {
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 128)){
    unsigned int blockDim_x_0 = 128;
    unsigned int threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 128;
    unsigned int blockDim_y_0 = 1;
    unsigned int threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128 % 1;
    unsigned int blockDim_z_0 = 1;
    unsigned int threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 128;
    float normal_reduce_temp02[1];
    static float red_result3[257] __attribute__((shared));
    static float T_softmax_maxelem4[257] __attribute__((shared));
    float normal_reduce_temp0_15[1];
    static float red_result_16[257] __attribute__((shared));
    static float T_softmax_expsum7[257] __attribute__((shared));
    normal_reduce_temp02[0] = -3.40282306E+38F;
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0))]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 128)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 256)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 384)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 512)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 640)]);
    normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 768)]);
    if (((int)threadIdx_x_0) < 104) {
        normal_reduce_temp02[0] = max(normal_reduce_temp02[0], data1[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_0)) + 896)]);
    }
    float red_buf08[1];
    unsigned int mask9[1];
    float t010[1];
    float red_buf0_111[1];
    unsigned int mask_112[1];
    float t0_113[1];
    static float red_buf_staging14[4] __attribute__((shared));
    red_buf0_111[0] = normal_reduce_temp02[0];
    mask_112[0] = __activemask();
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 16, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 8, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 4, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 2, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    t0_113[0] = __shfl_down_sync(mask_112[0], red_buf0_111[0], 1, 32);
    red_buf0_111[0] = max(red_buf0_111[0], t0_113[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging14[(((int)threadIdx_x_0) >> 5)] = red_buf0_111[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf08[0] = red_buf_staging14[((int)threadIdx_x_0)];
    }
    mask9[0] = (__activemask() & (unsigned int)15);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 2, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    t010[0] = __shfl_down_sync(mask9[0], red_buf08[0], 1, 32);
    red_buf08[0] = max(red_buf08[0], t010[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result3)[0] = red_buf08[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_maxelem4[0] = ((volatile float *)red_result3)[0];
    }
    normal_reduce_temp0_15[0] = 0.F;
    asm ("bar.sync 1,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            normal_reduce_temp0_15[0] = (normal_reduce_temp0_15[0] + __expf((data1[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])));
        }
    }
    float red_buf0_215[1];
    unsigned int mask_216[1];
    float t0_217[1];
    float red_buf0_318[1];
    unsigned int mask_319[1];
    float t0_320[1];
    static float red_buf_staging_121[4] __attribute__((shared));
    red_buf0_318[0] = normal_reduce_temp0_15[0];
    mask_319[0] = __activemask();
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 16, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 8, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 4, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 2, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    t0_320[0] = __shfl_down_sync(mask_319[0], red_buf0_318[0], 1, 32);
    red_buf0_318[0] = (red_buf0_318[0] + t0_320[0]);
    if ((((int)threadIdx_x_0) % 32) == 0) {
        red_buf_staging_121[(((int)threadIdx_x_0) >> 5)] = red_buf0_318[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) < 4) {
        red_buf0_215[0] = red_buf_staging_121[((int)threadIdx_x_0)];
    }
    mask_216[0] = (__activemask() & (unsigned int)15);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 2, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    t0_217[0] = __shfl_down_sync(mask_216[0], red_buf0_215[0], 1, 32);
    red_buf0_215[0] = (red_buf0_215[0] + t0_217[0]);
    if (((int)threadIdx_x_0) == 0) {
        ((volatile float *)red_result_16)[0] = red_buf0_215[0];
    }
    asm ("bar.sync 1,128;");
    ;
    if (((int)threadIdx_x_0) == 0) {
        T_softmax_expsum7[0] = ((volatile float *)red_result_16)[0];
    }
    asm ("bar.sync 1,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_0) >> 3)) < 125) {
            T_softmax_norm0[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] = (__expf((data1[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_0))] - T_softmax_maxelem4[0])) / T_softmax_expsum7[0]);
        }
    }
}
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=128 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 256)){
    unsigned int blockDim_x_1 = 128;
    unsigned int threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) % 128;
    unsigned int blockDim_y_1 = 1;
    unsigned int threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) / 128 % 1;
    unsigned int blockDim_z_1 = 1;
    unsigned int threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 128) / 128;
    float normal_reduce_temp024[1];
    static float red_result25[257] __attribute__((shared));
    static float T_softmax_maxelem26[257] __attribute__((shared));
    float normal_reduce_temp0_127[1];
    static float red_result_128[257] __attribute__((shared));
    static float T_softmax_expsum29[257] __attribute__((shared));
    normal_reduce_temp024[0] = -3.40282306E+38F;
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1))]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 128)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 256)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 384)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 512)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 640)]);
    normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 768)]);
    if (((int)threadIdx_x_1) < 104) {
        normal_reduce_temp024[0] = max(normal_reduce_temp024[0], data23[(((((int)blockIdx.x) * 1000) + ((int)threadIdx_x_1)) + 896)]);
    }
    float red_buf030[1];
    unsigned int mask31[1];
    float t032[1];
    float red_buf0_133[1];
    unsigned int mask_134[1];
    float t0_135[1];
    static float red_buf_staging36[4] __attribute__((shared));
    red_buf0_133[0] = normal_reduce_temp024[0];
    mask_134[0] = __activemask();
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 16, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 8, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 4, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 2, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    t0_135[0] = __shfl_down_sync(mask_134[0], red_buf0_133[0], 1, 32);
    red_buf0_133[0] = max(red_buf0_133[0], t0_135[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging36[(((int)threadIdx_x_1) >> 5)] = red_buf0_133[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf030[0] = red_buf_staging36[((int)threadIdx_x_1)];
    }
    mask31[0] = (__activemask() & (unsigned int)15);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 2, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    t032[0] = __shfl_down_sync(mask31[0], red_buf030[0], 1, 32);
    red_buf030[0] = max(red_buf030[0], t032[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result25)[0] = red_buf030[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_maxelem26[0] = ((volatile float *)red_result25)[0];
    }
    normal_reduce_temp0_127[0] = 0.F;
    asm ("bar.sync 2,128;");
    ;
    for (int k_outer = 0; k_outer < 8; ++k_outer) {
        if (((k_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            normal_reduce_temp0_127[0] = (normal_reduce_temp0_127[0] + __expf((data23[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])));
        }
    }
    float red_buf0_237[1];
    unsigned int mask_238[1];
    float t0_239[1];
    float red_buf0_340[1];
    unsigned int mask_341[1];
    float t0_342[1];
    static float red_buf_staging_143[4] __attribute__((shared));
    red_buf0_340[0] = normal_reduce_temp0_127[0];
    mask_341[0] = __activemask();
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 16, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 8, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 4, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 2, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    t0_342[0] = __shfl_down_sync(mask_341[0], red_buf0_340[0], 1, 32);
    red_buf0_340[0] = (red_buf0_340[0] + t0_342[0]);
    if ((((int)threadIdx_x_1) % 32) == 0) {
        red_buf_staging_143[(((int)threadIdx_x_1) >> 5)] = red_buf0_340[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) < 4) {
        red_buf0_237[0] = red_buf_staging_143[((int)threadIdx_x_1)];
    }
    mask_238[0] = (__activemask() & (unsigned int)15);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 2, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    t0_239[0] = __shfl_down_sync(mask_238[0], red_buf0_237[0], 1, 32);
    red_buf0_237[0] = (red_buf0_237[0] + t0_239[0]);
    if (((int)threadIdx_x_1) == 0) {
        ((volatile float *)red_result_128)[0] = red_buf0_237[0];
    }
    asm ("bar.sync 2,128;");
    ;
    if (((int)threadIdx_x_1) == 0) {
        T_softmax_expsum29[0] = ((volatile float *)red_result_128)[0];
    }
    asm ("bar.sync 2,128;");
    ;
    for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
        if (((i2_outer * 16) + (((int)threadIdx_x_1) >> 3)) < 125) {
            T_softmax_norm22[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] = (__expf((data23[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx_x_1))] - T_softmax_maxelem26[0])) / T_softmax_expsum29[0]);
        }
    }
}
}
