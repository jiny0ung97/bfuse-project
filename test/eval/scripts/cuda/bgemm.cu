
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[1];
  __shared__ float A_shared[8];
  __shared__ float B_shared[64];
  float A_shared_local[1];
  float B_shared_local[1];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((int)blockIdx.z) * 512) + (k_outer * 8)) + ((int)threadIdx.x))];
    #pragma unroll
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      B_shared[((ax1_inner * 8) + ((int)threadIdx.x))] = B[(((((((int)blockIdx.z) * 512000) + (((int)blockIdx.x) * 4096)) + (ax1_inner * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      A_shared_local[0] = A_shared[k_inner];
      B_shared_local[0] = B_shared[((((int)threadIdx.x) * 8) + k_inner)];
      T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (A_shared_local[0] * B_shared_local[0]));
    }
  }
  T_batch_matmul_NT[(((((int)blockIdx.z) * 1000) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[0];
}

