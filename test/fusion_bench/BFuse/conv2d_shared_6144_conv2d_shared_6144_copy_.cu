
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(256) void conv2d_shared_6144_conv2d_shared_6144_copy_fused_kernel_bfuse_idx_0(float *__restrict conv2d_shared_6144_A_, float *__restrict conv2d_shared_6144_B_, float *__restrict conv2d_shared_6144_W_, float *__restrict conv2d_shared_6144_copy_A_, float *__restrict conv2d_shared_6144_copy_B_, float *__restrict conv2d_shared_6144_copy_W_)
{
  /*
   * KernelID_ means...
   * 0: conv2d_shared_6144
   * 1: conv2d_shared_6144_copy
   */
  int gridDim_x_;
  int blockIdx_x_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 3024) && ((((int)blockIdx.x - 0) / 84) % 2 == 0))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 1568;
  }
  else if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 3024) && ((((int)blockIdx.x - 0) / 84) % 2 == 1))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 1;
    gridDim_x_ = 1568;
  }
  else if ((int)blockIdx.x >= 3024 && (int)blockIdx.x < 3080)
  {
    NewBlockIdx_ = (int)blockIdx.x - 1512;
    KernelID_  = 0;
    gridDim_x_ = 1568;
  }
  else if ((int)blockIdx.x >= 3080 && (int)blockIdx.x < 3136)
  {
    NewBlockIdx_ = (int)blockIdx.x - 1568;
    KernelID_  = 1;
    gridDim_x_ = 1568;
  }
  blockIdx_x_ = NewBlockIdx_;

  static float union_shared_0_[3072] __attribute__((shared));
  static float union_shared_1_[3072] __attribute__((shared));


  // conv2d_shared_6144
  if ((KernelID_ == 0) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 256)))
  {
      float B_local[64];
      float Apad_shared_local[8];
      float W_shared_local[8];
      for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
          for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
              B_local[((ff_c_init * 4) + nn_c_init)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.F;
          }
      }
      for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
          for (int ry = 0; ry < 3; ++ry) {
              for (int rx = 0; rx < 3; ++rx) {
                  asm ("bar.sync 0, 256;");
                  for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
                      if (((int)threadIdx.x) < 128) {
                          *(float4 *)(union_shared_0_ + ((((int)threadIdx.x) * 8) + (ax3_inner_outer * 4))) = (((((1 <= ((((int)blockIdx_x_) / 112) + ry)) && (((((int)blockIdx_x_) / 112) + ry) < 15)) && (1 <= (((((int)blockIdx_x_) % 112) >> 3) + rx))) && ((((((int)blockIdx_x_) % 112) >> 3) + rx) < 15)) ? *(float4 *)(conv2d_shared_6144_A_ + (((((((((ry * 917504) + ((((int)blockIdx_x_) >> 3) * 65536)) + (rx * 65536)) + (rc_outer * 2048)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer * 4)) - 983040)) : make_float4(0.F, 0.F, 0.F, 0.F));
                      }
                  }
                  for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
                      if (((int)threadIdx.x) < 128) {
                          *(float4 *)(union_shared_1_ + ((((int)threadIdx.x) * 8) + (ax3_inner_outer_1 * 4))) = *(float4 *)(conv2d_shared_6144_W_ + (((((((ry * 393216) + (rx * 131072)) + (rc_outer * 4096)) + ((((int)threadIdx.x) >> 4) * 512)) + (((((int)blockIdx_x_) & 7) >> 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer_1 * 4)));
                      }
                  }
                  asm ("bar.sync 0, 256;");
                  for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
                      for (int ax3 = 0; ax3 < 4; ++ax3) {
                          Apad_shared_local[ax3] = union_shared_0_[(((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3)];
                          Apad_shared_local[(ax3 + 4)] = union_shared_0_[((((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3) + 64)];
                      }
                      for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
                          W_shared_local[ax3_1] = union_shared_1_[(((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1)];
                          W_shared_local[(ax3_1 + 4)] = union_shared_1_[((((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1) + 64)];
                      }
                      for (int ff_c = 0; ff_c < 4; ++ff_c) {
                          for (int nn_c = 0; nn_c < 4; ++nn_c) {
                              B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
                              B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
                          }
                      }
                  }
              }
          }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
          for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
              conv2d_shared_6144_B_[(((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
              conv2d_shared_6144_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16384)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
              conv2d_shared_6144_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
              conv2d_shared_6144_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16448)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
          }
      }
  }
  // conv2d_shared_6144_copy
  else if ((KernelID_ == 1) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 256)))
  {
      float B_local[64];
      float Apad_shared_local[8];
      float W_shared_local[8];
      for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
          for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
              B_local[((ff_c_init * 4) + nn_c_init)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.F;
          }
      }
      for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
          for (int ry = 0; ry < 3; ++ry) {
              for (int rx = 0; rx < 3; ++rx) {
                  asm ("bar.sync 0, 256;");
                  for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
                      if (((int)threadIdx.x) < 128) {
                          *(float4 *)(union_shared_0_ + ((((int)threadIdx.x) * 8) + (ax3_inner_outer * 4))) = (((((1 <= ((((int)blockIdx_x_) / 112) + ry)) && (((((int)blockIdx_x_) / 112) + ry) < 15)) && (1 <= (((((int)blockIdx_x_) % 112) >> 3) + rx))) && ((((((int)blockIdx_x_) % 112) >> 3) + rx) < 15)) ? *(float4 *)(conv2d_shared_6144_copy_A_ + (((((((((ry * 917504) + ((((int)blockIdx_x_) >> 3) * 65536)) + (rx * 65536)) + (rc_outer * 2048)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer * 4)) - 983040)) : make_float4(0.F, 0.F, 0.F, 0.F));
                      }
                  }
                  for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
                      if (((int)threadIdx.x) < 128) {
                          *(float4 *)(union_shared_1_ + ((((int)threadIdx.x) * 8) + (ax3_inner_outer_1 * 4))) = *(float4 *)(conv2d_shared_6144_copy_W_ + (((((((ry * 393216) + (rx * 131072)) + (rc_outer * 4096)) + ((((int)threadIdx.x) >> 4) * 512)) + (((((int)blockIdx_x_) & 7) >> 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer_1 * 4)));
                      }
                  }
                  asm ("bar.sync 0, 256;");
                  for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
                      for (int ax3 = 0; ax3 < 4; ++ax3) {
                          Apad_shared_local[ax3] = union_shared_0_[(((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3)];
                          Apad_shared_local[(ax3 + 4)] = union_shared_0_[((((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3) + 64)];
                      }
                      for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
                          W_shared_local[ax3_1] = union_shared_1_[(((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1)];
                          W_shared_local[(ax3_1 + 4)] = union_shared_1_[((((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1) + 64)];
                      }
                      for (int ff_c = 0; ff_c < 4; ++ff_c) {
                          for (int nn_c = 0; nn_c < 4; ++nn_c) {
                              B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
                              B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
                          }
                      }
                  }
              }
          }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
          for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
              conv2d_shared_6144_copy_B_[(((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
              conv2d_shared_6144_copy_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16384)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
              conv2d_shared_6144_copy_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
              conv2d_shared_6144_copy_B_[((((((((((int)blockIdx_x_) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx_x_) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16448)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
          }
      }
  }
}
