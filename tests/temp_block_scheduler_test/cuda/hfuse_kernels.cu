
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ __launch_bounds__(96) void bgemm_0_conv2d_0_fused_hfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_0_conv2d_nchw_, float *__restrict conv2d_0_data_, float *__restrict conv2d_0_kernel_)
{
  uint streamingMultiprocessorId;
  asm("mov.u32 %0, %smid;" : "=r"(streamingMultiprocessorId));
  printf("Block: %d | SM: %d - Here!\n", blockIdx.x, streamingMultiprocessorId);

  // bgemm_0
  if (((int)threadIdx.x >= 0 && (int)threadIdx.x < 64) && ((int)blockIdx.x >= 0 && (int)blockIdx.x < 1024))
  {
      int blockIdx_x_ = (int)blockIdx.x % 8;
      int blockIdx_y_ = (int)blockIdx.x / 8 % 8;
      int blockIdx_z_ = (int)blockIdx.x / 64;
      int threadIdx_x_ = ((int)threadIdx.x - 0) % 8;
      int threadIdx_y_ = ((int)threadIdx.x - 0) / 8 % 8;
      int threadIdx_z_ = ((int)threadIdx.x - 0) / 64;

      float T_batch_matmul_NT_local[64];
      static float A_shared[512] __attribute__((shared));
      static float B_shared[512] __attribute__((shared));
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          asm ("bar.sync 1, 64;");
          A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          asm ("bar.sync 1, 64;");
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
  }
  // conv2d_0
  if (((int)threadIdx.x >= 64 && (int)threadIdx.x < 80) && ((int)blockIdx.x >= 0 && (int)blockIdx.x < 1815))
  {
      int blockIdx_x_ = (int)blockIdx.x % 55;
      int blockIdx_y_ = (int)blockIdx.x / 55 % 11;
      int blockIdx_z_ = (int)blockIdx.x / 605;
      int threadIdx_x_ = ((int)threadIdx.x - 64) % 1;
      int threadIdx_y_ = ((int)threadIdx.x - 64) / 1 % 1;
      int threadIdx_z_ = ((int)threadIdx.x - 64) / 1;

      float conv2d_nchw_local[20];
      static float pad_temp_shared[18] __attribute__((shared));
      static float kernel_shared[128] __attribute__((shared));
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[16] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[18] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[17] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[19] = 0.F;
      for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[(((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2))];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[(((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2))];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18))];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 9)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 576)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 585)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1152)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1161)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1728)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1737)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 1)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 1)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 10)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 577)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 586)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1153)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1162)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1729)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1738)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 2)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 2)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 2)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 11)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 578)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 587)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1154)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1163)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1730)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1739)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 112)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 112)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 3)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 12)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 579)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 588)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1155)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1164)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1731)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1740)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 113)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 113)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 4)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 13)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 580)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 589)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1156)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1165)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1732)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1741)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 114)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 114)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 5)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 14)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 581)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 590)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1157)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1166)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1733)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1742)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 224)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 224)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 6)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 15)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 582)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 591)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1158)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1167)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1734)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1743)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 225)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 225)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 7)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 16)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 583)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 592)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1159)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1168)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1735)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1744)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          asm ("bar.sync 2, 32;");
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((rc_outer * 25088) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 226)];
          }
          if (((int)threadIdx_z_) < 9) {
              pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((rc_outer * 25088) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 226)];
          }
          kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 8)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 17)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 584)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 593)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1160)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1169)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1736)];
          kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[((((((int)blockIdx_z_) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1745)];
          asm ("bar.sync 2, 32;");
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
      }
      conv2d_0_conv2d_nchw_[((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_))] = conv2d_nchw_local[0];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96800)] = conv2d_nchw_local[10];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 55)] = conv2d_nchw_local[2];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96855)] = conv2d_nchw_local[12];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 110)] = conv2d_nchw_local[4];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96910)] = conv2d_nchw_local[14];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 165)] = conv2d_nchw_local[6];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96965)] = conv2d_nchw_local[16];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 220)] = conv2d_nchw_local[8];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 97020)] = conv2d_nchw_local[18];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3025)] = conv2d_nchw_local[1];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99825)] = conv2d_nchw_local[11];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3080)] = conv2d_nchw_local[3];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99880)] = conv2d_nchw_local[13];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3135)] = conv2d_nchw_local[5];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99935)] = conv2d_nchw_local[15];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3190)] = conv2d_nchw_local[7];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99990)] = conv2d_nchw_local[17];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3245)] = conv2d_nchw_local[9];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 100045)] = conv2d_nchw_local[19];
  }
}
