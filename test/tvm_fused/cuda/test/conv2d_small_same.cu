
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64, 8) conv2d_small(float* __restrict__ A, float* __restrict__ B, float* __restrict__ W) {
  float B_local[256];
  __shared__ float Apad_shared[2048];
  __shared__ float W_shared[1024];
  float Apad_shared_local[16];
  float W_shared_local[16];
  for (int ff_c_init = 0; ff_c_init < 8; ++ff_c_init) {
    for (int nn_c_init = 0; nn_c_init < 8; ++nn_c_init) {
      B_local[((ff_c_init * 8) + nn_c_init)] = 0.000000e+00f;
      B_local[(((ff_c_init * 8) + nn_c_init) + 128)] = 0.000000e+00f;
      B_local[(((ff_c_init * 8) + nn_c_init) + 64)] = 0.000000e+00f;
      B_local[(((ff_c_init * 8) + nn_c_init) + 192)] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner_outer = 0; ax3_inner_outer < 4; ++ax3_inner_outer) {
        *(float4*)(Apad_shared + ((((((int)threadIdx.y) * 256) + (ax2_inner * 128)) + (((int)threadIdx.x) * 16)) + (ax3_inner_outer * 4))) = (((((58 <= ((int)blockIdx.z)) && (((int)blockIdx.z) < 3306)) && (1 <= (((int)blockIdx.z) % 58))) && ((((int)blockIdx.z) % 58) < 57)) ? *(float4*)(A + (((((((((((int)blockIdx.z) / 58) * 458752) + ((((int)blockIdx.z) % 58) * 8192)) + (rc_outer * 2048)) + (((int)threadIdx.y) * 256)) + (ax2_inner * 128)) + (((int)threadIdx.x) * 16)) + (ax3_inner_outer * 4)) - 466944)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
      }
    }
    for (int ax2_inner_1 = 0; ax2_inner_1 < 2; ++ax2_inner_1) {
      for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
        *(float4*)(W_shared + ((((((int)threadIdx.y) * 128) + (ax2_inner_1 * 64)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer_1 * 4))) = *(float4*)(W + (((((rc_outer * 1024) + (((int)threadIdx.y) * 128)) + (ax2_inner_1 * 64)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer_1 * 4)));
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      for (int ax3 = 0; ax3 < 8; ++ax3) {
        Apad_shared_local[ax3] = Apad_shared[(((rc_inner * 128) + (((int)threadIdx.x) * 8)) + ax3)];
        Apad_shared_local[(ax3 + 8)] = Apad_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 8)) + ax3) + 64)];
      }
      for (int ax3_1 = 0; ax3_1 < 8; ++ax3_1) {
        W_shared_local[ax3_1] = W_shared[(((rc_inner * 64) + (((int)threadIdx.y) * 8)) + ax3_1)];
      }
      for (int ff_c = 0; ff_c < 8; ++ff_c) {
        for (int nn_c = 0; nn_c < 8; ++nn_c) {
          B_local[((ff_c * 8) + nn_c)] = (B_local[((ff_c * 8) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
          B_local[(((ff_c * 8) + nn_c) + 64)] = (B_local[(((ff_c * 8) + nn_c) + 64)] + (Apad_shared_local[(nn_c + 8)] * W_shared_local[ff_c]));
        }
      }
    }
  }
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 8; ++ff_inner_inner_inner) {
    for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 8; ++nn_inner_inner_inner) {
      B[(((((((int)blockIdx.z) * 8192) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx.x) * 8)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 8) + nn_inner_inner_inner)];
      B[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx.x) * 8)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 8) + nn_inner_inner_inner) + 64)];
    }
  }
}

