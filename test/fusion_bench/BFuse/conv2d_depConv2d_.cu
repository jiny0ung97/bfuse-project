
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(112, 4) void conv2d_depConv2d_fused_kernel_bfuse_idx_0(float *__restrict conv2d_conv2d_nchw_, float *__restrict conv2d_data_, float *__restrict conv2d_kernel_, float *__restrict depConv2d_DepthwiseConv2d_, float *__restrict depConv2d_data_, float *__restrict depConv2d_kernel_)
{
  /*
   * KernelID_ means...
   * 0: conv2d
   * 1: depConv2d
   * Kernel's Thread Blocks are 19152
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_ = -1;
  
//   if (((int)blockIdx.x < 16016) && ((int)blockIdx.x % 1596 / 84 >= 0) && ((int)blockIdx.x % 1596 / 84 < 1))
//   {
//     NewBlockIdx_ = ((int)blockIdx.x / 1596) * 84 + (int)blockIdx.x % 1596 - 0;
//     KernelID_  = 0;
//     gridDim_x_ = 1;
//     gridDim_y_ = 28;
//     gridDim_z_ = 32;
//     blockDim_x_ = 56;
//     blockDim_y_ = 1;
//     blockDim_z_ = 2;
//   }
//   else if (((int)blockIdx.x < 17308) && ((int)blockIdx.x % 1596 / 84 >= 1) && ((int)blockIdx.x % 1596 / 84 < 19))
//   {
//     NewBlockIdx_ = ((int)blockIdx.x / 1596) * 1512 + (int)blockIdx.x % 1596 - 84;
//     KernelID_  = 1;
//     gridDim_x_ = 1;
//     gridDim_y_ = 1;
//     gridDim_z_ = 16384;
//     blockDim_x_ = 28;
//     blockDim_y_ = 4;
//     blockDim_z_ = 1;
//   }
//   blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
//   blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
//   blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
//   threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
//   threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
//   threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

  static float union_shared_0_[3249] __attribute__((shared));
  static float union_shared_1_[256] __attribute__((shared));


  // conv2d
  if (((int)threadIdx.x < 112) && ((int)blockIdx.x < 16016) && ((int)blockIdx.x % 1596 / 84 >= 0) && ((int)blockIdx.x % 1596 / 84 < 1))
  {
      NewBlockIdx_ = ((int)blockIdx.x / 1596) * 84 + (int)blockIdx.x % 1596 - 0;
      KernelID_  = 0;
      gridDim_x_ = 1;
      gridDim_y_ = 28;
      gridDim_z_ = 32;
      blockDim_x_ = 56;
      blockDim_y_ = 1;
      blockDim_z_ = 2;
      blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
      blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
      blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
      threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
      threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
      threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);
      float conv2d_nchw_local[64];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[16] = 0.F;
      conv2d_nchw_local[18] = 0.F;
      conv2d_nchw_local[20] = 0.F;
      conv2d_nchw_local[22] = 0.F;
      conv2d_nchw_local[24] = 0.F;
      conv2d_nchw_local[26] = 0.F;
      conv2d_nchw_local[28] = 0.F;
      conv2d_nchw_local[30] = 0.F;
      conv2d_nchw_local[32] = 0.F;
      conv2d_nchw_local[34] = 0.F;
      conv2d_nchw_local[36] = 0.F;
      conv2d_nchw_local[38] = 0.F;
      conv2d_nchw_local[40] = 0.F;
      conv2d_nchw_local[42] = 0.F;
      conv2d_nchw_local[44] = 0.F;
      conv2d_nchw_local[46] = 0.F;
      conv2d_nchw_local[48] = 0.F;
      conv2d_nchw_local[50] = 0.F;
      conv2d_nchw_local[52] = 0.F;
      conv2d_nchw_local[54] = 0.F;
      conv2d_nchw_local[56] = 0.F;
      conv2d_nchw_local[58] = 0.F;
      conv2d_nchw_local[60] = 0.F;
      conv2d_nchw_local[62] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      conv2d_nchw_local[17] = 0.F;
      conv2d_nchw_local[19] = 0.F;
      conv2d_nchw_local[21] = 0.F;
      conv2d_nchw_local[23] = 0.F;
      conv2d_nchw_local[25] = 0.F;
      conv2d_nchw_local[27] = 0.F;
      conv2d_nchw_local[29] = 0.F;
      conv2d_nchw_local[31] = 0.F;
      conv2d_nchw_local[33] = 0.F;
      conv2d_nchw_local[35] = 0.F;
      conv2d_nchw_local[37] = 0.F;
      conv2d_nchw_local[39] = 0.F;
      conv2d_nchw_local[41] = 0.F;
      conv2d_nchw_local[43] = 0.F;
      conv2d_nchw_local[45] = 0.F;
      conv2d_nchw_local[47] = 0.F;
      conv2d_nchw_local[49] = 0.F;
      conv2d_nchw_local[51] = 0.F;
      conv2d_nchw_local[53] = 0.F;
      conv2d_nchw_local[55] = 0.F;
      conv2d_nchw_local[57] = 0.F;
      conv2d_nchw_local[59] = 0.F;
      conv2d_nchw_local[61] = 0.F;
      conv2d_nchw_local[63] = 0.F;
      for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
          for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
              __syncthreads();
              union_shared_0_[((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4))] = ((((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) && (1 <= (((int)threadIdx_x_) % 14))) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 57)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 1)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 56)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 2)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 55)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 3)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 54)] : 0.F);
              if ((((((int)threadIdx_x_) * 3) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3))] = conv2d_kernel_[(((((((int)threadIdx_z_) * 18432) + (((((int)threadIdx_x_) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx_x_) * 3) & 3) * 9)) + (ry_outer * 3))];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 1) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 1)] = conv2d_kernel_[(((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 1) & 3) * 9)) + (ry_outer * 3))];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 2) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 42) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 2)] = conv2d_kernel_[(((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 2) & 3) * 9)) + (ry_outer * 3))];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
              __syncthreads();
              union_shared_0_[((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4))] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 56)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 1)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 55)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 2)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 54)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 3)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 53)] : 0.F);
              if ((((((int)threadIdx_x_) * 3) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3))] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + (((((int)threadIdx_x_) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx_x_) * 3) & 3) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 1) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 1)] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 1) & 3) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 2) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 42) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 2)] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 2) & 3) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
              __syncthreads();
              union_shared_0_[((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4))] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 55)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 1)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 54)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 2)] = (((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 53)] : 0.F);
              union_shared_0_[(((((int)threadIdx_z_) * 224) + (((int)threadIdx_x_) * 4)) + 3)] = ((((1 <= (((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx_y_) * 2) + ((((int)threadIdx_x_) % 28) / 14)) + ry_outer) < 57)) && ((((int)threadIdx_x_) % 14) < 13)) ? conv2d_data_[((((((((((int)blockIdx_z_) * 200704) + (rc_outer * 12544)) + (((int)threadIdx_z_) * 6272)) + ((((int)threadIdx_x_) / 28) * 3136)) + (((int)blockIdx_y_) * 112)) + (ry_outer * 56)) + ((((int)threadIdx_x_) % 28) * 4)) - 52)] : 0.F);
              if ((((((int)threadIdx_x_) * 3) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3))] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + (((((int)threadIdx_x_) * 3) >> 2) * 576)) + (rc_outer * 36)) + (((((int)threadIdx_x_) * 3) & 3) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 1) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 43) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 1)] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 1) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 1) & 3) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              if (((((((int)threadIdx_x_) * 3) + 2) >> 7) + ((int)threadIdx_z_)) < 2) {
                  if (((int)threadIdx_x_) < 42) {
                      union_shared_1_[(((((int)threadIdx_z_) * 128) + (((int)threadIdx_x_) * 3)) + 2)] = conv2d_kernel_[((((((((int)threadIdx_z_) * 18432) + ((((((int)threadIdx_x_) * 3) + 2) >> 2) * 576)) + (rc_outer * 36)) + ((((((int)threadIdx_x_) * 3) + 2) & 3) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[((int)threadIdx_x_)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[(((int)threadIdx_z_) * 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 8)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 16)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 24)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 32)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 40)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 48)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 56)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 64)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 72)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 80)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 88)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 96)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 104)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 112)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 120)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 128)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 136)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 144)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 152)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 160)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 168)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 176)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 184)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 192)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 200)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 208)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 216)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 224)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 232)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 240)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 56)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 248)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 112)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 9)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 17)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 25)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 33)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 41)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 49)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 57)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 65)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 73)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 81)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 89)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 97)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 105)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 113)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 121)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 129)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 137)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 145)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 153)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 161)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 169)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 177)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 185)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 193)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 201)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 209)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 217)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 225)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 233)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 241)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 168)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 249)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 224)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 10)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 18)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 26)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 34)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 42)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 50)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 58)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 66)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 74)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 82)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 90)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 98)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 106)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 114)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 122)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 130)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 138)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 146)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 154)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 162)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 170)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 178)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 186)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 194)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 202)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 210)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 218)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 226)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 234)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 242)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 280)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 250)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[56] = (conv2d_nchw_local[56] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[58] = (conv2d_nchw_local[58] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[60] = (conv2d_nchw_local[60] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[62] = (conv2d_nchw_local[62] + (union_shared_0_[(((int)threadIdx_x_) + 336)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 11)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 19)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 27)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 35)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 43)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 51)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 59)]));
              conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 67)]));
              conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 75)]));
              conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 83)]));
              conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 91)]));
              conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 99)]));
              conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 107)]));
              conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 115)]));
              conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 123)]));
              conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 131)]));
              conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 139)]));
              conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 147)]));
              conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 155)]));
              conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 163)]));
              conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 171)]));
              conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 179)]));
              conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 187)]));
              conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 195)]));
              conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 203)]));
              conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 211)]));
              conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 219)]));
              conv2d_nchw_local[57] = (conv2d_nchw_local[57] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 227)]));
              conv2d_nchw_local[59] = (conv2d_nchw_local[59] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 235)]));
              conv2d_nchw_local[61] = (conv2d_nchw_local[61] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 243)]));
              conv2d_nchw_local[63] = (conv2d_nchw_local[63] + (union_shared_0_[(((int)threadIdx_x_) + 392)] * union_shared_1_[((((int)threadIdx_z_) * 4) + 251)]));
          }
      }
      conv2d_conv2d_nchw_[((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 6272)] = conv2d_nchw_local[2];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 12544)] = conv2d_nchw_local[4];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 18816)] = conv2d_nchw_local[6];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 25088)] = conv2d_nchw_local[8];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 31360)] = conv2d_nchw_local[10];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 37632)] = conv2d_nchw_local[12];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 43904)] = conv2d_nchw_local[14];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 50176)] = conv2d_nchw_local[16];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 56448)] = conv2d_nchw_local[18];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 62720)] = conv2d_nchw_local[20];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 68992)] = conv2d_nchw_local[22];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 75264)] = conv2d_nchw_local[24];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 81536)] = conv2d_nchw_local[26];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 87808)] = conv2d_nchw_local[28];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 94080)] = conv2d_nchw_local[30];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 100352)] = conv2d_nchw_local[32];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 106624)] = conv2d_nchw_local[34];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 112896)] = conv2d_nchw_local[36];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 119168)] = conv2d_nchw_local[38];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 125440)] = conv2d_nchw_local[40];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 131712)] = conv2d_nchw_local[42];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 137984)] = conv2d_nchw_local[44];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 144256)] = conv2d_nchw_local[46];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 150528)] = conv2d_nchw_local[48];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 156800)] = conv2d_nchw_local[50];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 163072)] = conv2d_nchw_local[52];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 169344)] = conv2d_nchw_local[54];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 175616)] = conv2d_nchw_local[56];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 181888)] = conv2d_nchw_local[58];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 188160)] = conv2d_nchw_local[60];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 194432)] = conv2d_nchw_local[62];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 56)] = conv2d_nchw_local[1];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 6328)] = conv2d_nchw_local[3];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 12600)] = conv2d_nchw_local[5];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 18872)] = conv2d_nchw_local[7];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 25144)] = conv2d_nchw_local[9];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 31416)] = conv2d_nchw_local[11];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 37688)] = conv2d_nchw_local[13];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 43960)] = conv2d_nchw_local[15];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 50232)] = conv2d_nchw_local[17];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 56504)] = conv2d_nchw_local[19];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 62776)] = conv2d_nchw_local[21];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 69048)] = conv2d_nchw_local[23];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 75320)] = conv2d_nchw_local[25];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 81592)] = conv2d_nchw_local[27];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 87864)] = conv2d_nchw_local[29];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 94136)] = conv2d_nchw_local[31];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 100408)] = conv2d_nchw_local[33];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 106680)] = conv2d_nchw_local[35];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 112952)] = conv2d_nchw_local[37];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 119224)] = conv2d_nchw_local[39];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 125496)] = conv2d_nchw_local[41];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 131768)] = conv2d_nchw_local[43];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 138040)] = conv2d_nchw_local[45];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 144312)] = conv2d_nchw_local[47];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 150584)] = conv2d_nchw_local[49];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 156856)] = conv2d_nchw_local[51];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 163128)] = conv2d_nchw_local[53];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 169400)] = conv2d_nchw_local[55];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 175672)] = conv2d_nchw_local[57];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 181944)] = conv2d_nchw_local[59];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 188216)] = conv2d_nchw_local[61];
      conv2d_conv2d_nchw_[(((((((int)blockIdx_z_) * 200704) + (((int)threadIdx_z_) * 3136)) + (((int)blockIdx_y_) * 112)) + ((int)threadIdx_x_)) + 194488)] = conv2d_nchw_local[63];
  }
  // depConv2d
  else if (((int)threadIdx.x < 112) && ((int)blockIdx.x < 17308) && ((int)blockIdx.x % 1596 / 84 >= 1) && ((int)blockIdx.x % 1596 / 84 < 19))
  {
      NewBlockIdx_ = ((int)blockIdx.x / 1596) * 1512 + (int)blockIdx.x % 1596 - 84;
      KernelID_  = 1;
      gridDim_x_ = 1;
      gridDim_y_ = 1;
      gridDim_z_ = 16384;
      blockDim_x_ = 28;
      blockDim_y_ = 4;
      blockDim_z_ = 1;
      blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
      blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
      blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
      threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
      threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
      threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);
      float PaddedInput_shared_local[45];
      float kernel_shared_local[9];
      float DepthwiseConv2d_local[7];
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 30; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
          if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_)) < 3249) {
              if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 4) + ((int)threadIdx_y_)) < 117) {
                  union_shared_0_[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_))] = (((57 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_)) % 57))) ? depConv2d_data_[((((((int)blockIdx_z_) * 3136) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_)) / 57) * 56)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx_y_) * 28)) + ((int)threadIdx_x_)) % 57)) - 57)] : 0.F);
              }
          }
      }
      if (((((int)threadIdx_y_) * 28) + ((int)threadIdx_x_)) < 9) {
          if (((int)threadIdx_y_) < 1) {
              union_shared_1_[((((int)threadIdx_y_) * 28) + ((int)threadIdx_x_))] = depConv2d_kernel_[(((((int)threadIdx_y_) * 28) + ((((int)blockIdx_z_) & 127) * 9)) + ((int)threadIdx_x_))];
          }
      }
      __syncthreads();
      for (int ax2 = 0; ax2 < 15; ++ax2) {
          for (int ax3 = 0; ax3 < 3; ++ax3) {
              PaddedInput_shared_local[((ax2 * 3) + ax3)] = union_shared_0_[((((((int)threadIdx_y_) * 798) + (ax2 * 57)) + (((int)threadIdx_x_) * 2)) + ax3)];
          }
      }
      for (int ax2_1 = 0; ax2_1 < 3; ++ax2_1) {
          for (int ax3_1 = 0; ax3_1 < 3; ++ax3_1) {
              kernel_shared_local[((ax2_1 * 3) + ax3_1)] = union_shared_1_[((ax2_1 * 3) + ax3_1)];
          }
      }
      for (int i_c = 0; i_c < 7; ++i_c) {
          DepthwiseConv2d_local[i_c] = 0.F;
          for (int di = 0; di < 3; ++di) {
              for (int dj = 0; dj < 3; ++dj) {
                  DepthwiseConv2d_local[i_c] = (DepthwiseConv2d_local[i_c] + (PaddedInput_shared_local[(((i_c * 6) + (di * 3)) + dj)] * kernel_shared_local[((di * 3) + dj)]));
              }
          }
      }
      for (int i_inner_inner_inner = 0; i_inner_inner_inner < 7; ++i_inner_inner_inner) {
          depConv2d_DepthwiseConv2d_[((((((int)blockIdx_z_) * 784) + (((int)threadIdx_y_) * 196)) + (i_inner_inner_inner * 28)) + ((int)threadIdx_x_))] = DepthwiseConv2d_local[i_inner_inner_inner];
      }
  }
}
