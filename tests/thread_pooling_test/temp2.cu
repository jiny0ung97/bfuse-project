#include "hip/hip_runtime.h"
// TODO:

#include <stdio.h>

extern "C" __global__ __launch_bounds__(104) void bgemm_0_conv2d_3_fused_bfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_3_conv2d_nchw_, float *__restrict conv2d_3_data_, float *__restrict conv2d_3_kernel_)
{
  uint streamingMultiprocessorId;
  asm("mov.u32 %0, %smid;" : "=r"(streamingMultiprocessorId));
  printf("Block: %d | SM: %d - Here!\n", blockIdx.x, streamingMultiprocessorId);

  typedef struct bgemm_0 {
    float A_shared[512];
    float B_shared[512];
  } bgemm_0_Ty_;
  typedef struct conv2d_3 {
    float pad_temp_shared[208];
    float kernel_shared[512];
  } conv2d_3_Ty_;
  typedef union bgemm_0_conv2d_3 {
    bgemm_0_Ty_ bgemm_0;
    conv2d_3_Ty_ conv2d_3;
  } bgemm_0_conv2d_3_Ty_;

  __shared__ bgemm_0_conv2d_3_Ty_ SU_;

  // bgemm_0
  // GridSize  = 2048
  // BlockSize = 64
  // per block = 25
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 84) && ((int)threadIdx.x < 64))
  {
    int blocksPerBlock  = 25;
    int startBlockBound = (int)blockIdx.x * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x + 1) * blocksPerBlock;
    if (endBlockBound >= 2048) {
        endBlockBound = 2048;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
        int blockIdx_  = Idx;
        int threadIdx_ = (int)threadIdx.x;

        int gridDim_x_  = 8;
        int gridDim_y_  = 8;
        int gridDim_z_  = 32;
        int blockDim_x_ = 8;
        int blockDim_y_ = 8;
        int blockDim_z_ = 1;

        int blockIdx_x_  = blockIdx_ % gridDim_x_;
        int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
        int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
        int threadIdx_x_ = threadIdx_ % blockDim_x_;
        int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
        int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

        float T_batch_matmul_NT_local[64];
        float A_shared_local[8];
        float B_shared_local[8];
        for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
            for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
                T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
            }
        }
        for (int k_outer = 0; k_outer < 8; ++k_outer) {
            __syncthreads();
            SU.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
            SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
            SU.bgemm_0.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
            SU.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
            __syncthreads();
            for (int k_inner = 0; k_inner < 8; ++k_inner) {
                A_shared_local[0] = SU.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
                A_shared_local[1] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
                A_shared_local[2] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
                A_shared_local[3] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
                A_shared_local[4] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
                A_shared_local[5] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
                A_shared_local[6] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
                A_shared_local[7] = SU.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
                B_shared_local[0] = SU.bgemm_0.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
                B_shared_local[1] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
                B_shared_local[2] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
                B_shared_local[3] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
                B_shared_local[4] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
                B_shared_local[5] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
                B_shared_local[6] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
                B_shared_local[7] = SU.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
                for (int i_c = 0; i_c < 8; ++i_c) {
                    T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                    T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                    T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                    T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                    T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                    T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                    T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                    T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
                }
            }
        }
        for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
            bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
            bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
        }
    }
  }
  // conv2d_3
  // GridSize  = 416
  // BlockSize = 104
  // per block = 5
  else if (((int)blockIdx.x >= 84 && (int)blockIdx.x < 168) && ((int)threadIdx.x < 104))
  {
    int blocksPerBlock  = 5;
    int startBlockBound = ((int)blockIdx.x - 84) * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x - 84 + 1) * blocksPerBlock;
    if (endBlockBound >= 416) {
        endBlockBound = 416;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
        int blockIdx_  = Idx;
        int threadIdx_ = (int)threadIdx.x;

        int gridDim_x_  = 2;
        int gridDim_y_  = 13;
        int gridDim_z_  = 16;
        int blockDim_x_ = 13;
        int blockDim_y_ = 1;
        int blockDim_z_ = 8;

        int blockIdx_x_  = blockIdx_ % gridDim_x_;
        int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
        int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
        int threadIdx_x_ = threadIdx_ % blockDim_x_;
        int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
        int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

        float conv2d_nchw_local[16];
        conv2d_nchw_local[0] = 0.F;
        conv2d_nchw_local[2] = 0.F;
        conv2d_nchw_local[4] = 0.F;
        conv2d_nchw_local[6] = 0.F;
        conv2d_nchw_local[8] = 0.F;
        conv2d_nchw_local[10] = 0.F;
        conv2d_nchw_local[12] = 0.F;
        conv2d_nchw_local[14] = 0.F;
        conv2d_nchw_local[1] = 0.F;
        conv2d_nchw_local[3] = 0.F;
        conv2d_nchw_local[5] = 0.F;
        conv2d_nchw_local[7] = 0.F;
        conv2d_nchw_local[9] = 0.F;
        conv2d_nchw_local[11] = 0.F;
        conv2d_nchw_local[13] = 0.F;
        conv2d_nchw_local[15] = 0.F;
        for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
            for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
                __syncthreads();
                SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13))];
                SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13))];
                SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3))];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3))];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3))];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3))];
                if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                    if (((int)threadIdx_x_) < 12) {
                        SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3))];
                    }
                }
                __syncthreads();
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
                __syncthreads();
                SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 1)];
                SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 1)];
                SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 1)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1)];
                if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                    if (((int)threadIdx_x_) < 12) {
                        SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1)];
                    }
                }
                __syncthreads();
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
                __syncthreads();
                SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 2)];
                SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 2)];
                SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 2)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2)];
                SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2)];
                if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                    if (((int)threadIdx_x_) < 12) {
                        SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2)];
                    }
                }
                __syncthreads();
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
                conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
                conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
                conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
                conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
                conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
                conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
                conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
                conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
                conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
            }
        }
        conv2d_3_conv2d_nchw_[(((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5408)] = conv2d_nchw_local[2];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10816)] = conv2d_nchw_local[4];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16224)] = conv2d_nchw_local[6];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21632)] = conv2d_nchw_local[8];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27040)] = conv2d_nchw_local[10];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32448)] = conv2d_nchw_local[12];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37856)] = conv2d_nchw_local[14];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 26)] = conv2d_nchw_local[1];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5434)] = conv2d_nchw_local[3];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10842)] = conv2d_nchw_local[5];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16250)] = conv2d_nchw_local[7];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21658)] = conv2d_nchw_local[9];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27066)] = conv2d_nchw_local[11];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32474)] = conv2d_nchw_local[13];
        conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37882)] = conv2d_nchw_local[15];
    }
  }
}

// TODO:

extern "C" __global__ __launch_bounds__(104) void bgemm_1_conv2d_3_fused_bfuse(float *__restrict bgemm_1_A_, float *__restrict bgemm_1_B_, float *__restrict bgemm_1_T_batch_matmul_NT_, float *__restrict conv2d_3_conv2d_nchw_, float *__restrict conv2d_3_data_, float *__restrict conv2d_3_kernel_)
{
  uint streamingMultiprocessorId;
  asm("mov.u32 %0, %smid;" : "=r"(streamingMultiprocessorId));
  printf("Block: %d | SM: %d - Here!\n", blockIdx.x, streamingMultiprocessorId);

  typedef struct bgemm_1 {
    float A_shared[512];
    float B_shared[512];
  } bgemm_1_Ty_;
  typedef struct conv2d_3 {
    float pad_temp_shared[208];
    float kernel_shared[512];
  } conv2d_3_Ty_;
  typedef union bgemm_1_conv2d_3 {
    bgemm_1_Ty_ bgemm_1;
    conv2d_3_Ty_ conv2d_3;
  } bgemm_1_conv2d_3_Ty_;

  __shared__ bgemm_1_conv2d_3_Ty_ SU_;

  // bgemm_1
  // GridSize  = 256
  // BlockSize = 64
  // per block = 4
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 84) && ((int)threadIdx.x < 64))
  {
    int blocksPerBlock  = 4;
    int startBlockBound = (int)blockIdx.x * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x + 1) * blocksPerBlock;
    if (endBlockBound >= 256) {
        endBlockBound = 256;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
      int blockIdx_  = Idx;
      int threadIdx_ = (int)threadIdx.x;

      int gridDim_x_  = 1;
      int gridDim_y_  = 8;
      int gridDim_z_  = 32;
      int blockDim_x_ = 8;
      int blockDim_y_ = 8;
      int blockDim_z_ = 1;

      int blockIdx_x_  = blockIdx_ % gridDim_x_;
      int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
      int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
      int threadIdx_x_ = threadIdx_ % blockDim_x_;
      int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
      int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

      float T_batch_matmul_NT_local[64];
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 64; ++k_outer) {
          __syncthreads();
          SU.bgemm_1.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_1_A_[(((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 512)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 1024)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 1536)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 2048)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 2560)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 3072)];
          SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_1_A_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 3584)];
          SU.bgemm_1.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_1_B_[((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 512)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 1024)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 1536)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 2048)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 2560)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 3072)];
          SU.bgemm_1.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_1_B_[(((((((int)blockIdx_z_) * 32768) + (((int)threadIdx_y_) * 4096)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 3584)];
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = SU.bgemm_1.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = SU.bgemm_1.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = SU.bgemm_1.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = SU.bgemm_1.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_1_T_batch_matmul_NT_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_1_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (i_inner_inner * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
    }
  }
  // conv2d_3
  // GridSize  = 416
  // BlockSize = 104
  // per block = 5
  else if (((int)blockIdx.x >= 84 && (int)blockIdx.x < 168) && ((int)threadIdx.x < 104))
  {
    int blocksPerBlock  = 5;
    int startBlockBound = ((int)blockIdx.x - 84) * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x - 84 + 1) * blocksPerBlock;
    if (endBlockBound >= 416) {
        endBlockBound = 416;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
      int blockIdx_  = Idx;
      int threadIdx_ = (int)threadIdx.x;

      int gridDim_x_  = 2;
      int gridDim_y_  = 13;
      int gridDim_z_  = 16;
      int blockDim_x_ = 13;
      int blockDim_y_ = 1;
      int blockDim_z_ = 8;

      int blockIdx_x_  = blockIdx_ % gridDim_x_;
      int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
      int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
      int threadIdx_x_ = threadIdx_ % blockDim_x_;
      int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
      int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

      float conv2d_nchw_local[16];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
          for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13))];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13))];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3))];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3))];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 1)];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 1)];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 2)];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 2)];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
          }
      }
      conv2d_3_conv2d_nchw_[(((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5408)] = conv2d_nchw_local[2];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10816)] = conv2d_nchw_local[4];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16224)] = conv2d_nchw_local[6];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21632)] = conv2d_nchw_local[8];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27040)] = conv2d_nchw_local[10];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32448)] = conv2d_nchw_local[12];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37856)] = conv2d_nchw_local[14];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 26)] = conv2d_nchw_local[1];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5434)] = conv2d_nchw_local[3];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10842)] = conv2d_nchw_local[5];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16250)] = conv2d_nchw_local[7];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21658)] = conv2d_nchw_local[9];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27066)] = conv2d_nchw_local[11];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32474)] = conv2d_nchw_local[13];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37882)] = conv2d_nchw_local[15];
    }
  }
}

// TODO:

extern "C" __global__ __launch_bounds__(104) void bgemm_2_conv2d_3_fused_bfuse(float *__restrict bgemm_2_A_, float *__restrict bgemm_2_B_, float *__restrict bgemm_2_T_batch_matmul_NT_, float *__restrict conv2d_3_conv2d_nchw_, float *__restrict conv2d_3_data_, float *__restrict conv2d_3_kernel_)
{
  uint streamingMultiprocessorId;
  asm("mov.u32 %0, %smid;" : "=r"(streamingMultiprocessorId));
  printf("Block: %d | SM: %d - Here!\n", blockIdx.x, streamingMultiprocessorId);

  typedef struct bgemm_2 {
    float A_shared[512];
    float B_shared[512];
  } bgemm_2_Ty_;
  typedef struct conv2d_3 {
    float pad_temp_shared[208];
    float kernel_shared[512];
  } conv2d_3_Ty_;
  typedef union bgemm_2_conv2d_3 {
    bgemm_2_Ty_ bgemm_2;
    conv2d_3_Ty_ conv2d_3;
  } bgemm_2_conv2d_3_Ty_;

  __shared__ bgemm_2_conv2d_3_Ty_ SU_;

  // bgemm_2
  // GridSize  = 512
  // BlockSize = 64
  // per block = 7
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 84) && ((int)threadIdx.x < 64))
  {
    int blocksPerBlock  = 7;
    int startBlockBound = (int)blockIdx.x * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x + 1) * blocksPerBlock;
    if (endBlockBound >= 512) {
        endBlockBound = 512;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
      int blockIdx_  = Idx;
      int threadIdx_ = (int)threadIdx.x;

      int gridDim_x_  = 4;
      int gridDim_y_  = 4;
      int gridDim_z_  = 32;
      int blockDim_x_ = 8;
      int blockDim_y_ = 8;
      int blockDim_z_ = 1;

      int blockIdx_x_  = blockIdx_ % gridDim_x_;
      int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
      int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
      int threadIdx_x_ = threadIdx_ % blockDim_x_;
      int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
      int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

      float T_batch_matmul_NT_local[64];
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 10; ++k_outer) {
          __syncthreads();
          SU.bgemm_2.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_2_A_[(((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 80)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 160)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 240)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 400)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 480)];
          SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_2_A_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_y_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 560)];
          SU.bgemm_2.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_2_B_[(((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 80)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 160)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 240)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 400)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 480)];
          SU.bgemm_2.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_2_B_[((((((((int)blockIdx_z_) * 20480) + (((int)blockIdx_x_) * 5120)) + (((int)threadIdx_y_) * 640)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 560)];
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = SU.bgemm_2.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = SU.bgemm_2.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = SU.bgemm_2.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = SU.bgemm_2.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_2_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_2_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 65536) + (((int)blockIdx_y_) * 16384)) + (((int)threadIdx_y_) * 2048)) + (i_inner_inner * 256)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
    }
  }
  // conv2d_3
  // GridSize  = 416
  // BlockSize = 104
  // per block = 5
  else if (((int)blockIdx.x >= 84 && (int)blockIdx.x < 168) && ((int)threadIdx.x < 104))
  {
    int blocksPerBlock  = 5;
    int startBlockBound = ((int)blockIdx.x - 84) * blocksPerBlock;
    int endBlockBound   = ((int)blockIdx.x - 84 + 1) * blocksPerBlock;
    if (endBlockBound >= 416) {
        endBlockBound = 416;
    }
    for (int Idx = startBlockBound; Idx < endBlockBound; ++Idx) {
      int blockIdx_  = Idx;
      int threadIdx_ = (int)threadIdx.x;

      int gridDim_x_  = 2;
      int gridDim_y_  = 13;
      int gridDim_z_  = 16;
      int blockDim_x_ = 13;
      int blockDim_y_ = 1;
      int blockDim_z_ = 8;

      int blockIdx_x_  = blockIdx_ % gridDim_x_;
      int blockIdx_y_  = blockIdx_ / gridDim_x_ % gridDim_y_;
      int blockIdx_z_  = blockIdx_ / (gridDim_x_ * gridDim_y_);
      int threadIdx_x_ = threadIdx_ % blockDim_x_;
      int threadIdx_y_ = threadIdx_ / blockDim_x_ % blockDim_y_;
      int threadIdx_z_ = threadIdx_ / (blockDim_x_ * blockDim_y_);

      float conv2d_nchw_local[16];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
          for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13))];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13))];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3))];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3))];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3))];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 1)];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 1)];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 2)];
              SU.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 2)];
              SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
          }
      }
      conv2d_3_conv2d_nchw_[(((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5408)] = conv2d_nchw_local[2];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10816)] = conv2d_nchw_local[4];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16224)] = conv2d_nchw_local[6];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21632)] = conv2d_nchw_local[8];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27040)] = conv2d_nchw_local[10];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32448)] = conv2d_nchw_local[12];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37856)] = conv2d_nchw_local[14];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 26)] = conv2d_nchw_local[1];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5434)] = conv2d_nchw_local[3];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10842)] = conv2d_nchw_local[5];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16250)] = conv2d_nchw_local[7];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21658)] = conv2d_nchw_local[9];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27066)] = conv2d_nchw_local[11];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32474)] = conv2d_nchw_local[13];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37882)] = conv2d_nchw_local[15];
    }
  }
}

// TODO: