
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(64) void bgemm_0_conv2d_0_fused_bfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_0_conv2d_nchw_, float *__restrict conv2d_0_data_, float *__restrict conv2d_0_kernel_)
{
  /*
   * KernelID_ means...
   * 0: bgemm_0
   * 1: conv2d_0
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x < 327680) && ((int)blockIdx.x % 5 >= 0) && ((int)blockIdx.x % 5 < 1))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 5) * 1 + ((int)blockIdx.x % 5 - 0);
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x < 327680) && ((int)blockIdx.x % 5 >= 1) && ((int)blockIdx.x % 5 < 5))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 5) * 4 + ((int)blockIdx.x % 5 - 1);
    KernelID_  = 1;
    gridDim_x_ = 55;
    gridDim_y_ = 11;
    gridDim_z_ = 3072;
    blockDim_x_ = 1;
    blockDim_y_ = 1;
    blockDim_z_ = 16;
  }
  else if ((int)blockIdx.x >= 327680 && (int)blockIdx.x < 327680)
  {
    NewBlockIdx_ = (int)blockIdx.x - 327680 + 65536;
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 327680 && (int)blockIdx.x < 1924096)
  {
    NewBlockIdx_ = (int)blockIdx.x - 327680 + 262144;
    KernelID_  = 1;
    gridDim_x_ = 55;
    gridDim_y_ = 11;
    gridDim_z_ = 3072;
    blockDim_x_ = 1;
    blockDim_y_ = 1;
    blockDim_z_ = 16;
  }
  blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
  blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
  blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
  threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
  threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
  threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

  typedef struct bgemm_0 {
     float A_shared[512];
     float B_shared[512];
  } bgemm_0Ty_;
  typedef struct conv2d_0 {
     float pad_temp_shared[18];
     float kernel_shared[128];
  } conv2d_0Ty_;
  typedef union ShrdUnion {
    bgemm_0Ty_ bgemm_0;
    conv2d_0Ty_ conv2d_0;
  } ShrdUnionTy_;

  __shared__ ShrdUnionTy_ SU_;

  // bgemm_0
  if ((KernelID_ == 0) && ((int)threadIdx.x < 64))
  {
      float T_batch_matmul_NT_local[64];
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          __syncthreads();
          SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          SU_.bgemm_0.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = SU_.bgemm_0.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
  }
  // conv2d_0
  else if ((KernelID_ == 1) && ((int)threadIdx.x < 16))
  {
      float conv2d_nchw_local[20];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[16] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[18] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[17] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[19] = 0.F;
      for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[(((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2))];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[(((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2))];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18))];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 9)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 576)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 585)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1152)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1161)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1728)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1737)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 1)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 1)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 10)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 577)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 586)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1153)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1162)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1729)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1738)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 2)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 2)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 2)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 11)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 578)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 587)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1154)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1163)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1730)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1739)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 112)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 112)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 3)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 12)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 579)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 588)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1155)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1164)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1731)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1740)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 113)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 113)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 4)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 13)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 580)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 589)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1156)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1165)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1732)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1741)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 114)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 114)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 5)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 14)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 581)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 590)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1157)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1166)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1733)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1742)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 224)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 224)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 6)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 15)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 582)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 591)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1158)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1167)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1734)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1743)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 225)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 225)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 7)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 16)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 583)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 592)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1159)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1168)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1735)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1744)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          __syncthreads();
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[(((int)threadIdx_z_) * 2)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + (((((int)threadIdx_z_) * 2) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + (((((int)threadIdx_z_) * 2) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 226)];
          }
          if (((int)threadIdx_z_) < 9) {
              SU_.conv2d_0.pad_temp_shared[((((int)threadIdx_z_) * 2) + 1)] = conv2d_0_data_[((((((((((int)blockIdx_z_) / 3) * 802816) + (rc_outer * 25088)) + ((((((int)threadIdx_z_) * 2) + 1) / 9) * 12544)) + (((int)blockIdx_y_) * 1120)) + ((((((int)threadIdx_z_) * 2) + 1) % 9) * 112)) + (((int)blockIdx_x_) * 2)) + 226)];
          }
          SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 8)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 8)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 1)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 17)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 2)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 584)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 3)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 593)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 4)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1160)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 5)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1169)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 6)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1736)];
          SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 8) + 7)] = conv2d_0_kernel_[(((((((int)blockIdx_z_) % 3) * 36864) + (((int)threadIdx_z_) * 2304)) + (rc_outer * 18)) + 1745)];
          __syncthreads();
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[(((int)threadIdx_z_) * 4)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 64)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[0] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[2] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[4] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[6] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 2)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[8] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 66)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 1)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 65)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_0.pad_temp_shared[9] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_0.pad_temp_shared[11] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_0.pad_temp_shared[13] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (SU_.conv2d_0.pad_temp_shared[15] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 3)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (SU_.conv2d_0.pad_temp_shared[17] * SU_.conv2d_0.kernel_shared[((((int)threadIdx_z_) * 4) + 67)]));
      }
      conv2d_0_conv2d_nchw_[((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_))] = conv2d_nchw_local[0];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96800)] = conv2d_nchw_local[10];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 55)] = conv2d_nchw_local[2];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96855)] = conv2d_nchw_local[12];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 110)] = conv2d_nchw_local[4];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96910)] = conv2d_nchw_local[14];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 165)] = conv2d_nchw_local[6];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 96965)] = conv2d_nchw_local[16];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 220)] = conv2d_nchw_local[8];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 97020)] = conv2d_nchw_local[18];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3025)] = conv2d_nchw_local[1];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99825)] = conv2d_nchw_local[11];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3080)] = conv2d_nchw_local[3];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99880)] = conv2d_nchw_local[13];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3135)] = conv2d_nchw_local[5];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99935)] = conv2d_nchw_local[15];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3190)] = conv2d_nchw_local[7];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 99990)] = conv2d_nchw_local[17];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 3245)] = conv2d_nchw_local[9];
      conv2d_0_conv2d_nchw_[(((((((int)blockIdx_z_) * 193600) + (((int)threadIdx_z_) * 6050)) + (((int)blockIdx_y_) * 275)) + ((int)blockIdx_x_)) + 100045)] = conv2d_nchw_local[19];
  }
}
