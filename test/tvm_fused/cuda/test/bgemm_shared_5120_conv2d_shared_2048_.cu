
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ void __launch_bounds__(128) bgemm_shared_5120(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[32];
  __shared__ float A_shared[4096];
  __shared__ float B_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    T_batch_matmul_NT_local[(i_c_outer_inner_init * 8)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 16)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 1)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 17)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 2)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 18)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 3)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 19)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 4)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 20)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 5)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 21)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 6)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 22)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 7)] = 0.000000e+00f;
    T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 23)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    *(float2*)(A_shared + (((int)threadIdx.x) * 2)) = *(float2*)(A + (((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 16384));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 24576));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 32768));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 40960));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 49152));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 57344));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2048)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 65536));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2304)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 73728));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2560)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 81920));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2816)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 90112));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3072)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 98304));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3328)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 106496));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3584)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 114688));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3840)) = *(float2*)(A + ((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 122880));
    B_shared[((int)threadIdx.x)] = B[((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    B_shared[(((int)threadIdx.x) + 128)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    B_shared[(((int)threadIdx.x) + 256)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    B_shared[(((int)threadIdx.x) + 384)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    B_shared[(((int)threadIdx.x) + 512)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
    B_shared[(((int)threadIdx.x) + 640)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
    B_shared[(((int)threadIdx.x) + 768)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
    B_shared[(((int)threadIdx.x) + 896)] = B[(((((((((int)blockIdx.x) >> 8) * 1048576) + ((((int)blockIdx.x) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 32; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        T_batch_matmul_NT_local[(i_c_outer_inner * 8)] = (T_batch_matmul_NT_local[(i_c_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * B_shared[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
        T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * B_shared[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_batch_matmul_NT[(((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner)] = T_batch_matmul_NT_local[((i_inner * 2) + j_inner)];
      T_batch_matmul_NT[((((((((((int)blockIdx.x) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner) + 65536)] = T_batch_matmul_NT_local[(((i_inner * 2) + j_inner) + 16)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) conv2d_shared_2048(float* __restrict__ A, float* __restrict__ B, float* __restrict__ W) {
  float B_local[64];
  __shared__ float Apad_shared[1024];
  __shared__ float W_shared[1024];
  float Apad_shared_local[8];
  float W_shared_local[8];
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
      B_local[((ff_c_init * 4) + nn_c_init)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    for (int ry = 0; ry < 3; ++ry) {
      for (int rx = 0; rx < 3; ++rx) {
        __syncthreads();
        for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
          if (((int)threadIdx.x) < 128) {
            *(float4*)(Apad_shared + ((((int)threadIdx.x) * 8) + (ax3_inner_outer * 4))) = (((((1 <= ((((int)blockIdx.x) / 112) + ry)) && (((((int)blockIdx.x) / 112) + ry) < 15)) && (1 <= (((((int)blockIdx.x) % 112) >> 3) + rx))) && ((((((int)blockIdx.x) % 112) >> 3) + rx) < 15)) ? *(float4*)(A + (((((((((ry * 917504) + ((((int)blockIdx.x) >> 3) * 65536)) + (rx * 65536)) + (rc_outer * 2048)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer * 4)) - 983040)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
          }
        }
        for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
          if (((int)threadIdx.x) < 128) {
            *(float4*)(W_shared + ((((int)threadIdx.x) * 8) + (ax3_inner_outer_1 * 4))) = *(float4*)(W + (((((((ry * 393216) + (rx * 131072)) + (rc_outer * 4096)) + ((((int)threadIdx.x) >> 4) * 512)) + (((((int)blockIdx.x) & 7) >> 1) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + (ax3_inner_outer_1 * 4)));
          }
        }
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int ax3 = 0; ax3 < 4; ++ax3) {
            Apad_shared_local[ax3] = Apad_shared[(((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3)];
            Apad_shared_local[(ax3 + 4)] = Apad_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 4)) + ax3) + 64)];
          }
          for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
            W_shared_local[ax3_1] = W_shared[(((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1)];
            W_shared_local[(ax3_1 + 4)] = W_shared[((((rc_inner * 128) + ((((int)threadIdx.x) >> 4) * 4)) + ax3_1) + 64)];
          }
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            for (int nn_c = 0; nn_c < 4; ++nn_c) {
              B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
              B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
              B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
              B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
    for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
      B[(((((((((int)blockIdx.x) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
      B[((((((((((int)blockIdx.x) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16384)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
      B[((((((((((int)blockIdx.x) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
      B[((((((((((int)blockIdx.x) >> 1) * 32768) + ((((int)threadIdx.x) >> 4) * 1024)) + (ff_inner_inner_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + nn_inner_inner_inner) + 16448)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
    }
  }
}