
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(50) default_function_kernel(float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  float normal_reduce_temp0[1];
  __shared__ float red_buf0[50];
  __shared__ float T_softmax_maxelem[1];
  float normal_reduce_temp0_1[1];
  __shared__ float red_buf0_1[50];
  __shared__ float T_softmax_expsum[1];
  normal_reduce_temp0[0] = -3.402823e+38f;
  for (int k_outer = 0; k_outer < 20; ++k_outer) {
    normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + (k_outer * 50)) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[((int)threadIdx.x)] = normal_reduce_temp0[0];
  __syncthreads();
  if (((int)threadIdx.x) < 18) {
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 32)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    float w_16_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 16)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_16_0;
    float w_8_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 8)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_8_0;
    float w_4_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 4)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_4_0;
    float w_2_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 2)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_2_0;
    float w_1_0 = max(((volatile float*)red_buf0)[((int)threadIdx.x)], ((volatile float*)red_buf0)[(((int)threadIdx.x) + 1)]);
    ((volatile float*)red_buf0)[((int)threadIdx.x)] = w_1_0;
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_maxelem[0] = ((volatile float*)red_buf0)[0];
  }
  normal_reduce_temp0_1[0] = 0.000000e+00f;
  __syncthreads();
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[((((int)blockIdx.x) * 1000) + ((int)threadIdx.x))] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 50)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 100)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 150)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 200)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 250)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 300)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 350)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 400)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 450)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 500)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 550)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 600)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 650)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 700)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 750)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 800)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 850)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 900)] - T_softmax_maxelem[0])));
  normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 950)] - T_softmax_maxelem[0])));
  __syncthreads();
  ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = normal_reduce_temp0_1[0];
  __syncthreads();
  if (((int)threadIdx.x) < 18) {
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 32)]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    float w_16_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 16)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_16_0_1;
    float w_8_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 8)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_8_0_1;
    float w_4_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 4)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_4_0_1;
    float w_2_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 2)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_2_0_1;
    float w_1_0_1 = (((volatile float*)red_buf0_1)[((int)threadIdx.x)] + ((volatile float*)red_buf0_1)[(((int)threadIdx.x) + 1)]);
    ((volatile float*)red_buf0_1)[((int)threadIdx.x)] = w_1_0_1;
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_expsum[0] = ((volatile float*)red_buf0_1)[0];
  }
  __syncthreads();
  for (int i1_outer = 0; i1_outer < 20; ++i1_outer) {
    T_softmax_norm[(((((int)blockIdx.x) * 1000) + (i1_outer * 50)) + ((int)threadIdx.x))] = (__expf((data[(((((int)blockIdx.x) * 1000) + (i1_outer * 50)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])) / T_softmax_expsum[0]);
  }
}

