
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((((int)blockIdx.x) / 729) * 200704) + (((((int)blockIdx.x) % 729) / 27) * 7168)) + (rv0 * 3584)) + ((((int)blockIdx.x) % 27) * 128)) + (rv1 * 64)) + ((int)threadIdx.x))]);
    }
  }
}

