
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) softmax_shared_2060(float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  float normal_reduce_temp0[1];
  __shared__ float red_result[1+512];//
  __shared__ float T_softmax_maxelem[1+512];//
  float normal_reduce_temp0_1[1];
  __shared__ float red_result_1[1+512];//
  __shared__ float T_softmax_expsum[1+512];//
  normal_reduce_temp0[0] = -3.402823e+38f;
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[((((int)blockIdx.x) * 1000) + ((int)threadIdx.x))]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 128)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 256)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 384)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 512)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 640)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 768)]);
  if (((int)threadIdx.x) < 104) {
    normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 896)]);
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[4];
  red_buf0_1[0] = normal_reduce_temp0[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int)threadIdx.x) % 32) == 0) {
    red_buf_staging[(((int)threadIdx.x) >> 5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) < 4) {
    red_buf0[0] = red_buf_staging[((int)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)15);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int)threadIdx.x) == 0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_maxelem[0] = ((volatile float*)red_result)[0];
  }
  normal_reduce_temp0_1[0] = 0.000000e+00f;
  __syncthreads();
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    if (((k_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
      normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[4];
  red_buf0_3[0] = normal_reduce_temp0_1[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int)threadIdx.x) % 32) == 0) {
    red_buf_staging_1[(((int)threadIdx.x) >> 5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) < 4) {
    red_buf0_2[0] = red_buf_staging_1[((int)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)15);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int)threadIdx.x) == 0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_expsum[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
    if (((i2_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
      T_softmax_norm[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] = (__expf((data[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])) / T_softmax_expsum[0]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(128) softmax_shared_2060_copy(float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  float normal_reduce_temp0[1];
  __shared__ float red_result[1+512];//
  __shared__ float T_softmax_maxelem[1+512];//
  float normal_reduce_temp0_1[1];
  __shared__ float red_result_1[1+512];//
  __shared__ float T_softmax_expsum[1+512];//
  normal_reduce_temp0[0] = -3.402823e+38f;
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[((((int)blockIdx.x) * 1000) + ((int)threadIdx.x))]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 128)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 256)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 384)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 512)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 640)]);
  normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 768)]);
  if (((int)threadIdx.x) < 104) {
    normal_reduce_temp0[0] = max(normal_reduce_temp0[0], data[(((((int)blockIdx.x) * 1000) + ((int)threadIdx.x)) + 896)]);
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[4];
  red_buf0_1[0] = normal_reduce_temp0[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int)threadIdx.x) % 32) == 0) {
    red_buf_staging[(((int)threadIdx.x) >> 5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) < 4) {
    red_buf0[0] = red_buf_staging[((int)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)15);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int)threadIdx.x) == 0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_maxelem[0] = ((volatile float*)red_result)[0];
  }
  normal_reduce_temp0_1[0] = 0.000000e+00f;
  __syncthreads();
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    if (((k_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
      normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((data[(((((int)blockIdx.x) * 1000) + (k_outer * 128)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[4];
  red_buf0_3[0] = normal_reduce_temp0_1[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int)threadIdx.x) % 32) == 0) {
    red_buf_staging_1[(((int)threadIdx.x) >> 5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) < 4) {
    red_buf0_2[0] = red_buf_staging_1[((int)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)15);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int)threadIdx.x) == 0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_softmax_expsum[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
    if (((i2_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
      T_softmax_norm[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] = (__expf((data[(((((int)blockIdx.x) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] - T_softmax_maxelem[0])) / T_softmax_expsum[0]);
    }
  }
}