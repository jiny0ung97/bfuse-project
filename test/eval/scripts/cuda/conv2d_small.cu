
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ W) {
  float B_local[64];
  __shared__ float Apad_shared[1024];
  __shared__ float W_shared[512];
  float Apad_shared_local[8];
  float W_shared_local[8];
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
      B_local[((ff_c_init * 4) + nn_c_init)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
      if (((int)threadIdx.y) < 8) {
        *(float4*)(Apad_shared + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer * 4))) = (((((58 <= ((int)blockIdx.z)) && (((int)blockIdx.z) < 3306)) && (1 <= (((int)blockIdx.z) % 58))) && ((((int)blockIdx.z) % 58) < 57)) ? *(float4*)(A + ((((((((((int)blockIdx.z) / 58) * 458752) + ((((int)blockIdx.z) % 58) * 8192)) + (rc_outer * 1024)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer * 4)) - 466944)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
      }
    }
    if (((int)threadIdx.y) < 8) {
      *(float4*)(W_shared + ((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 4))) = *(float4*)(W + (((rc_outer * 512) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ax3 = 0; ax3 < 4; ++ax3) {
        Apad_shared_local[ax3] = Apad_shared[(((rc_inner * 128) + (((int)threadIdx.x) * 4)) + ax3)];
        Apad_shared_local[(ax3 + 4)] = Apad_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 4)) + ax3) + 64)];
      }
      for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
        W_shared_local[ax3_1] = W_shared[(((rc_inner * 64) + (((int)threadIdx.y) * 4)) + ax3_1)];
      }
      for (int ff_c = 0; ff_c < 4; ++ff_c) {
        for (int nn_c = 0; nn_c < 4; ++nn_c) {
          B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
          B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
        }
      }
    }
  }
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
    for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
      B[(((((((int)blockIdx.z) * 8192) + (((int)threadIdx.y) * 512)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
      B[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.y) * 512)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
    }
  }
}

