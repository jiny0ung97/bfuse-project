
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) conv2d_large(float* __restrict__ A, float* __restrict__ B, float* __restrict__ W) {
  float B_local[64];
  __shared__ float Apad_shared[512];
  __shared__ float W_shared[512];
  float Apad_shared_local[8];
  float W_shared_local[8];
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
      B_local[((ff_c_init * 4) + nn_c_init)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.000000e+00f;
      B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    for (int ry = 0; ry < 3; ++ry) {
      for (int rx = 0; rx < 3; ++rx) {
        __syncthreads();
        for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
          *(float4*)(Apad_shared + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer * 4))) = (((((1 <= ((((int)blockIdx.z) / 56) + ry)) && (((((int)blockIdx.z) / 56) + ry) < 57)) && (1 <= (rx + (((int)blockIdx.z) % 56)))) && ((rx + (((int)blockIdx.z) % 56)) < 57)) ? *(float4*)(A + ((((((((ry * 229376) + (((int)blockIdx.z) * 4096)) + (rx * 4096)) + (rc_outer * 512)) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer * 4)) - 233472)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
        }
        for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
          *(float4*)(W_shared + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer_1 * 4))) = *(float4*)(W + ((((((ry * 12288) + (rx * 4096)) + (rc_outer * 512)) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer_1 * 4)));
        }
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int ax3 = 0; ax3 < 4; ++ax3) {
            Apad_shared_local[ax3] = Apad_shared[(((rc_inner * 64) + (((int)threadIdx.x) * 4)) + ax3)];
            Apad_shared_local[(ax3 + 4)] = Apad_shared[((((rc_inner * 64) + (((int)threadIdx.x) * 4)) + ax3) + 32)];
          }
          for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
            W_shared_local[ax3_1] = W_shared[(((rc_inner * 64) + (((int)threadIdx.y) * 4)) + ax3_1)];
            W_shared_local[(ax3_1 + 4)] = W_shared[((((rc_inner * 64) + (((int)threadIdx.y) * 4)) + ax3_1) + 32)];
          }
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            for (int nn_c = 0; nn_c < 4; ++nn_c) {
              B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
              B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
              B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
              B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
    for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
      B[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.y) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
      B[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.y) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 2048)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
      B[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.y) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 32)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
      B[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.y) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 2080)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
    }
  }
}

