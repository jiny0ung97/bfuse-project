
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(128) void bgemm_shared_8192_bgemm_shared_8192_copy_fused_kernel_bfuse_idx_0(float *__restrict bgemm_shared_8192_A_, float *__restrict bgemm_shared_8192_B_, float *__restrict bgemm_shared_8192_T_batch_matmul_NT_, float *__restrict bgemm_shared_8192_copy_A_, float *__restrict bgemm_shared_8192_copy_B_, float *__restrict bgemm_shared_8192_copy_T_batch_matmul_NT_)
{
  /*
   * KernelID_ means...
   * 0: bgemm_shared_8192
   * 1: bgemm_shared_8192_copy
   */
  int gridDim_x_;
  int blockIdx_x_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 65520) && ((((int)blockIdx.x - 0) / 84) % 2 == 0))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 32768;
  }
  else if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 65520) && ((((int)blockIdx.x - 0) / 84) % 2 == 1))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 1;
    gridDim_x_ = 32768;
  }
  else if ((int)blockIdx.x >= 65520 && (int)blockIdx.x < 65528)
  {
    NewBlockIdx_ = (int)blockIdx.x - 32760;
    KernelID_  = 0;
    gridDim_x_ = 32768;
  }
  else if ((int)blockIdx.x >= 65528 && (int)blockIdx.x < 65536)
  {
    NewBlockIdx_ = (int)blockIdx.x - 32768;
    KernelID_  = 1;
    gridDim_x_ = 32768;
  }
  blockIdx_x_ = NewBlockIdx_;

  static float union_shared_0_[4096] __attribute__((shared));
  static float union_shared_1_[4096] __attribute__((shared));


  // bgemm_shared_8192
  if ((KernelID_ == 0) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 128)))
  {
      float T_batch_matmul_NT_local[32];
      for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
          T_batch_matmul_NT_local[(i_c_outer_inner_init * 8)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 16)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 1)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 17)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 2)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 18)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 3)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 19)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 4)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 20)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 5)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 21)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 6)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 22)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 7)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 23)] = 0.F;
      }
      for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
          asm ("bar.sync 0, 128;");
          *(float2 *)(union_shared_0_ + (((int)threadIdx.x) * 2)) = *(float2 *)(bgemm_shared_8192_A_ + (((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 256)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 512)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 16384));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 768)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 24576));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1024)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 32768));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1280)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 40960));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1536)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 49152));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1792)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 57344));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2048)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 65536));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2304)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 73728));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2560)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 81920));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2816)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 90112));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3072)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 98304));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3328)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 106496));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3584)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 114688));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3840)) = *(float2 *)(bgemm_shared_8192_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 122880));
          union_shared_1_[((int)threadIdx.x)] = bgemm_shared_8192_B_[((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
          union_shared_1_[(((int)threadIdx.x) + 128)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
          union_shared_1_[(((int)threadIdx.x) + 256)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
          union_shared_1_[(((int)threadIdx.x) + 384)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
          union_shared_1_[(((int)threadIdx.x) + 512)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
          union_shared_1_[(((int)threadIdx.x) + 640)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
          union_shared_1_[(((int)threadIdx.x) + 768)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
          union_shared_1_[(((int)threadIdx.x) + 896)] = bgemm_shared_8192_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
          asm ("bar.sync 0, 128;");
          for (int k_outer_inner = 0; k_outer_inner < 32; ++k_outer_inner) {
              for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
                  T_batch_matmul_NT_local[(i_c_outer_inner * 8)] = (T_batch_matmul_NT_local[(i_c_outer_inner * 8)] + (union_shared_0_[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] + (union_shared_0_[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
              }
          }
      }
      for (int i_inner = 0; i_inner < 8; ++i_inner) {
          for (int j_inner = 0; j_inner < 2; ++j_inner) {
              bgemm_shared_8192_T_batch_matmul_NT_[(((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx_x_) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner)] = T_batch_matmul_NT_local[((i_inner * 2) + j_inner)];
              bgemm_shared_8192_T_batch_matmul_NT_[((((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx_x_) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner) + 65536)] = T_batch_matmul_NT_local[(((i_inner * 2) + j_inner) + 16)];
          }
      }
  }
  // bgemm_shared_8192_copy
  else if ((KernelID_ == 1) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 128)))
  {
      float T_batch_matmul_NT_local[32];
      for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
          T_batch_matmul_NT_local[(i_c_outer_inner_init * 8)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 16)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 1)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 17)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 2)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 18)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 3)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 19)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 4)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 20)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 5)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 21)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 6)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 22)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 7)] = 0.F;
          T_batch_matmul_NT_local[((i_c_outer_inner_init * 8) + 23)] = 0.F;
      }
      for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
          asm ("bar.sync 0, 128;");
          *(float2 *)(union_shared_0_ + (((int)threadIdx.x) * 2)) = *(float2 *)(bgemm_shared_8192_copy_A_ + (((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 256)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 512)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 16384));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 768)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 24576));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1024)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 32768));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1280)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 40960));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1536)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 49152));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 1792)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 57344));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2048)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 65536));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2304)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 73728));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2560)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 81920));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 2816)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 90112));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3072)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 98304));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3328)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 106496));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3584)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 114688));
          *(float2 *)(union_shared_0_ + ((((int)threadIdx.x) * 2) + 3840)) = *(float2 *)(bgemm_shared_8192_copy_A_ + ((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 122880));
          union_shared_1_[((int)threadIdx.x)] = bgemm_shared_8192_copy_B_[((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
          union_shared_1_[(((int)threadIdx.x) + 128)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
          union_shared_1_[(((int)threadIdx.x) + 256)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
          union_shared_1_[(((int)threadIdx.x) + 384)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
          union_shared_1_[(((int)threadIdx.x) + 512)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
          union_shared_1_[(((int)threadIdx.x) + 640)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
          union_shared_1_[(((int)threadIdx.x) + 768)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
          union_shared_1_[(((int)threadIdx.x) + 896)] = bgemm_shared_8192_copy_B_[(((((((((int)blockIdx_x_) >> 8) * 1048576) + ((((int)blockIdx_x_) & 31) * 32768)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
          asm ("bar.sync 0, 128;");
          for (int k_outer_inner = 0; k_outer_inner < 32; ++k_outer_inner) {
              for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
                  T_batch_matmul_NT_local[(i_c_outer_inner * 8)] = (T_batch_matmul_NT_local[(i_c_outer_inner * 8)] + (union_shared_0_[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 16)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 1)] + (union_shared_0_[((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 17)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2048)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 2)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 18)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 3)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 32)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 19)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2080)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 4)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 20)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 5)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 64)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 21)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2112)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 6)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 22)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * union_shared_1_[(((((int)threadIdx.x) & 15) * 64) + k_outer_inner)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 7)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 96)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
                  T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] = (T_batch_matmul_NT_local[((i_c_outer_inner * 8) + 23)] + (union_shared_0_[(((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + k_outer_inner) + 2144)] * union_shared_1_[((((((int)threadIdx.x) & 15) * 64) + k_outer_inner) + 32)]));
              }
          }
      }
      for (int i_inner = 0; i_inner < 8; ++i_inner) {
          for (int j_inner = 0; j_inner < 2; ++j_inner) {
              bgemm_shared_8192_copy_T_batch_matmul_NT_[(((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx_x_) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner)] = T_batch_matmul_NT_local[((i_inner * 2) + j_inner)];
              bgemm_shared_8192_copy_T_batch_matmul_NT_[((((((((((int)blockIdx_x_) >> 5) * 131072) + ((((int)threadIdx.x) >> 4) * 8192)) + (i_inner * 1024)) + ((((int)blockIdx_x_) & 31) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_inner) + 65536)] = T_batch_matmul_NT_local[(((i_inner * 2) + j_inner) + 16)];
          }
      }
  }
}
