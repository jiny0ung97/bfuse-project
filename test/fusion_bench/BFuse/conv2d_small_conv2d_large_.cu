
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(256) void conv2d_small_conv2d_large_fused_kernel_bfuse_idx_0(float *__restrict conv2d_small_A_, float *__restrict conv2d_small_B_, float *__restrict conv2d_small_W_, float *__restrict conv2d_large_A_, float *__restrict conv2d_large_B_, float *__restrict conv2d_large_W_)
{
  /*
   * KernelID_ means...
   * 0: conv2d_small
   * 1: conv2d_large
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 6216) && ((((int)blockIdx.x - 0) / 84) % 2 == 0))
  {
    NewBlockIdx_ = 0 + (((int)blockIdx.x - 0) / 168) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 1;
    gridDim_x_ = 1;
    gridDim_y_ = 1;
    gridDim_z_ = 3136;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 6216) && ((((int)blockIdx.x - 0) / 84) % 2 == 1))
  {
    NewBlockIdx_ = 0 + (((int)blockIdx.x - 0) / 168) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 1;
    gridDim_y_ = 1;
    gridDim_z_ = 3364;
    blockDim_x_ = 16;
    blockDim_y_ = 16;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x >= 6216 && (int)blockIdx.x < 6468) && ((((int)blockIdx.x - 6216) / 84) % 1 == 0))
  {
    NewBlockIdx_ = 3108 + (((int)blockIdx.x - 6216) / 84) * 84 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 1;
    gridDim_y_ = 1;
    gridDim_z_ = 3364;
    blockDim_x_ = 16;
    blockDim_y_ = 16;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 6468 && (int)blockIdx.x < 6496)
  {
    NewBlockIdx_ = (int)blockIdx.x - 3360;
    KernelID_  = 1;
    gridDim_x_ = 1;
    gridDim_y_ = 1;
    gridDim_z_ = 3136;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 6496 && (int)blockIdx.x < 6500)
  {
    NewBlockIdx_ = (int)blockIdx.x - 3136;
    KernelID_  = 0;
    gridDim_x_ = 1;
    gridDim_y_ = 1;
    gridDim_z_ = 3364;
    blockDim_x_ = 16;
    blockDim_y_ = 16;
    blockDim_z_ = 1;
  }
  blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
  blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
  blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
  threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
  threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
  threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

//   static float union_shared_0_[1024] __attribute__((shared));
//   static float union_shared_1_[512] __attribute__((shared));
    static float union_shared_0_[2048] __attribute__((shared));
  static float union_shared_1_[2048] __attribute__((shared));


  // conv2d_small
  if ((KernelID_ == 0) && ((int)threadIdx.x < 256))
  {
      float B_local[64];
      float Apad_shared_local[8];
      float W_shared_local[8];
      for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
          for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
              B_local[((ff_c_init * 4) + nn_c_init)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.F;
          }
      }
      for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
          __syncthreads();
          for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
              if (((int)threadIdx_y_) < 8) {
                  *(float4 *)(union_shared_0_ + (((((int)threadIdx_y_) * 128) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer * 4))) = (((((58 <= ((int)blockIdx_z_)) && (((int)blockIdx_z_) < 3306)) && (1 <= (((int)blockIdx_z_) % 58))) && ((((int)blockIdx_z_) % 58) < 57)) ? *(float4 *)(conv2d_small_A_ + ((((((((((int)blockIdx_z_) / 58) * 458752) + ((((int)blockIdx_z_) % 58) * 8192)) + (rc_outer * 1024)) + (((int)threadIdx_y_) * 128)) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer * 4)) - 466944)) : make_float4(0.F, 0.F, 0.F, 0.F));
              }
          }
          if (((int)threadIdx_y_) < 8) {
              *(float4 *)(union_shared_1_ + ((((int)threadIdx_y_) * 64) + (((int)threadIdx_x_) * 4))) = *(float4 *)(conv2d_small_W_ + (((rc_outer * 512) + (((int)threadIdx_y_) * 64)) + (((int)threadIdx_x_) * 4)));
          }
          __syncthreads();
          for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
              for (int ax3 = 0; ax3 < 4; ++ax3) {
                  Apad_shared_local[ax3] = union_shared_0_[(((rc_inner * 128) + (((int)threadIdx_x_) * 4)) + ax3)];
                  Apad_shared_local[(ax3 + 4)] = union_shared_0_[((((rc_inner * 128) + (((int)threadIdx_x_) * 4)) + ax3) + 64)];
              }
              for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
                  W_shared_local[ax3_1] = union_shared_1_[(((rc_inner * 64) + (((int)threadIdx_y_) * 4)) + ax3_1)];
              }
              for (int ff_c = 0; ff_c < 4; ++ff_c) {
                  for (int nn_c = 0; nn_c < 4; ++nn_c) {
                      B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
                      B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
                  }
              }
          }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
          for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
              conv2d_small_B_[(((((((int)blockIdx_z_) * 8192) + (((int)threadIdx_y_) * 512)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
              conv2d_small_B_[((((((((int)blockIdx_z_) * 8192) + (((int)threadIdx_y_) * 512)) + (ff_inner_inner_inner * 128)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner) + 64)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
          }
      }
  }
  // conv2d_large
  else if ((KernelID_ == 1) && ((int)threadIdx.x < 64))
  {
      float B_local[64];
      float Apad_shared_local[8];
      float W_shared_local[8];
      for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
          for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
              B_local[((ff_c_init * 4) + nn_c_init)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.F;
              B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.F;
          }
      }
      for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
          for (int ry = 0; ry < 3; ++ry) {
              for (int rx = 0; rx < 3; ++rx) {
                  __syncthreads();
                  for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
                      *(float4 *)(union_shared_0_ + (((((int)threadIdx_y_) * 64) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer * 4))) = (((((1 <= ((((int)blockIdx_z_) / 56) + ry)) && (((((int)blockIdx_z_) / 56) + ry) < 57)) && (1 <= (rx + (((int)blockIdx_z_) % 56)))) && ((rx + (((int)blockIdx_z_) % 56)) < 57)) ? *(float4 *)(conv2d_large_A_ + ((((((((ry * 229376) + (((int)blockIdx_z_) * 4096)) + (rx * 4096)) + (rc_outer * 512)) + (((int)threadIdx_y_) * 64)) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer * 4)) - 233472)) : make_float4(0.F, 0.F, 0.F, 0.F));
                  }
                  for (int ax3_inner_outer_1 = 0; ax3_inner_outer_1 < 2; ++ax3_inner_outer_1) {
                      *(float4 *)(union_shared_1_ + (((((int)threadIdx_y_) * 64) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer_1 * 4))) = *(float4 *)(conv2d_large_W_ + ((((((ry * 12288) + (rx * 4096)) + (rc_outer * 512)) + (((int)threadIdx_y_) * 64)) + (((int)threadIdx_x_) * 8)) + (ax3_inner_outer_1 * 4)));
                  }
                  __syncthreads();
                  for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
                      for (int ax3 = 0; ax3 < 4; ++ax3) {
                          Apad_shared_local[ax3] = union_shared_0_[(((rc_inner * 64) + (((int)threadIdx_x_) * 4)) + ax3)];
                          Apad_shared_local[(ax3 + 4)] = union_shared_0_[((((rc_inner * 64) + (((int)threadIdx_x_) * 4)) + ax3) + 32)];
                      }
                      for (int ax3_1 = 0; ax3_1 < 4; ++ax3_1) {
                          W_shared_local[ax3_1] = union_shared_1_[(((rc_inner * 64) + (((int)threadIdx_y_) * 4)) + ax3_1)];
                          W_shared_local[(ax3_1 + 4)] = union_shared_1_[((((rc_inner * 64) + (((int)threadIdx_y_) * 4)) + ax3_1) + 32)];
                      }
                      for (int ff_c = 0; ff_c < 4; ++ff_c) {
                          for (int nn_c = 0; nn_c < 4; ++nn_c) {
                              B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
                              B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
                              B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
                          }
                      }
                  }
              }
          }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
          for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
              conv2d_large_B_[(((((((int)blockIdx_z_) * 4096) + (((int)threadIdx_y_) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
              conv2d_large_B_[((((((((int)blockIdx_z_) * 4096) + (((int)threadIdx_y_) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner) + 2048)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
              conv2d_large_B_[((((((((int)blockIdx_z_) * 4096) + (((int)threadIdx_y_) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner) + 32)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
              conv2d_large_B_[((((((((int)blockIdx_z_) * 4096) + (((int)threadIdx_y_) * 256)) + (ff_inner_inner_inner * 64)) + (((int)threadIdx_x_) * 4)) + nn_inner_inner_inner) + 2080)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
          }
      }
  }
}
