
#include <hip/hip_runtime.h>


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

extern "C" __global__ __launch_bounds__(128) void softmax_shared_1036_softmax_shared_1036_copy_fused_kernel_bfuse_idx_0(float *__restrict softmax_shared_1036_T_softmax_norm_, float *__restrict softmax_shared_1036_data_, float *__restrict softmax_shared_1036_copy_T_softmax_norm_, float *__restrict softmax_shared_1036_copy_data_)
{
  /*
   * KernelID_ means...
   * 0: softmax_shared_1036
   * 1: softmax_shared_1036_copy
   */
  int gridDim_x_;
  int blockIdx_x_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 168) && ((((int)blockIdx.x - 0) / 84) % 2 == 0))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 0;
    gridDim_x_ = 128;
  }
  else if (((int)blockIdx.x >= 0 && (int)blockIdx.x < 168) && ((((int)blockIdx.x - 0) / 84) % 2 == 1))
  {
    NewBlockIdx_ = 0 + ((int)blockIdx.x - (int)blockIdx.x % 168) / 2 + (int)blockIdx.x % 84;
    KernelID_  = 1;
    gridDim_x_ = 128;
  }
  else if ((int)blockIdx.x >= 168 && (int)blockIdx.x < 212)
  {
    NewBlockIdx_ = (int)blockIdx.x - 84;
    KernelID_  = 0;
    gridDim_x_ = 128;
  }
  else if ((int)blockIdx.x >= 212 && (int)blockIdx.x < 256)
  {
    NewBlockIdx_ = (int)blockIdx.x - 128;
    KernelID_  = 1;
    gridDim_x_ = 128;
  }
  blockIdx_x_ = NewBlockIdx_;

  static float union_shared_0_[257] __attribute__((shared));
  static float union_shared_1_[257] __attribute__((shared));
  static float union_shared_2_[257] __attribute__((shared));
  static float union_shared_3_[257] __attribute__((shared));
  static float union_shared_4_[4] __attribute__((shared));
  static float union_shared_5_[4] __attribute__((shared));


  // softmax_shared_1036
  if ((KernelID_ == 0) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 128)))
  {
      float normal_reduce_temp0[1];
      float normal_reduce_temp0_1[1];
      normal_reduce_temp0[0] = -3.40282306E+38F;
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x))]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 128)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 256)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 384)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 512)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 640)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 768)]);
      if (((int)threadIdx.x) < 104) {
          normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 896)]);
      }
      float red_buf0[1];
      unsigned int mask[1];
      float t0[1];
      float red_buf0_1[1];
      unsigned int mask_1[1];
      float t0_1[1];
      red_buf0_1[0] = normal_reduce_temp0[0];
      mask_1[0] = __activemask();
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      if ((((int)threadIdx.x) % 32) == 0) {
          union_shared_4_[(((int)threadIdx.x) >> 5)] = red_buf0_1[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) < 4) {
          red_buf0[0] = union_shared_4_[((int)threadIdx.x)];
      }
      mask[0] = (__activemask() & (unsigned int)15);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
      red_buf0[0] = max(red_buf0[0], t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
      red_buf0[0] = max(red_buf0[0], t0[0]);
      if (((int)threadIdx.x) == 0) {
          ((volatile float *)union_shared_0_)[0] = red_buf0[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) == 0) {
          union_shared_1_[0] = ((volatile float *)union_shared_0_)[0];
      }
      normal_reduce_temp0_1[0] = 0.F;
      asm ("bar.sync 0, 128;");
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          if (((k_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
              normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + (k_outer * 128)) + ((int)threadIdx.x))] - union_shared_1_[0])));
          }
      }
      float red_buf0_2[1];
      unsigned int mask_2[1];
      float t0_2[1];
      float red_buf0_3[1];
      unsigned int mask_3[1];
      float t0_3[1];
      red_buf0_3[0] = normal_reduce_temp0_1[0];
      mask_3[0] = __activemask();
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      if ((((int)threadIdx.x) % 32) == 0) {
          union_shared_5_[(((int)threadIdx.x) >> 5)] = red_buf0_3[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) < 4) {
          red_buf0_2[0] = union_shared_5_[((int)threadIdx.x)];
      }
      mask_2[0] = (__activemask() & (unsigned int)15);
      t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
      red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
      t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
      red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
      if (((int)threadIdx.x) == 0) {
          ((volatile float *)union_shared_2_)[0] = red_buf0_2[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) == 0) {
          union_shared_3_[0] = ((volatile float *)union_shared_2_)[0];
      }
      asm ("bar.sync 0, 128;");
      for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
          if (((i2_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
              softmax_shared_1036_T_softmax_norm_[(((((int)blockIdx_x_) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] = (__expf((softmax_shared_1036_data_[(((((int)blockIdx_x_) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] - union_shared_1_[0])) / union_shared_3_[0]);
          }
      }
  }
  // softmax_shared_1036_copy
  else if ((KernelID_ == 1) && (((int)threadIdx.x >= 0 && (int)threadIdx.x < 128)))
  {
      float normal_reduce_temp0[1];
      float normal_reduce_temp0_1[1];
      normal_reduce_temp0[0] = -3.40282306E+38F;
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x))]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 128)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 256)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 384)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 512)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 640)]);
      normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 768)]);
      if (((int)threadIdx.x) < 104) {
          normal_reduce_temp0[0] = max(normal_reduce_temp0[0], softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + ((int)threadIdx.x)) + 896)]);
      }
      float red_buf0[1];
      unsigned int mask[1];
      float t0[1];
      float red_buf0_1[1];
      unsigned int mask_1[1];
      float t0_1[1];
      red_buf0_1[0] = normal_reduce_temp0[0];
      mask_1[0] = __activemask();
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
      red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
      if ((((int)threadIdx.x) % 32) == 0) {
          union_shared_4_[(((int)threadIdx.x) >> 5)] = red_buf0_1[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) < 4) {
          red_buf0[0] = union_shared_4_[((int)threadIdx.x)];
      }
      mask[0] = (__activemask() & (unsigned int)15);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
      red_buf0[0] = max(red_buf0[0], t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
      red_buf0[0] = max(red_buf0[0], t0[0]);
      if (((int)threadIdx.x) == 0) {
          ((volatile float *)union_shared_0_)[0] = red_buf0[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) == 0) {
          union_shared_1_[0] = ((volatile float *)union_shared_0_)[0];
      }
      normal_reduce_temp0_1[0] = 0.F;
      asm ("bar.sync 0, 128;");
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          if (((k_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
              normal_reduce_temp0_1[0] = (normal_reduce_temp0_1[0] + __expf((softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + (k_outer * 128)) + ((int)threadIdx.x))] - union_shared_1_[0])));
          }
      }
      float red_buf0_2[1];
      unsigned int mask_2[1];
      float t0_2[1];
      float red_buf0_3[1];
      unsigned int mask_3[1];
      float t0_3[1];
      red_buf0_3[0] = normal_reduce_temp0_1[0];
      mask_3[0] = __activemask();
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
      red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
      if ((((int)threadIdx.x) % 32) == 0) {
          union_shared_5_[(((int)threadIdx.x) >> 5)] = red_buf0_3[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) < 4) {
          red_buf0_2[0] = union_shared_5_[((int)threadIdx.x)];
      }
      mask_2[0] = (__activemask() & (unsigned int)15);
      t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
      red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
      t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
      red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
      if (((int)threadIdx.x) == 0) {
          ((volatile float *)union_shared_2_)[0] = red_buf0_2[0];
      }
      asm ("bar.sync 0, 128;");
      if (((int)threadIdx.x) == 0) {
          union_shared_3_[0] = ((volatile float *)union_shared_2_)[0];
      }
      asm ("bar.sync 0, 128;");
      for (int i2_outer = 0; i2_outer < 8; ++i2_outer) {
          if (((i2_outer * 16) + (((int)threadIdx.x) >> 3)) < 125) {
              softmax_shared_1036_copy_T_softmax_norm_[(((((int)blockIdx_x_) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] = (__expf((softmax_shared_1036_copy_data_[(((((int)blockIdx_x_) * 1000) + (i2_outer * 128)) + ((int)threadIdx.x))] - union_shared_1_[0])) / union_shared_3_[0]);
          }
      }
  }
}
