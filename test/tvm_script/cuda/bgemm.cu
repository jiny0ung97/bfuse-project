
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[4];
  __shared__ float A_shared[32];
  __shared__ float B_shared[1024];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  T_batch_matmul_NT_local[2] = 0.000000e+00f;
  T_batch_matmul_NT_local[1] = 0.000000e+00f;
  T_batch_matmul_NT_local[3] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 16) {
      *(float2*)(A_shared + (((int)threadIdx.x) * 2)) = *(float2*)(A + ((((((int)blockIdx.x) >> 5) * 512) + (k_outer_outer * 32)) + (((int)threadIdx.x) * 2)));
    }
    B_shared[((int)threadIdx.x)] = B[((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    B_shared[(((int)threadIdx.x) + 128)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    B_shared[(((int)threadIdx.x) + 256)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    B_shared[(((int)threadIdx.x) + 384)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    B_shared[(((int)threadIdx.x) + 512)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    B_shared[(((int)threadIdx.x) + 640)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240)];
    B_shared[(((int)threadIdx.x) + 768)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    B_shared[(((int)threadIdx.x) + 896)] = B[(((((((int)blockIdx.x) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 32; ++k_outer_inner) {
      if (((int)threadIdx.x) < 16) {
        T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (A_shared[k_outer_inner] * B_shared[((((int)threadIdx.x) * 64) + k_outer_inner)]));
      }
      if (((int)threadIdx.x) < 16) {
        T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (A_shared[k_outer_inner] * B_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 32)]));
      }
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    if (((int)threadIdx.x) < 16) {
      T_batch_matmul_NT[(((((int)blockIdx.x) * 32) + (((int)threadIdx.x) * 2)) + j_inner)] = T_batch_matmul_NT_local[j_inner];
    }
  }
}

