
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) bgemm_0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT);
extern "C" __global__ void __launch_bounds__(64) bgemm_0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[64];
  __shared__ float A_shared[512];
  __shared__ float B_shared[512];
  float A_shared_local[8];
  float B_shared_local[8];
  for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
      T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    __syncthreads();
    A_shared[((((int)threadIdx.y) * 64) + ((int)threadIdx.x))] = A[(((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 8)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 64)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 16)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 128)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 24)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 192)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 32)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 256)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 40)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 320)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 48)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 384)];
    A_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 56)] = A[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.y) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 448)];
    B_shared[((((int)threadIdx.y) * 64) + ((int)threadIdx.x))] = B[(((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x))];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 8)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 64)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 16)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 128)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 24)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 192)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 32)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 256)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 40)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 320)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 48)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 384)];
    B_shared[(((((int)threadIdx.y) * 64) + ((int)threadIdx.x)) + 56)] = B[((((((((int)blockIdx.z) * 32768) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.y) * 512)) + (k_outer * 8)) + ((int)threadIdx.x)) + 448)];
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      A_shared_local[0] = A_shared[((((int)threadIdx.y) * 64) + k_inner)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 8)];
      A_shared_local[2] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 16)];
      A_shared_local[3] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 24)];
      A_shared_local[4] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 32)];
      A_shared_local[5] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 40)];
      A_shared_local[6] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 48)];
      A_shared_local[7] = A_shared[(((((int)threadIdx.y) * 64) + k_inner) + 56)];
      B_shared_local[0] = B_shared[((((int)threadIdx.x) * 64) + k_inner)];
      B_shared_local[1] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 8)];
      B_shared_local[2] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 16)];
      B_shared_local[3] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 24)];
      B_shared_local[4] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 32)];
      B_shared_local[5] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 40)];
      B_shared_local[6] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 48)];
      B_shared_local[7] = B_shared[(((((int)threadIdx.x) * 64) + k_inner) + 56)];
      for (int i_c = 0; i_c < 8; ++i_c) {
        T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
        T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
        T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
        T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
        T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
        T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
        T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
        T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
      }
    }
  }
  for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
    T_batch_matmul_NT[((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
    T_batch_matmul_NT[(((((((((int)blockIdx.z) * 262144) + (((int)blockIdx.y) * 32768)) + (((int)threadIdx.y) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
  }
}


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) conv2d_0(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel);
extern "C" __global__ void __launch_bounds__(16) conv2d_0(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nchw_local[20];
  __shared__ float pad_temp_shared[18];
  __shared__ float kernel_shared[128];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[(((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2))];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[(((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2))];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[(((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18))];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 9)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 576)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 585)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1152)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1161)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1728)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1737)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 1)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 1)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 10)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 577)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 586)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1153)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1162)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1729)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1738)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 2)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 2)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 2)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 11)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 578)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 587)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1154)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1163)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1730)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1739)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 112)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 112)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 3)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 12)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 579)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 588)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1155)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1164)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1731)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1740)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 113)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 113)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 4)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 13)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 580)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 589)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1156)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1165)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1732)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1741)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 114)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 114)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 5)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 14)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 581)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 590)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1157)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1166)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1733)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1742)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 224)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 224)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 6)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 15)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 582)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 591)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1158)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1167)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1734)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1743)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 225)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 225)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 7)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 16)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 583)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 592)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1159)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1168)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1735)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1744)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    __syncthreads();
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[(((int)threadIdx.z) * 2)] = data[((((((rc_outer * 25088) + (((((int)threadIdx.z) * 2) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + (((((int)threadIdx.z) * 2) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 226)];
    }
    if (((int)threadIdx.z) < 9) {
      pad_temp_shared[((((int)threadIdx.z) * 2) + 1)] = data[((((((rc_outer * 25088) + ((((((int)threadIdx.z) * 2) + 1) / 9) * 12544)) + (((int)blockIdx.y) * 1120)) + ((((((int)threadIdx.z) * 2) + 1) % 9) * 112)) + (((int)blockIdx.x) * 2)) + 226)];
    }
    kernel_shared[(((int)threadIdx.z) * 8)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 8)];
    kernel_shared[((((int)threadIdx.z) * 8) + 1)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 17)];
    kernel_shared[((((int)threadIdx.z) * 8) + 2)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 584)];
    kernel_shared[((((int)threadIdx.z) * 8) + 3)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 593)];
    kernel_shared[((((int)threadIdx.z) * 8) + 4)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1160)];
    kernel_shared[((((int)threadIdx.z) * 8) + 5)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1169)];
    kernel_shared[((((int)threadIdx.z) * 8) + 6)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1736)];
    kernel_shared[((((int)threadIdx.z) * 8) + 7)] = kernel[((((((int)blockIdx.z) * 36864) + (((int)threadIdx.z) * 2304)) + (rc_outer * 18)) + 1745)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[8] * kernel_shared[(((int)threadIdx.z) * 4)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 2)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.z) * 4) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 1)]));
    conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
    conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 3)]));
    conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.z) * 4) + 67)]));
  }
  conv2d_nchw[((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x))] = conv2d_nchw_local[0];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 96800)] = conv2d_nchw_local[10];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 55)] = conv2d_nchw_local[2];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 96855)] = conv2d_nchw_local[12];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 110)] = conv2d_nchw_local[4];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 96910)] = conv2d_nchw_local[14];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 165)] = conv2d_nchw_local[6];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 96965)] = conv2d_nchw_local[16];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 220)] = conv2d_nchw_local[8];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 97020)] = conv2d_nchw_local[18];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 3025)] = conv2d_nchw_local[1];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 99825)] = conv2d_nchw_local[11];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 3080)] = conv2d_nchw_local[3];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 99880)] = conv2d_nchw_local[13];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 3135)] = conv2d_nchw_local[5];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 99935)] = conv2d_nchw_local[15];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 3190)] = conv2d_nchw_local[7];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 99990)] = conv2d_nchw_local[17];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 3245)] = conv2d_nchw_local[9];
  conv2d_nchw[(((((((int)blockIdx.z) * 193600) + (((int)threadIdx.z) * 6050)) + (((int)blockIdx.y) * 275)) + ((int)blockIdx.x)) + 100045)] = conv2d_nchw_local[19];
}

