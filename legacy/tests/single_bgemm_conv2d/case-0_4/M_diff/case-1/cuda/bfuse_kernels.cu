
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(152) void bgemm_0_conv2d_4_fused_bfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_4_conv2d_nchw_, float *__restrict conv2d_4_data_, float *__restrict conv2d_4_kernel_)
{
  /*
   * KernelID_ means...
   * 0: bgemm_0
   * 1: conv2d_4
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x < 131072) && ((int)blockIdx.x % 2 >= 0) && ((int)blockIdx.x % 2 < 1))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 2) * 1 + ((int)blockIdx.x % 2 - 0);
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x < 131072) && ((int)blockIdx.x % 2 >= 1) && ((int)blockIdx.x % 2 < 2))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 2) * 1 + ((int)blockIdx.x % 2 - 1);
    KernelID_  = 1;
    gridDim_x_ = 3;
    gridDim_y_ = 57;
    gridDim_z_ = 2048;
    blockDim_x_ = 19;
    blockDim_y_ = 1;
    blockDim_z_ = 8;
  }
  else if ((int)blockIdx.x >= 131072 && (int)blockIdx.x < 131072)
  {
    NewBlockIdx_ = (int)blockIdx.x - 131072 + 65536;
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 131072 && (int)blockIdx.x < 415744)
  {
    NewBlockIdx_ = (int)blockIdx.x - 131072 + 65536;
    KernelID_  = 1;
    gridDim_x_ = 3;
    gridDim_y_ = 57;
    gridDim_z_ = 2048;
    blockDim_x_ = 19;
    blockDim_y_ = 1;
    blockDim_z_ = 8;
  }
  blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
  blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
  blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
  threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
  threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
  threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

  typedef struct bgemm_0 {
     float A_shared[512];
     float B_shared[512];
  } bgemm_0Ty_;
  typedef struct conv2d_4 {
     float pad_temp_shared[450];
     float kernel_shared[576];
  } conv2d_4Ty_;
  typedef union ShrdUnion {
    bgemm_0Ty_ bgemm_0;
    conv2d_4Ty_ conv2d_4;
  } ShrdUnionTy_;

  __shared__ ShrdUnionTy_ SU_;

  // bgemm_0
  if ((KernelID_ == 0) && ((int)threadIdx.x < 64))
  {
      float T_batch_matmul_NT_local[64];
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          __syncthreads();
          SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          SU_.bgemm_0.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = SU_.bgemm_0.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
  }
  // conv2d_4
  else if ((KernelID_ == 1) && ((int)threadIdx.x < 152))
  {
      float conv2d_nchw_local[4];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
          __syncthreads();
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              if (((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) < 150) {
                  SU_.conv2d_4.pad_temp_shared[(((((int)threadIdx_z_) * 57) + (((int)threadIdx_x_) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = conv2d_4_data_[((((((((((int)blockIdx_z_) >> 1) * 824464) + (rc_outer * 103058)) + ((((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) / 75) * 51529)) + (((int)blockIdx_y_) * 908)) + (((((((int)threadIdx_z_) * 19) + ((int)threadIdx_x_)) % 75) / 25) * 227)) + (((int)blockIdx_x_) * 76)) + ((((((int)threadIdx_z_) * 57) + (((int)threadIdx_x_) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 75))];
              }
          }
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) {
              if (((((int)threadIdx_x_) / 18) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 18) {
                      SU_.conv2d_4.kernel_shared[(((((int)threadIdx_z_) * 72) + (((int)threadIdx_x_) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1)] = conv2d_4_kernel_[((((((((int)blockIdx_z_) & 1) * 4608) + (((int)threadIdx_z_) * 576)) + ((((((int)threadIdx_x_) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 >> 1)) / 9) * 144)) + (rc_outer * 18)) + (((((int)threadIdx_x_) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) % 18))];
                  }
              }
          }
          __syncthreads();
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
                  for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
                      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_4.pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * SU_.conv2d_4.kernel_shared[((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
                      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_4.pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * SU_.conv2d_4.kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
                      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_4.pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * SU_.conv2d_4.kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
                      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_4.pad_temp_shared[((((rc_inner * 225) + (ry_inner * 75)) + (((int)threadIdx_x_) * 4)) + rx_inner)] * SU_.conv2d_4.kernel_shared[(((((((int)threadIdx_z_) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
                  }
              }
          }
      }
      conv2d_4_conv2d_nchw_[(((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 25992)] = conv2d_nchw_local[1];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 51984)] = conv2d_nchw_local[2];
      conv2d_4_conv2d_nchw_[((((((((int)blockIdx_z_) * 103968) + (((int)threadIdx_z_) * 3249)) + (((int)blockIdx_y_) * 57)) + (((int)blockIdx_x_) * 19)) + ((int)threadIdx_x_)) + 77976)] = conv2d_nchw_local[3];
  }
}
