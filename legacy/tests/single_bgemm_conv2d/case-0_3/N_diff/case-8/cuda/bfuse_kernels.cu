
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(104) void bgemm_0_conv2d_3_fused_bfuse(float *__restrict bgemm_0_A_, float *__restrict bgemm_0_B_, float *__restrict bgemm_0_T_batch_matmul_NT_, float *__restrict conv2d_3_conv2d_nchw_, float *__restrict conv2d_3_data_, float *__restrict conv2d_3_kernel_)
{
  /*
   * KernelID_ means...
   * 0: bgemm_0
   * 1: conv2d_3
   */
  int gridDim_x_, gridDim_y_, gridDim_z_;
  int blockIdx_x_, blockIdx_y_, blockIdx_z_;
  int blockDim_x_, blockDim_y_, blockDim_z_;
  int threadIdx_x_, threadIdx_y_, threadIdx_z_;
  int NewBlockIdx_;
  int KernelID_;
  
  if (((int)blockIdx.x < 131072) && ((int)blockIdx.x % 16 >= 0) && ((int)blockIdx.x % 16 < 8))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 16) * 8 + ((int)blockIdx.x % 16 - 0);
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if (((int)blockIdx.x < 131072) && ((int)blockIdx.x % 16 >= 8) && ((int)blockIdx.x % 16 < 16))
  {
    NewBlockIdx_ = int((int)blockIdx.x / 16) * 8 + ((int)blockIdx.x % 16 - 8);
    KernelID_  = 1;
    gridDim_x_ = 2;
    gridDim_y_ = 13;
    gridDim_z_ = 4096;
    blockDim_x_ = 13;
    blockDim_y_ = 1;
    blockDim_z_ = 8;
  }
  else if ((int)blockIdx.x >= 131072 && (int)blockIdx.x < 131072)
  {
    NewBlockIdx_ = (int)blockIdx.x - 131072 + 65536;
    KernelID_  = 0;
    gridDim_x_ = 8;
    gridDim_y_ = 8;
    gridDim_z_ = 1024;
    blockDim_x_ = 8;
    blockDim_y_ = 8;
    blockDim_z_ = 1;
  }
  else if ((int)blockIdx.x >= 131072 && (int)blockIdx.x < 172032)
  {
    NewBlockIdx_ = (int)blockIdx.x - 131072 + 65536;
    KernelID_  = 1;
    gridDim_x_ = 2;
    gridDim_y_ = 13;
    gridDim_z_ = 4096;
    blockDim_x_ = 13;
    blockDim_y_ = 1;
    blockDim_z_ = 8;
  }
  blockIdx_x_ = NewBlockIdx_ % gridDim_x_;
  blockIdx_y_ = NewBlockIdx_ / gridDim_x_ % gridDim_y_;
  blockIdx_z_ = NewBlockIdx_ / (gridDim_x_ * gridDim_y_);
  threadIdx_x_ = (int)threadIdx.x % blockDim_x_;
  threadIdx_y_ = (int)threadIdx.x / blockDim_x_ % blockDim_y_;
  threadIdx_z_ = (int)threadIdx.x / (blockDim_x_ * blockDim_y_);

  typedef struct bgemm_0 {
     float A_shared[512];
     float B_shared[512];
  } bgemm_0Ty_;
  typedef struct conv2d_3 {
     float pad_temp_shared[208];
     float kernel_shared[512];
  } conv2d_3Ty_;
  typedef union ShrdUnion {
    bgemm_0Ty_ bgemm_0;
    conv2d_3Ty_ conv2d_3;
  } ShrdUnionTy_;

  __shared__ ShrdUnionTy_ SU_;

  // bgemm_0
  if ((KernelID_ == 0) && ((int)threadIdx.x < 64))
  {
      float T_batch_matmul_NT_local[64];
      float A_shared_local[8];
      float B_shared_local[8];
      for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
          for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
              T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.F;
          }
      }
      for (int k_outer = 0; k_outer < 8; ++k_outer) {
          __syncthreads();
          SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_A_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_A_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_y_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          SU_.bgemm_0.B_shared[((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_))] = bgemm_0_B_[(((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_))];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 8)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 64)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 16)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 128)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 24)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 192)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 32)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 256)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 40)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 320)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 48)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 384)];
          SU_.bgemm_0.B_shared[(((((int)threadIdx_y_) * 64) + ((int)threadIdx_x_)) + 56)] = bgemm_0_B_[((((((((int)blockIdx_z_) * 32768) + (((int)blockIdx_x_) * 4096)) + (((int)threadIdx_y_) * 512)) + (k_outer * 8)) + ((int)threadIdx_x_)) + 448)];
          __syncthreads();
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
              A_shared_local[0] = SU_.bgemm_0.A_shared[((((int)threadIdx_y_) * 64) + k_inner)];
              A_shared_local[1] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 8)];
              A_shared_local[2] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 16)];
              A_shared_local[3] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 24)];
              A_shared_local[4] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 32)];
              A_shared_local[5] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 40)];
              A_shared_local[6] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 48)];
              A_shared_local[7] = SU_.bgemm_0.A_shared[(((((int)threadIdx_y_) * 64) + k_inner) + 56)];
              B_shared_local[0] = SU_.bgemm_0.B_shared[((((int)threadIdx_x_) * 64) + k_inner)];
              B_shared_local[1] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 8)];
              B_shared_local[2] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 16)];
              B_shared_local[3] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 24)];
              B_shared_local[4] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 32)];
              B_shared_local[5] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 40)];
              B_shared_local[6] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 48)];
              B_shared_local[7] = SU_.bgemm_0.B_shared[(((((int)threadIdx_x_) * 64) + k_inner) + 56)];
              for (int i_c = 0; i_c < 8; ++i_c) {
                  T_batch_matmul_NT_local[(i_c * 8)] = (T_batch_matmul_NT_local[(i_c * 8)] + (A_shared_local[i_c] * B_shared_local[0]));
                  T_batch_matmul_NT_local[((i_c * 8) + 1)] = (T_batch_matmul_NT_local[((i_c * 8) + 1)] + (A_shared_local[i_c] * B_shared_local[1]));
                  T_batch_matmul_NT_local[((i_c * 8) + 2)] = (T_batch_matmul_NT_local[((i_c * 8) + 2)] + (A_shared_local[i_c] * B_shared_local[2]));
                  T_batch_matmul_NT_local[((i_c * 8) + 3)] = (T_batch_matmul_NT_local[((i_c * 8) + 3)] + (A_shared_local[i_c] * B_shared_local[3]));
                  T_batch_matmul_NT_local[((i_c * 8) + 4)] = (T_batch_matmul_NT_local[((i_c * 8) + 4)] + (A_shared_local[i_c] * B_shared_local[4]));
                  T_batch_matmul_NT_local[((i_c * 8) + 5)] = (T_batch_matmul_NT_local[((i_c * 8) + 5)] + (A_shared_local[i_c] * B_shared_local[5]));
                  T_batch_matmul_NT_local[((i_c * 8) + 6)] = (T_batch_matmul_NT_local[((i_c * 8) + 6)] + (A_shared_local[i_c] * B_shared_local[6]));
                  T_batch_matmul_NT_local[((i_c * 8) + 7)] = (T_batch_matmul_NT_local[((i_c * 8) + 7)] + (A_shared_local[i_c] * B_shared_local[7]));
              }
          }
      }
      for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
          bgemm_0_T_batch_matmul_NT_[((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8))] = T_batch_matmul_NT_local[(i_inner_inner * 8)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 1)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 1)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 2)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 2)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 3)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 3)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 4)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 4)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 5)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 5)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 6)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 6)];
          bgemm_0_T_batch_matmul_NT_[(((((((((int)blockIdx_z_) * 262144) + (((int)blockIdx_y_) * 32768)) + (((int)threadIdx_y_) * 4096)) + (i_inner_inner * 512)) + (((int)blockIdx_x_) * 64)) + (((int)threadIdx_x_) * 8)) + 7)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + 7)];
      }
  }
  // conv2d_3
  else if ((KernelID_ == 1) && ((int)threadIdx.x < 104))
  {
      float conv2d_nchw_local[16];
      conv2d_nchw_local[0] = 0.F;
      conv2d_nchw_local[2] = 0.F;
      conv2d_nchw_local[4] = 0.F;
      conv2d_nchw_local[6] = 0.F;
      conv2d_nchw_local[8] = 0.F;
      conv2d_nchw_local[10] = 0.F;
      conv2d_nchw_local[12] = 0.F;
      conv2d_nchw_local[14] = 0.F;
      conv2d_nchw_local[1] = 0.F;
      conv2d_nchw_local[3] = 0.F;
      conv2d_nchw_local[5] = 0.F;
      conv2d_nchw_local[7] = 0.F;
      conv2d_nchw_local[9] = 0.F;
      conv2d_nchw_local[11] = 0.F;
      conv2d_nchw_local[13] = 0.F;
      conv2d_nchw_local[15] = 0.F;
      for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
          for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
              __syncthreads();
              SU_.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13))];
              SU_.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[(((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13))];
              SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3))];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3))];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3))];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3))];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[(((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3))];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU_.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 1)];
              SU_.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 1)];
              SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 1)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 1)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 1)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              __syncthreads();
              SU_.conv2d_3.pad_temp_shared[((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2))] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + (((((int)threadIdx_x_) * 2) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + ((((int)threadIdx_x_) * 2) % 13)) + 2)];
              SU_.conv2d_3.pad_temp_shared[(((((int)threadIdx_z_) * 26) + (((int)threadIdx_x_) * 2)) + 1)] = conv2d_3_data_[((((((((((((int)blockIdx_z_) >> 2) * 100352) + (rc_outer * 6272)) + (((int)threadIdx_z_) * 784)) + (((int)blockIdx_y_) * 56)) + ((((((int)threadIdx_x_) * 2) + 1) / 13) * 28)) + (ry_outer * 28)) + (((int)blockIdx_x_) * 13)) + (((((int)threadIdx_x_) * 2) + 1) % 13)) + 2)];
              SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5))] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + (((((int)threadIdx_x_) * 5) >> 3) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx_x_) * 5) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 1)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 1) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 1) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 2)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 2) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 2) & 7) * 9)) + (ry_outer * 3)) + 2)];
              SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 3)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 3) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 3) & 7) * 9)) + (ry_outer * 3)) + 2)];
              if (((((((int)threadIdx_x_) * 5) + 4) >> 6) + ((int)threadIdx_z_)) < 8) {
                  if (((int)threadIdx_x_) < 12) {
                      SU_.conv2d_3.kernel_shared[(((((int)threadIdx_z_) * 64) + (((int)threadIdx_x_) * 5)) + 4)] = conv2d_3_kernel_[((((((((((int)blockIdx_z_) & 3) * 73728) + (((int)threadIdx_z_) * 9216)) + ((((((int)threadIdx_x_) * 5) + 4) >> 3) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx_x_) * 5) + 4) & 7) * 9)) + (ry_outer * 3)) + 2)];
                  }
              }
              __syncthreads();
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[((int)threadIdx_x_)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[(((int)threadIdx_z_) * 8)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 64)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 128)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 192)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 256)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 320)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 384)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 13)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 448)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 26)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 1)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 65)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 129)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 193)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 257)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 321)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 385)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 39)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 449)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 52)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 2)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 66)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 130)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 194)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 258)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 322)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 386)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 65)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 450)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 78)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 3)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 67)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 131)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 195)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 259)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 323)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 387)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 91)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 451)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 104)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 4)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 68)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 132)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 196)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 260)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 324)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 388)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 117)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 452)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 130)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 5)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 69)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 133)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 197)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 261)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 325)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 389)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 143)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 453)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 156)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 6)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 70)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 134)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 198)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 262)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 326)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 390)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 169)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 454)]));
              conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 182)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
              conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 7)]));
              conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 71)]));
              conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 135)]));
              conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 199)]));
              conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 263)]));
              conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 327)]));
              conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 391)]));
              conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (SU_.conv2d_3.pad_temp_shared[(((int)threadIdx_x_) + 195)] * SU_.conv2d_3.kernel_shared[((((int)threadIdx_z_) * 8) + 455)]));
          }
      }
      conv2d_3_conv2d_nchw_[(((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_))] = conv2d_nchw_local[0];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5408)] = conv2d_nchw_local[2];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10816)] = conv2d_nchw_local[4];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16224)] = conv2d_nchw_local[6];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21632)] = conv2d_nchw_local[8];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27040)] = conv2d_nchw_local[10];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32448)] = conv2d_nchw_local[12];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37856)] = conv2d_nchw_local[14];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 26)] = conv2d_nchw_local[1];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 5434)] = conv2d_nchw_local[3];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 10842)] = conv2d_nchw_local[5];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 16250)] = conv2d_nchw_local[7];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 21658)] = conv2d_nchw_local[9];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 27066)] = conv2d_nchw_local[11];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 32474)] = conv2d_nchw_local[13];
      conv2d_3_conv2d_nchw_[((((((((int)blockIdx_z_) * 43264) + (((int)threadIdx_z_) * 676)) + (((int)blockIdx_y_) * 52)) + (((int)blockIdx_x_) * 13)) + ((int)threadIdx_x_)) + 37882)] = conv2d_nchw_local[15];
  }
}
