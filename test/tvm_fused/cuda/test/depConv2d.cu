
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) depConv2d(float* __restrict__ DepthwiseConv2d, float* __restrict__ data, float* __restrict__ kernel) {
  __shared__ float PaddedInput_shared[3249];
  __shared__ float kernel_shared[9];
  float PaddedInput_shared_local[45];
  float kernel_shared_local[9];
  float DepthwiseConv2d_local[7];
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 30; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) < 3249) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 4) + ((int)threadIdx.y)) < 117) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x))] = (((57 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) % 57))) ? data[((((((int)blockIdx.z) * 3136) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) / 57) * 56)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) % 57)) - 57)] : 0.000000e+00f);
      }
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      kernel_shared[((((int)threadIdx.y) * 28) + ((int)threadIdx.x))] = kernel[(((((int)threadIdx.y) * 28) + ((((int)blockIdx.z) & 127) * 9)) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax2 = 0; ax2 < 15; ++ax2) {
    #pragma unroll
    for (int ax3 = 0; ax3 < 3; ++ax3) {
      PaddedInput_shared_local[((ax2 * 3) + ax3)] = PaddedInput_shared[((((((int)threadIdx.y) * 798) + (ax2 * 57)) + (((int)threadIdx.x) * 2)) + ax3)];
    }
  }
  #pragma unroll
  for (int ax2_1 = 0; ax2_1 < 3; ++ax2_1) {
    #pragma unroll
    for (int ax3_1 = 0; ax3_1 < 3; ++ax3_1) {
      kernel_shared_local[((ax2_1 * 3) + ax3_1)] = kernel_shared[((ax2_1 * 3) + ax3_1)];
    }
  }
  #pragma unroll
  for (int i_c = 0; i_c < 7; ++i_c) {
    DepthwiseConv2d_local[i_c] = 0.000000e+00f;
    #pragma unroll
    for (int di = 0; di < 3; ++di) {
      #pragma unroll
      for (int dj = 0; dj < 3; ++dj) {
        DepthwiseConv2d_local[i_c] = (DepthwiseConv2d_local[i_c] + (PaddedInput_shared_local[(((i_c * 6) + (di * 3)) + dj)] * kernel_shared_local[((di * 3) + dj)]));
      }
    }
  }
  #pragma unroll
  for (int i_inner_inner_inner = 0; i_inner_inner_inner < 7; ++i_inner_inner_inner) {
    DepthwiseConv2d[((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 196)) + (i_inner_inner_inner * 28)) + ((int)threadIdx.x))] = DepthwiseConv2d_local[i_inner_inner_inner];
  }
}

