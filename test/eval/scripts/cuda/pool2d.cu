
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ pool_avg, float* __restrict__ pool_sum) {
  pool_avg[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * 1.111111e-01f);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_sum) {
  pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + (((((1 <= (((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 1225) / 35) + rv0)) && ((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 1225) / 35) + rv0) < 36)) && (1 <= (rv1 + (((((int)blockIdx.x) * 29) + ((int)threadIdx.x)) % 35)))) && ((rv1 + (((((int)blockIdx.x) * 29) + ((int)threadIdx.x)) % 35)) < 36)) ? data[(((((((int)blockIdx.x) * 64) + (rv0 * 35)) + ((int)threadIdx.x)) + rv1) - 36)] : 0.000000e+00f));
    }
  }
}

