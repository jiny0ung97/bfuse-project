
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include "operation.h"

#define CHECK_CUDA(call)                                              \
  do                                                                  \
  {                                                                   \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess)                                       \
    {                                                                 \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)
//----------------------------------------------------------------------------------------------------
float *I1_gpu, *F1_gpu, *O1_gpu;
float *I2_gpu, *F2_gpu, *O2_gpu;

hipStream_t S1, S2;
//----------------------------------------------------------------------------------------------------
void initialize_kernel1(int *I_shape, int *F_shape, int *O_shape)
{
  CHECK_CUDA(hipMalloc(&I1_gpu, I_shape[0] * I_shape[1] * I_shape[2] * I_shape[3] * sizeof(float)));
  CHECK_CUDA(hipMalloc(&F1_gpu, F_shape[0] * F_shape[1] * F_shape[2] * F_shape[3] * sizeof(float)));
  CHECK_CUDA(hipMalloc(&O1_gpu, O_shape[0] * O_shape[1] * O_shape[2] * O_shape[3] * sizeof(float)));

  CHECK_CUDA(hipStreamCreate(&S1));
}
//----------------------------------------------------------------------------------------------------
void initialize_kernel2(int *I_shape, int *F_shape, int *O_shape)
{
  CHECK_CUDA(hipMalloc(&I2_gpu, I_shape[0] * I_shape[1] * I_shape[2] * I_shape[3] * sizeof(float)));
  CHECK_CUDA(hipMalloc(&F2_gpu, F_shape[0] * F_shape[1] * F_shape[2] * F_shape[3] * sizeof(float)));
  CHECK_CUDA(hipMalloc(&O2_gpu, O_shape[0] * O_shape[1] * O_shape[2] * O_shape[3] * sizeof(float)));

  CHECK_CUDA(hipStreamCreate(&S2));
}
//----------------------------------------------------------------------------------------------------
void finalize_kernel1()
{
  CHECK_CUDA(hipStreamDestroy(S1));

  CHECK_CUDA(hipFree(O1_gpu));
  CHECK_CUDA(hipFree(F1_gpu));
  CHECK_CUDA(hipFree(I1_gpu));
}
//----------------------------------------------------------------------------------------------------
void finalize_kernel2()
{
  CHECK_CUDA(hipStreamDestroy(S2));

  CHECK_CUDA(hipFree(O2_gpu));
  CHECK_CUDA(hipFree(F2_gpu));
  CHECK_CUDA(hipFree(I2_gpu));
}
//----------------------------------------------------------------------------------------------------
void run_kernel1(float *I, float *F, float *O, int *I_shape, int *F_shape, int *O_shape,
                 void (*func)(float*, float*, float*), unsigned int *gridDim, unsigned int *blockDim, size_t K)
{
  // GPU Memory copy (H2D)
  CHECK_CUDA(hipMemcpy(I1_gpu, I, I_shape[0] * I_shape[1] * I_shape[2] * I_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F1_gpu, F, F_shape[0] * F_shape[1] * F_shape[2] * F_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  // Kernel Launch
  dim3 GridDim{gridDim[0], gridDim[1], gridDim[2]};
  dim3 BlockDim{blockDim[0], blockDim[1], blockDim[2]};
  
  switch (K)
  {
  case 0:
    func<<<GridDim, BlockDim>>>(I1_gpu, F1_gpu, O1_gpu);
    break;
  }

  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());

  // GPU Memory copy (D2H)
  CHECK_CUDA(hipMemcpy(O, O1_gpu, O_shape[0] * O_shape[1] * O_shape[2] * O_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
}
//----------------------------------------------------------------------------------------------------
void run_kernel2(float *I, float *F, float *O, int *I_shape, int *F_shape, int *O_shape,
                 void (*func)(float*, float*, float*), unsigned int *gridDim, unsigned int *blockDim, size_t K)
{
  // GPU Memory copy (H2D)
  CHECK_CUDA(hipMemcpy(I2_gpu, I, I_shape[0] * I_shape[1] * I_shape[2] * I_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F2_gpu, F, F_shape[0] * F_shape[1] * F_shape[2] * F_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  // Kernel Launch
  dim3 GridDim{gridDim[0], gridDim[1], gridDim[2]};
  dim3 BlockDim{blockDim[0], blockDim[1], blockDim[2]};
  
  switch (K)
  {
  case 0:
    func<<<GridDim, BlockDim>>>(O2_gpu, I2_gpu, F2_gpu);
    break;
  }


  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());

  // GPU Memory copy (D2H)
  CHECK_CUDA(hipMemcpy(O, O2_gpu, O_shape[0] * O_shape[1] * O_shape[2] * O_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
}
//----------------------------------------------------------------------------------------------------
void run_parallel(float *I1, float *F1, float *O1, float *I2, float *F2, float *O2,
                  int *I1_shape, int *F1_shape, int *O1_shape, int *I2_shape, int *F2_shape, int *O2_shape,
                  void (*func1)(float*, float*, float*), void (*func2)(float*, float*, float*),
                  unsigned int *gridDim1, unsigned int *blockDim1, unsigned int *gridDim2, unsigned int *blockDim2, size_t K1, size_t K2)
{
  // GPU Memory copy (H2D)
  CHECK_CUDA(hipMemcpy(I1_gpu, I1, I1_shape[0] * I1_shape[1] * I1_shape[2] * I1_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F1_gpu, F1, F1_shape[0] * F1_shape[1] * F1_shape[2] * F1_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(I2_gpu, I2, I2_shape[0] * I2_shape[1] * I2_shape[2] * I2_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F2_gpu, F2, F2_shape[0] * F2_shape[1] * F2_shape[2] * F2_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  // Kernel Launch
  dim3 GridDim1{gridDim1[0], gridDim1[1], gridDim1[2]};
  dim3 BlockDim1{blockDim1[0], blockDim1[1], blockDim1[2]};

  dim3 GridDim2{gridDim2[0], gridDim2[1], gridDim2[2]};
  dim3 BlockDim2{blockDim2[0], blockDim2[1], blockDim2[2]};
  
  switch (K1)
  {
  case 0:
    switch (K2)
    {
    case 0:
      func1<<<GridDim1, BlockDim1, 0, S1>>>(I1_gpu, F1_gpu, O1_gpu);
      func2<<<GridDim2, BlockDim2, 0, S2>>>(O2_gpu, I2_gpu, F2_gpu);
      break;
    }
    break;
  }

  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());

  // GPU Memory copy (D2H)
  CHECK_CUDA(hipMemcpy(O1, O1_gpu, O1_shape[0] * O1_shape[1] * O1_shape[2] * O1_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(O2, O2_gpu, O2_shape[0] * O2_shape[1] * O2_shape[2] * O2_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
}
//----------------------------------------------------------------------------------------------------
void run_fuse(float *I1, float *F1, float *O1, float *I2, float *F2, float *O2,
              int *I1_shape, int *F1_shape, int *O1_shape, int *I2_shape, int *F2_shape, int *O2_shape,
              void (*func)(float*, float*, float*, float*, float*, float*), unsigned int *gridDim, unsigned int *blockDim, size_t K1, size_t K2)
{
  // GPU Memory copy (H2D)
  CHECK_CUDA(hipMemcpy(I1_gpu, I1, I1_shape[0] * I1_shape[1] * I1_shape[2] * I1_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F1_gpu, F1, F1_shape[0] * F1_shape[1] * F1_shape[2] * F1_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(I2_gpu, I2, I2_shape[0] * I2_shape[1] * I2_shape[2] * I2_shape[3] * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F2_gpu, F2, F2_shape[0] * F2_shape[1] * F2_shape[2] * F2_shape[3] * sizeof(float), hipMemcpyHostToDevice));

  // Kernel Launch
  dim3 GridDim{gridDim[0], gridDim[1], gridDim[2]};
  dim3 BlockDim{blockDim[0], blockDim[1], blockDim[2]};
  
  switch (K1)
  {
  case 0:
    switch (K2)
    {
    case 0:
      func<<<GridDim, BlockDim>>>(I1_gpu, F1_gpu, O1_gpu, O2_gpu, I2_gpu, F2_gpu);
      break;
    }
    break;
  }

  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());

  // GPU Memory copy (D2H)
  CHECK_CUDA(hipMemcpy(O1, O1_gpu, O1_shape[0] * O1_shape[1] * O1_shape[2] * O1_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(O2, O2_gpu, O2_shape[0] * O2_shape[1] * O2_shape[2] * O2_shape[3] * sizeof(float), hipMemcpyDeviceToHost));
}
//----------------------------------------------------------------------------------------------------
